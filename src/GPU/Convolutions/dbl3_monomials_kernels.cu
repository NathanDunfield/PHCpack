#include "hip/hip_runtime.h"
// The file dbl3_monomials_kernels.cu defines the kernels specified
// in dbl3_monomials_kernels.h.

/* The algorithm to compute forward, backward, and cross products
 * (denoted respectively by arrays f, b, and c)
 * for a monomial cff*x[0]*x[1]* .. *x[n-1] goes as follows:
 *
 * f[0] := cff*x[0]
 * for i from 1 to n-1 do f[i] := f[i-1]*x[i]
 * if n > 2 then
 *    b[0] := x[n-1]*x[n-2]
 *    for i from 1 to n-3 do b[i] := b[i-1]*x[n-2-i]
 *    b[n-3] := b[n-3]*cff
 *    if n = 3 then
 *       c[0] = f[0]*x[2]
 *    else
 *       for i from 0 to n-4 do c[i] := f[i]*b[n-4-i]
 *       c[n-3] := f[n-3]*x[n-1]
 *
 * Compared to the evaluation and differentiation of a product of variables,
 * (without coefficient cff), two extra multiplications must be done,
 * but this is better than n+1 multiplications with cff afterwards. */

#include "double_double_gpufun.cu"
#include "triple_double_gpufun.cu"
#include "dbl3_convolutions_kernels.h"
#include "dbl3_monomials_kernels.h"

__device__ void dbl3_convolute
 ( double *xhi, double *xmi, double *xlo,
   double *yhi, double *ymi, double *ylo,
   double *zhi, double *zmi, double *zlo, int dim, int k )
{
   double prdhi,prdmi,prdlo;

   // z[k] = x[0]*y[k];
   tdg_mul(xhi[0],xmi[0],xlo[0],yhi[k],ymi[k],ylo[k],&zhi[k],&zmi[k],&zlo[k]);

   for(int i=1; i<=k; i++) // z[k] = z[k] + x[i]*y[k-i];
   {
      tdg_mul(xhi[i],xmi[i],xlo[i],yhi[k-i],ymi[k-i],ylo[k-i],
              &prdhi,&prdmi,&prdlo);
      tdg_inc(&zhi[k],&zmi[k],&zlo[k],prdhi,prdmi,prdlo);
   }
}

__device__ void cmplx3_convolute
 ( double *xrehi, double *xremi, double *xrelo,
   double *ximhi, double *ximmi, double *ximlo,
   double *yrehi, double *yremi, double *yrelo,
   double *yimhi, double *yimmi, double *yimlo,
   double *zrehi, double *zremi, double *zrelo,
   double *zimhi, double *zimmi, double *zimlo, int dim, int k )
{
   double xrhi,xihi,yrhi,yihi,zrhi,zihi,acchi;
   double xrmi,ximi,yrmi,yimi,zrmi,zimi,accmi;
   double xrlo,xilo,yrlo,yilo,zrlo,zilo,acclo;

   // z[k] = x[0]*y[k]
   xrhi = xrehi[0]; xrmi = xremi[0]; xrlo = xrelo[0];
   xihi = ximhi[0]; ximi = ximmi[0]; xilo = ximlo[0];
   yrhi = yrehi[k]; yrmi = yremi[k]; yrlo = yrelo[k];
   yihi = yimhi[k]; yimi = yimmi[k]; yilo = yimlo[k];

   tdg_mul(xrhi,xrmi,xrlo,yrhi,yrmi,yrlo,&zrhi,&zrmi,&zrlo);     // zr = xr*yr
   tdg_mul(xihi,ximi,xilo,yihi,yimi,yilo,&acchi,&accmi,&acclo); // acc = xi*yi
   tdg_minus(&acchi,&accmi,&acclo);
   tdg_inc(&zrhi,&zrmi,&zrlo,acchi,accmi,acclo);         // zr = xr*yr - xi*yi
   tdg_mul(xrhi,xrmi,xrlo,yihi,yimi,yilo,&zihi,&zimi,&zilo);     // zi = xr*yi
   tdg_mul(xihi,ximi,xilo,yrhi,yrmi,yrlo,&acchi,&accmi,&acclo); // acc = xi*yr
   tdg_inc(&zihi,&zimi,&zilo,acchi,accmi,acclo);         // zr = xr*yr + xi*yi

   zrehi[k] = zrhi; zremi[k] = zrmi; zrelo[k] = zrlo;
   zimhi[k] = zihi; zimmi[k] = zimi; zimlo[k] = zilo;

   for(int i=1; i<=k; i++) // z[k] = z[k] + x[i]*y[k-i]
   {
      xrhi = xrehi[i]; xrmi = xremi[i]; xrlo = xrelo[i];
      xihi = ximhi[i]; ximi = ximmi[i]; xilo = ximlo[i];
      yrhi = yrehi[k-i]; yrmi = yremi[k-i]; yrlo = yrelo[k-i];
      yihi = yimhi[k-i]; yimi = yimmi[k-i]; yilo = yimlo[k-i];

      tdg_mul(xrhi,xrmi,xrlo,yrhi,yrmi,yrlo,&zrhi,&zrmi,&zrlo); // zr = xr*yr
      tdg_mul(xihi,ximi,xilo,yihi,yimi,yilo,&acchi,&accmi,&acclo);   // xi*yi
      tdg_minus(&acchi,&accmi,&acclo);
      tdg_inc(&zrhi,&zrmi,&zrlo,acchi,accmi,acclo);     // zr = xr*yr - xi*yi
      tdg_mul(xrhi,xrmi,xrlo,yihi,yimi,yilo,&zihi,&zimi,&zilo); // zi = xr*yi
      tdg_mul(xihi,ximi,xilo,yrhi,yrmi,yrlo,&acchi,&accmi,&acclo);   // xi*yr
      tdg_inc(&zihi,&zimi,&zilo,acchi,accmi,acclo);     // zr = xr*yr + xi*yi
      // zvre[k] += zr; zvim[k] += zi
      tdg_inc(&zrehi[k],&zremi[k],&zrelo[k],zrhi,zrmi,zrlo);
      tdg_inc(&zimhi[k],&zimmi[k],&zimlo[k],zihi,zimi,zilo);
   }
}

__global__ void GPU_dbl3_speel
 ( int nvr, int deg, int *idx, double *cffhi, double *cffmi, double *cfflo,
   double *inputhi, double *inputmi, double *inputlo,
   double *forwardhi, double *forwardmi, double *forwardlo,
   double *backwardhi, double *backwardmi, double *backwardlo,
   double *crosshi, double *crossmi, double *crosslo )
{
   const int k = threadIdx.x;
   const int deg1 = deg+1;
   int ix1,ix2;

   __shared__ double xvhi[td_shmemsize];
   __shared__ double xvmi[td_shmemsize];
   __shared__ double xvlo[td_shmemsize];
   __shared__ double yvhi[td_shmemsize];
   __shared__ double yvmi[td_shmemsize];
   __shared__ double yvlo[td_shmemsize];
   __shared__ double zvhi[td_shmemsize];
   __shared__ double zvmi[td_shmemsize];
   __shared__ double zvlo[td_shmemsize];
  
   xvhi[k] = cffhi[k]; xvmi[k] = cffmi[k]; xvlo[k] = cfflo[k];
   ix1 = idx[0]*deg1+k;
   yvhi[k] = inputhi[ix1]; yvmi[k] = inputmi[ix1]; yvlo[k] = inputlo[ix1]; 
   __syncthreads();
   dbl3_convolute(xvhi,xvmi,xvlo,yvhi,yvmi,yvlo,zvhi,zvmi,zvlo,deg1,k);
   forwardhi[k] = zvhi[k];
   forwardmi[k] = zvmi[k];
   forwardlo[k] = zvlo[k];                            // f[0] = cff*x[0]

   for(int i=1; i<nvr; i++)
   {
      xvhi[k] = zvhi[k]; xvmi[k] = zvmi[k]; xvlo[k] = zvlo[k];
      ix2 = idx[i]*deg1+k;
      yvhi[k] = inputhi[ix2]; yvmi[k] = inputmi[ix2]; yvlo[k] = inputlo[ix2];
      __syncthreads();
      dbl3_convolute(xvhi,xvmi,xvlo,yvhi,yvmi,yvlo,zvhi,zvmi,zvlo,deg1,k);
      forwardhi[i*deg1+k] = zvhi[k]; 
      forwardmi[i*deg1+k] = zvmi[k]; 
      forwardlo[i*deg1+k] = zvlo[k];                  // f[i] = f[i-1]*x[i]
   }
   if(nvr > 2)
   {
      ix1 = idx[nvr-1]*deg1+k;
      xvhi[k] = inputhi[ix1]; xvmi[k] = inputmi[ix1]; xvlo[k] = inputlo[ix1];
      ix2 = idx[nvr-2]*deg1+k;
      yvhi[k] = inputhi[ix2]; yvmi[k] = inputmi[ix2]; yvlo[k] = inputlo[ix2];
      __syncthreads();
      dbl3_convolute(xvhi,xvmi,xvlo,yvhi,yvmi,yvlo,zvhi,zvmi,zvlo,deg1,k);
      backwardhi[k] = zvhi[k];
      backwardmi[k] = zvmi[k];
      backwardlo[k] = zvlo[k];                       // b[0] = x[n-1]*x[n-2]
      for(int i=1; i<nvr-2; i++)
      {
         xvhi[k] = zvhi[k]; xvmi[k] = zvmi[k]; xvlo[k] = zvlo[k];
         ix2 = idx[nvr-2-i]*deg1+k;
         yvhi[k] = inputhi[ix2]; yvmi[k] = inputmi[ix2];
         yvlo[k] = inputlo[ix2];
         __syncthreads();
         dbl3_convolute(xvhi,xvmi,xvlo,yvhi,yvmi,yvlo,zvhi,zvmi,zvlo,deg1,k);
         backwardhi[i*deg1+k] = zvhi[k];
         backwardmi[i*deg1+k] = zvmi[k];
         backwardlo[i*deg1+k] = zvlo[k];             // b[i] = b[i-1]*x[n-2-i]
      }
      xvhi[k] = zvhi[k];  xvmi[k] = zvmi[k];  xvlo[k] = zvlo[k];
      yvhi[k] = cffhi[k]; yvmi[k] = cffmi[k]; yvlo[k] = cfflo[k];
      __syncthreads();
      dbl3_convolute(xvhi,xvmi,xvlo,yvhi,yvmi,yvlo,zvhi,zvmi,zvlo,deg1,k);
      ix2 = (nvr-3)*deg1+k;
      backwardhi[ix2] = zvhi[k];
      backwardmi[ix2] = zvmi[k];
      backwardlo[ix2] = zvlo[k];                    // b[n-3] = b[n-3]*cff

      if(nvr == 3)
      {
         xvhi[k] = forwardhi[k]; xvmi[k] = forwardmi[k];
         xvlo[k] = forwardlo[k];
         ix2 = idx[2]*deg1+k;
         yvhi[k] = inputhi[ix2]; yvmi[k] = inputmi[ix2]; 
         yvlo[k] = inputlo[ix2];
         __syncthreads();
         dbl3_convolute(xvhi,xvmi,xvlo,yvhi,yvmi,yvlo,zvhi,zvmi,zvlo,deg1,k);
         crosshi[k] = zvhi[k];
         crossmi[k] = zvmi[k];
         crosslo[k] = zvlo[k];                      // c[0] = f[0]*x[2]
      }
      else
      {
         for(int i=0; i<nvr-3; i++)
         {
            ix1 = i*deg1+k; 
            xvhi[k] = forwardhi[ix1]; xvmi[k] = forwardmi[ix1];
            xvlo[k] = forwardlo[ix1];
            ix2 = (nvr-4-i)*deg1+k;
            yvhi[k] = backwardhi[ix2]; yvmi[k] = backwardmi[ix2];
            yvlo[k] = backwardlo[ix2];
            __syncthreads();
            dbl3_convolute
               (xvhi,xvmi,xvlo,yvhi,yvmi,yvlo,zvhi,zvmi,zvlo,deg1,k);
            crosshi[i*deg1+k] = zvhi[k];
            crossmi[i*deg1+k] = zvmi[k];
            crosslo[i*deg1+k] = zvlo[k];            // c[i] = f[i]*b[n-4-i]
         }
         ix1 = (nvr-3)*deg1+k;
         xvhi[k] = forwardhi[ix1]; xvmi[k] = forwardmi[ix1];
         xvlo[k] = forwardlo[ix1];
         ix2 = idx[nvr-1]*deg1+k;
         yvhi[k] = inputhi[ix2]; yvmi[k] = inputmi[ix2];
         yvlo[k] = inputlo[ix2];
         __syncthreads();
         dbl3_convolute(xvhi,xvmi,xvlo,yvhi,yvmi,yvlo,zvhi,zvmi,zvlo,deg1,k);
         crosshi[(nvr-3)*deg1+k] = zvhi[k];
         crossmi[(nvr-3)*deg1+k] = zvmi[k];
         crosslo[(nvr-3)*deg1+k] = zvlo[k];         // c[n-3] = f[n-3]*x[n-1]
      }
   }
}

__global__ void GPU_cmplx3_speel
 ( int nvr, int deg, int *idx,
   double *cffrehi, double *cffremi, double *cffrelo,
   double *cffimhi, double *cffimmi, double *cffimlo,
   double *inputrehi, double *inputremi, double *inputrelo,
   double *inputimhi, double *inputimmi, double *inputimlo,
   double *forwardrehi, double *forwardremi, double *forwardrelo,
   double *forwardimhi, double *forwardimmi, double *forwardimlo,
   double *backwardrehi, double *backwardremi, double *backwardrelo,
   double *backwardimhi, double *backwardimmi, double *backwardimlo,
   double *crossrehi, double *crossremi, double *crossrelo,
   double *crossimhi, double *crossimmi, double *crossimlo )
{
}

void GPU_dbl3_evaldiff
 ( int BS, int dim, int nvr, int deg, int *idx,
   double *cffhi, double *cffmi, double *cfflo,
   double **inputhi, double **inputmi, double **inputlo,
   double **outputhi, double **outputmi, double **outputlo )
{
   const int deg1 = deg+1;            // length of all vectors
   double *inputhi_d;                 // inputhi_d is input on the device
   double *inputmi_d;                 // inputmi_d is input on the device
   double *inputlo_d;                 // inputlo_d is input on the device
   double *forwardhi_d;               // high forward products on the device
   double *forwardmi_d;               // middle forward products on the device
   double *forwardlo_d;               // low forward products on the device
   double *backwardhi_d;              // high backward products on the device
   double *backwardmi_d;              // middle backward products on the device
   double *backwardlo_d;              // low backward products on the device
   double *crosshi_d;                 // high cross products on the device
   double *crossmi_d;                 // middle cross products on the device
   double *crosslo_d;                 // low cross products on the device
   double *cffhi_d;                   // cffhi_d is cffhi on device
   double *cffmi_d;                   // cffmi_d is cffmi on device
   double *cfflo_d;                   // cfflo_d is cfflo on device
   int *idx_d;                        // idx_d is idx on device

   size_t szcff = deg1*sizeof(double);
   size_t szdim = dim*(deg1)*sizeof(double);
   size_t sznvr = nvr*(deg1)*sizeof(double);
   size_t sznvr2 = (nvr-2)*(deg1)*sizeof(double);
   size_t szidx = nvr*sizeof(int);

   hipMalloc((void**)&idx_d,szidx);
   hipMalloc((void**)&cffhi_d,szcff);
   hipMalloc((void**)&cffmi_d,szcff);
   hipMalloc((void**)&cfflo_d,szcff);
   hipMalloc((void**)&inputhi_d,szdim);
   hipMalloc((void**)&inputmi_d,szdim);
   hipMalloc((void**)&inputlo_d,szdim);
   hipMalloc((void**)&forwardhi_d,sznvr);
   hipMalloc((void**)&forwardmi_d,sznvr);
   hipMalloc((void**)&forwardlo_d,sznvr);
   hipMalloc((void**)&backwardhi_d,sznvr2);
   hipMalloc((void**)&backwardmi_d,sznvr2);
   hipMalloc((void**)&backwardlo_d,sznvr2);
   hipMalloc((void**)&crosshi_d,sznvr2);
   hipMalloc((void**)&crossmi_d,sznvr2);
   hipMalloc((void**)&crosslo_d,sznvr2);

   double *inputhi_h = new double[dim*(deg1)];
   double *inputmi_h = new double[dim*(deg1)];
   double *inputlo_h = new double[dim*(deg1)];
   int ix = 0;
   for(int i=0; i<dim; i++)
      for(int j=0; j<deg1; j++)
      {
         inputhi_h[ix] = inputhi[i][j];
         inputmi_h[ix] = inputmi[i][j];
         inputlo_h[ix++] = inputlo[i][j];
      }

   hipMemcpy(idx_d,idx,szidx,hipMemcpyHostToDevice);
   hipMemcpy(cffhi_d,cffhi,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffmi_d,cffmi,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cfflo_d,cfflo,szcff,hipMemcpyHostToDevice);
   hipMemcpy(inputhi_d,inputhi_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputmi_d,inputmi_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputlo_d,inputlo_h,szdim,hipMemcpyHostToDevice);

   if(BS = deg1)
   {
      GPU_dbl3_speel<<<1,BS>>>
         (nvr,deg,idx_d,cffhi_d,cffmi_d,cfflo_d,inputhi_d,inputmi_d,
          inputlo_d,forwardhi_d,forwardmi_d,forwardlo_d,backwardhi_d,
          backwardmi_d,backwardlo_d,crosshi_d,crossmi_d,crosslo_d);
   }
   double *forwardhi_h = new double[(deg1)*nvr];
   double *forwardmi_h = new double[(deg1)*nvr];
   double *forwardlo_h = new double[(deg1)*nvr];
   double *backwardhi_h = new double[(deg1)*(nvr-2)];
   double *backwardmi_h = new double[(deg1)*(nvr-2)];
   double *backwardlo_h = new double[(deg1)*(nvr-2)];
   double *crosshi_h = new double[(deg1)*(nvr-2)];
   double *crossmi_h = new double[(deg1)*(nvr-2)];
   double *crosslo_h = new double[(deg1)*(nvr-2)];
  
   hipMemcpy(forwardhi_h,forwardhi_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardmi_h,forwardmi_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardlo_h,forwardlo_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(backwardhi_h,backwardhi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardmi_h,backwardmi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardlo_h,backwardlo_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crosshi_h,crosshi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossmi_h,crossmi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crosslo_h,crosslo_d,sznvr2,hipMemcpyDeviceToHost);

   int offset = (nvr-1)*deg1;            // assign value of the monomial
   for(int i=0; i<deg1; i++)
   {
      outputhi[dim][i] = forwardhi_h[offset+i];
      outputmi[dim][i] = forwardmi_h[offset+i];
      outputlo[dim][i] = forwardlo_h[offset+i];
   }
   ix = idx[nvr-1];                      // derivative with respect to x[n-1]
   offset = (nvr-2)*deg1;
   for(int i=0; i<deg1; i++)
   {
      outputhi[ix][i] = forwardhi_h[offset+i];
      outputmi[ix][i] = forwardmi_h[offset+i];
      outputlo[ix][i] = forwardlo_h[offset+i];
   }
   ix = idx[0];                          // derivative with respect to x[0]
   offset = (nvr-3)*deg1;
   for(int i=0; i<deg1; i++)
   {
      outputhi[ix][i] = backwardhi_h[offset+i];
      outputmi[ix][i] = backwardmi_h[offset+i];
      outputlo[ix][i] = backwardlo_h[offset+i];
   }
   for(int k=1; k<nvr-1; k++)            // derivative with respect to x[k]
   {
      ix = idx[k]; offset = (k-1)*deg1;
      for(int i=0; i<deg1; i++)
      {
         outputhi[ix][i] = crosshi_h[offset+i];
         outputmi[ix][i] = crossmi_h[offset+i];
         outputlo[ix][i] = crosslo_h[offset+i];
      }
   }
}

void GPU_cmplx3_evaldiff
 ( int BS, int dim, int nvr, int deg, int *idx,
   double *cffrehi, double *cffremi, double *cffrelo,
   double *cffimhi, double *cffimmi, double *cffimlo,
   double **inputrehi, double **inputremi, double **inputrelo,
   double **inputimhi, double **inputimmi, double **inputimlo,
   double **outputrehi, double **outputremi, double **outputrelo,
   double **outputimhi, double **outputimmi, double **outputimlo )
{
}
