// The file dbl3_monomials_kernels.cu defines the kernels specified
// in dbl3_monomials_kernels.h.

/* The algorithm to compute forward, backward, and cross products
 * (denoted respectively by arrays f, b, and c)
 * for a monomial cff*x[0]*x[1]* .. *x[n-1] goes as follows:
 *
 * f[0] := cff*x[0]
 * for i from 1 to n-1 do f[i] := f[i-1]*x[i]
 * if n > 2 then
 *    b[0] := x[n-1]*x[n-2]
 *    for i from 1 to n-3 do b[i] := b[i-1]*x[n-2-i]
 *    b[n-3] := b[n-3]*cff
 *    if n = 3 then
 *       c[0] = f[0]*x[2]
 *    else
 *       for i from 0 to n-4 do c[i] := f[i]*b[n-4-i]
 *       c[n-3] := f[n-3]*x[n-1]
 *
 * Compared to the evaluation and differentiation of a product of variables,
 * (without coefficient cff), two extra multiplications must be done,
 * but this is better than n+1 multiplications with cff afterwards. */

#include "dbl3_convolutions_kernels.cu"
#include "dbl3_monomials_kernels.h"

void GPU_dbl3_speel
 ( int BS, int nvr, int deg, int *idx,
   double *cffhi, double *cffmi, double *cfflo,
   double *inputhi, double *inputmi, double *inputlo,
   double *forwardhi, double *forwardmi, double *forwardlo,
   double *backwardhi, double *backwardmi, double *backwardlo,
   double *crosshi, double *crossmi, double *crosslo )
{
   const int deg1 = deg+1;
   int ix1,ix2,ix3;

   ix1 = idx[0]*deg1;                                     // f[0] = cff*x[0]
   dbl3_convolute<<<1,BS>>>
      (cffhi,cffmi,cfflo,&inputhi[ix1],&inputmi[ix1],&inputlo[ix1],
       forwardhi,forwardmi,forwardlo,deg1);

   for(int i=1; i<nvr; i++)                            // f[i] = f[i-1]*x[i]
   {
      ix2 = idx[i]*deg1; ix3 = i*deg1; ix1 = ix3 - deg1;
      dbl3_convolute<<<1,BS>>>
         (&forwardhi[ix1],&forwardmi[ix1],&forwardlo[ix1],
          &inputhi[ix2],&inputmi[ix2],&inputlo[ix2],
          &forwardhi[ix3],&forwardmi[ix3],&forwardlo[ix3],deg1);
   }
   if(nvr > 2)
   {
      ix1 = idx[nvr-1]*deg1; ix2 = idx[nvr-2]*deg1;  // b[0] = x[n-1]*x[n-2]
      dbl3_convolute<<<1,BS>>>
         (&inputhi[ix1],&inputmi[ix1],&inputlo[ix1],
          &inputhi[ix2],&inputmi[ix2],&inputlo[ix2],
          backwardhi,backwardmi,backwardlo,deg1);

      for(int i=1; i<nvr-2; i++)                   // b[i] = b[i-1]*x[n-2-i]
      {
         ix2 = idx[nvr-2-i]*deg1; ix3 = i*deg1; ix1 = ix3 - deg1;
         dbl3_convolute<<<1,BS>>>
            (&backwardhi[ix1],&backwardmi[ix1],&backwardlo[ix1],
             &inputhi[ix2],&inputmi[ix2],&inputlo[ix2],
             &backwardhi[ix3],&backwardmi[ix3],&backwardlo[ix3],deg1);
      }
      ix3 = (nvr-3)*deg1; ix2 = (nvr-2)*deg1;         // b[n-2] = b[n-3]*cff
      dbl3_convolute<<<1,BS>>>
         (&backwardhi[ix3],&backwardmi[ix3],&backwardlo[ix3],
          cffhi,cffmi,cfflo,
          &backwardhi[ix2],&backwardmi[ix2],&backwardlo[ix2],deg1);

      if(nvr == 3)                                       // c[0] = f[0]*x[2]
      {
         ix2 = idx[2]*deg1;
         dbl3_convolute<<<1,BS>>>
            (forwardhi,forwardmi,forwardlo,
             &inputhi[ix2],&inputmi[ix2],&inputlo[ix2],
             crosshi,crossmi,crosslo,deg1);
      }
      else
      {
         for(int i=0; i<nvr-3; i++)                  // c[i] = f[i]*b[n-4-i]
         {
            ix1 = i*deg1; ix2 = (nvr-4-i)*deg1;
            dbl3_convolute<<<1,BS>>>
               (&forwardhi[ix1],&forwardmi[ix1],&forwardlo[ix1],
                &backwardhi[ix2],&backwardmi[ix2],&backwardlo[ix2],
                &crosshi[ix1],&crossmi[ix1],&crosslo[ix1],deg1);
         }
         ix1 = (nvr-3)*deg1; ix2 = idx[nvr-1]*deg1; // c[n-3] = f[n-3]*x[n-1]
         dbl3_convolute<<<1,BS>>>
            (&forwardhi[ix1],&forwardmi[ix1],&forwardlo[ix1],
             &inputhi[ix2],&inputmi[ix2],&inputlo[ix2],
             &crosshi[ix1],&crossmi[ix1],&crosslo[ix1],deg1);
      }
   }
}

void GPU_cmplx3_speel
 ( int BS, int nvr, int deg, int *idx,
   double *cffrehi, double *cffremi, double *cffrelo,
   double *cffimhi, double *cffimmi, double *cffimlo,
   double *inputrehi, double *inputremi, double *inputrelo,
   double *inputimhi, double *inputimmi, double *inputimlo,
   double *forwardrehi, double *forwardremi, double *forwardrelo,
   double *forwardimhi, double *forwardimmi, double *forwardimlo,
   double *backwardrehi, double *backwardremi, double *backwardrelo,
   double *backwardimhi, double *backwardimmi, double *backwardimlo,
   double *crossrehi, double *crossremi, double *crossrelo,
   double *crossimhi, double *crossimmi, double *crossimlo )
{
   const int deg1 = deg+1;
   int ix1,ix2,ix3;

   ix1 = idx[0]*deg1;                                     // f[0] = cff*x[0]
   cmplx3_convolute<<<1,BS>>>
      (cffrehi,cffremi,cffrelo,cffimhi,cffimmi,cffimlo,
       &inputrehi[ix1],&inputremi[ix1],&inputrelo[ix1],
       &inputimhi[ix1],&inputimmi[ix1],&inputimlo[ix1],
       forwardrehi,forwardremi,forwardrelo,
       forwardimhi,forwardimmi,forwardimlo,deg1); 

   for(int i=1; i<nvr; i++)                            // f[i] = f[i-i]*x[i]
   {
      ix2 = idx[i]*deg1; ix3 = i*deg1; ix1 = ix3 - deg1;
      cmplx3_convolute<<<1,BS>>>
         (&forwardrehi[ix1],&forwardremi[ix1],&forwardrelo[ix1],
          &forwardimhi[ix1],&forwardimmi[ix1],&forwardimlo[ix1],
          &inputrehi[ix2],&inputremi[ix2],&inputrelo[ix2],
          &inputimhi[ix2],&inputimmi[ix2],&inputimlo[ix2],
          &forwardrehi[ix3],&forwardremi[ix3],&forwardrelo[ix3],
          &forwardimhi[ix3],&forwardimmi[ix3],&forwardimlo[ix3],deg1);
   }
   if(nvr > 2)
   {
      ix1 = idx[nvr-1]*deg1; ix2 = idx[nvr-2]*deg1;  // b[0] = x[n-1]*x[n-2]
      cmplx3_convolute<<<1,BS>>>
         (&inputrehi[ix1],&inputremi[ix1],&inputrelo[ix1],
          &inputimhi[ix1],&inputimmi[ix1],&inputimlo[ix1],
          &inputrehi[ix2],&inputremi[ix2],&inputrelo[ix2],
          &inputimhi[ix2],&inputimmi[ix2],&inputimlo[ix2],
          backwardrehi,backwardremi,backwardrelo,
          backwardimhi,backwardimmi,backwardimlo,deg1);

      for(int i=1; i<nvr-2; i++)                   // b[i] = b[i-1]*x[n-2-i]
      {
         ix2 = idx[nvr-2-i]*deg1; ix3 = i*deg1; ix1 = ix3 - deg1;
         cmplx3_convolute<<<1,BS>>>
            (&backwardrehi[ix1],&backwardremi[ix1],&backwardrelo[ix1],
             &backwardimhi[ix1],&backwardimmi[ix1],&backwardimlo[ix1],
             &inputrehi[ix2],&inputremi[ix2],&inputrelo[ix2],
             &inputimhi[ix2],&inputimmi[ix2],&inputimlo[ix2],
             &backwardrehi[ix3],&backwardremi[ix3],&backwardrelo[ix3],
             &backwardimhi[ix3],&backwardimmi[ix3],&backwardimlo[ix3],deg1);
      }
      ix3 = (nvr-3)*deg1; ix2 = (nvr-2)*deg1;         // b[n-2] = b[n-3]*cff
      cmplx3_convolute<<<1,BS>>>
         (&backwardrehi[ix3],&backwardremi[ix3],&backwardrelo[ix3],
          &backwardimhi[ix3],&backwardimmi[ix3],&backwardimlo[ix3],
          cffrehi,cffremi,cffrelo,cffimhi,cffimmi,cffimlo,
          &backwardrehi[ix2],&backwardremi[ix2],&backwardrelo[ix2],
          &backwardimhi[ix2],&backwardimmi[ix2],&backwardimlo[ix2],deg1);

      if(nvr == 3)                                       // c[0] = f[0]*x[2]
      {
         ix2 = idx[2]*deg1;
         cmplx3_convolute<<<1,BS>>>
            (forwardrehi,forwardremi,forwardrelo,
             forwardimhi,forwardimmi,forwardimlo,
             &inputrehi[ix2],&inputremi[ix2],&inputrelo[ix2],
             &inputimhi[ix2],&inputimmi[ix2],&inputimlo[ix2],
             crossrehi,crossremi,crossrelo,
             crossimhi,crossimmi,crossimlo,deg1);
      }
      else
      {
         for(int i=0; i<nvr-3; i++)                  // c[i] = f[i]*b[n-4-i]
         {
            ix1 = i*deg1; ix2 = (nvr-4-i)*deg1;
            cmplx3_convolute<<<1,BS>>>
               (&forwardrehi[ix1],&forwardremi[ix1],&forwardrelo[ix1],
                &forwardimhi[ix1],&forwardimmi[ix1],&forwardimlo[ix1],
                &backwardrehi[ix2],&backwardremi[ix2],&backwardrelo[ix2],
                &backwardimhi[ix2],&backwardimmi[ix2],&backwardimlo[ix2],
                &crossrehi[ix1],&crossremi[ix1],&crossrelo[ix1],
                &crossimhi[ix1],&crossimmi[ix1],&crossimlo[ix1],deg1);
         }
         ix1 = (nvr-3)*deg1; ix2 = idx[nvr-1]*deg1; // c[n-3] = f[n-3]*x[n-1]
         cmplx3_convolute<<<1,BS>>>
            (&forwardrehi[ix1],&forwardremi[ix1],&forwardrelo[ix1],
             &forwardimhi[ix1],&forwardimmi[ix1],&forwardimlo[ix1],
             &inputrehi[ix2],&inputremi[ix2],&inputrelo[ix2],
             &inputimhi[ix2],&inputimmi[ix2],&inputimlo[ix2],
             &crossrehi[ix1],&crossremi[ix1],&crossrelo[ix1],
             &crossimhi[ix1],&crossimmi[ix1],&crossimlo[ix1],deg1);
      }
   }
}

void GPU_dbl3_evaldiff
 ( int BS, int dim, int nvr, int deg, int *idx,
   double *cffhi, double *cffmi, double *cfflo,
   double **inputhi, double **inputmi, double **inputlo,
   double **outputhi, double **outputmi, double **outputlo )
{
   const int deg1 = deg+1;            // length of all vectors
   double *inputhi_d;                 // inputhi_d is input on the device
   double *inputmi_d;                 // inputmi_d is input on the device
   double *inputlo_d;                 // inputlo_d is input on the device
   double *forwardhi_d;               // high forward products on the device
   double *forwardmi_d;               // middle forward products on the device
   double *forwardlo_d;               // low forward products on the device
   double *backwardhi_d;              // high backward products on the device
   double *backwardmi_d;              // middle backward products on the device
   double *backwardlo_d;              // low backward products on the device
   double *crosshi_d;                 // high cross products on the device
   double *crossmi_d;                 // middle cross products on the device
   double *crosslo_d;                 // low cross products on the device
   double *cffhi_d;                   // cffhi_d is cffhi on device
   double *cffmi_d;                   // cffmi_d is cffmi on device
   double *cfflo_d;                   // cfflo_d is cfflo on device

   size_t szcff = deg1*sizeof(double);
   size_t szdim = dim*(deg1)*sizeof(double);
   size_t sznvr = nvr*(deg1)*sizeof(double);
   size_t sznvr1 = (nvr-1)*(deg1)*sizeof(double);
   size_t sznvr2 = (nvr-2)*(deg1)*sizeof(double);

   hipMalloc((void**)&cffhi_d,szcff);
   hipMalloc((void**)&cffmi_d,szcff);
   hipMalloc((void**)&cfflo_d,szcff);
   hipMalloc((void**)&inputhi_d,szdim);
   hipMalloc((void**)&inputmi_d,szdim);
   hipMalloc((void**)&inputlo_d,szdim);
   hipMalloc((void**)&forwardhi_d,sznvr);
   hipMalloc((void**)&forwardmi_d,sznvr);
   hipMalloc((void**)&forwardlo_d,sznvr);
   hipMalloc((void**)&backwardhi_d,sznvr1);
   hipMalloc((void**)&backwardmi_d,sznvr1);
   hipMalloc((void**)&backwardlo_d,sznvr1);
   hipMalloc((void**)&crosshi_d,sznvr2);
   hipMalloc((void**)&crossmi_d,sznvr2);
   hipMalloc((void**)&crosslo_d,sznvr2);

   double *inputhi_h = new double[dim*(deg1)];
   double *inputmi_h = new double[dim*(deg1)];
   double *inputlo_h = new double[dim*(deg1)];
   int ix = 0;
   for(int i=0; i<dim; i++)
      for(int j=0; j<deg1; j++)
      {
         inputhi_h[ix] = inputhi[i][j];
         inputmi_h[ix] = inputmi[i][j];
         inputlo_h[ix++] = inputlo[i][j];
      }

   hipMemcpy(cffhi_d,cffhi,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffmi_d,cffmi,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cfflo_d,cfflo,szcff,hipMemcpyHostToDevice);
   hipMemcpy(inputhi_d,inputhi_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputmi_d,inputmi_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputlo_d,inputlo_h,szdim,hipMemcpyHostToDevice);

   if(BS == deg1)
   {
      GPU_dbl3_speel
         (BS,nvr,deg,idx,cffhi_d,cffmi_d,cfflo_d,inputhi_d,inputmi_d,
          inputlo_d,forwardhi_d,forwardmi_d,forwardlo_d,backwardhi_d,
          backwardmi_d,backwardlo_d,crosshi_d,crossmi_d,crosslo_d);
   }
   double *forwardhi_h = new double[(deg1)*nvr];
   double *forwardmi_h = new double[(deg1)*nvr];
   double *forwardlo_h = new double[(deg1)*nvr];
   double *backwardhi_h = new double[(deg1)*(nvr-1)];
   double *backwardmi_h = new double[(deg1)*(nvr-1)];
   double *backwardlo_h = new double[(deg1)*(nvr-1)];
   double *crosshi_h = new double[(deg1)*(nvr-2)];
   double *crossmi_h = new double[(deg1)*(nvr-2)];
   double *crosslo_h = new double[(deg1)*(nvr-2)];
  
   hipMemcpy(forwardhi_h,forwardhi_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardmi_h,forwardmi_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardlo_h,forwardlo_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(backwardhi_h,backwardhi_d,sznvr1,hipMemcpyDeviceToHost);
   hipMemcpy(backwardmi_h,backwardmi_d,sznvr1,hipMemcpyDeviceToHost);
   hipMemcpy(backwardlo_h,backwardlo_d,sznvr1,hipMemcpyDeviceToHost);
   hipMemcpy(crosshi_h,crosshi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossmi_h,crossmi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crosslo_h,crosslo_d,sznvr2,hipMemcpyDeviceToHost);

   int offset = (nvr-1)*deg1;            // assign value of the monomial
   for(int i=0; i<deg1; i++)
   {
      outputhi[dim][i] = forwardhi_h[offset+i];
      outputmi[dim][i] = forwardmi_h[offset+i];
      outputlo[dim][i] = forwardlo_h[offset+i];
   }
   ix = idx[nvr-1];                      // derivative with respect to x[n-1]
   offset = (nvr-2)*deg1;
   for(int i=0; i<deg1; i++)
   {
      outputhi[ix][i] = forwardhi_h[offset+i];
      outputmi[ix][i] = forwardmi_h[offset+i];
      outputlo[ix][i] = forwardlo_h[offset+i];
   }
   ix = idx[0];                          // derivative with respect to x[0]
   offset = (nvr-2)*deg1;
   for(int i=0; i<deg1; i++)
   {
      outputhi[ix][i] = backwardhi_h[offset+i];
      outputmi[ix][i] = backwardmi_h[offset+i];
      outputlo[ix][i] = backwardlo_h[offset+i];
   }
   for(int k=1; k<nvr-1; k++)            // derivative with respect to x[k]
   {
      ix = idx[k]; offset = (k-1)*deg1;
      for(int i=0; i<deg1; i++)
      {
         outputhi[ix][i] = crosshi_h[offset+i];
         outputmi[ix][i] = crossmi_h[offset+i];
         outputlo[ix][i] = crosslo_h[offset+i];
      }
   }
}

void GPU_cmplx3_evaldiff
 ( int BS, int dim, int nvr, int deg, int *idx,
   double *cffrehi, double *cffremi, double *cffrelo,
   double *cffimhi, double *cffimmi, double *cffimlo,
   double **inputrehi, double **inputremi, double **inputrelo,
   double **inputimhi, double **inputimmi, double **inputimlo,
   double **outputrehi, double **outputremi, double **outputrelo,
   double **outputimhi, double **outputimmi, double **outputimlo )
{
   const int deg1 = deg+1;          // length of all vectors
   double *inputrehi_d;             // inputrehi_d is inputrehi on the device
   double *inputremi_d;             // inputremi_d is inputremi on the device
   double *inputrelo_d;             // inputrelo_d is inputrelo on the device
   double *inputimhi_d;             // inputimhi_d is inputrehi on the device
   double *inputimmi_d;             // inputimmi_d is inputremi on the device
   double *inputimlo_d;             // inputimlo_d is inputrelo on the device
   double *forwardrehi_d;
   double *forwardremi_d;
   double *forwardrelo_d;
   double *forwardimhi_d;
   double *forwardimmi_d;
   double *forwardimlo_d;
   double *backwardrehi_d;
   double *backwardremi_d;
   double *backwardrelo_d;
   double *backwardimhi_d;
   double *backwardimmi_d;
   double *backwardimlo_d;
   double *crossrehi_d;
   double *crossremi_d;
   double *crossrelo_d;
   double *crossimhi_d;
   double *crossimmi_d;
   double *crossimlo_d;
   double *cffrehi_d;               // cffrehi_d is cffrehi on the device
   double *cffremi_d;               // cffremi_d is cffremi on the device
   double *cffrelo_d;               // cffrelo_d is cffrelo on the device
   double *cffimhi_d;               // cffimhi_d is cffimhi on the device
   double *cffimmi_d;               // cffimmi_d is cffimmi on the device
   double *cffimlo_d;               // cffimlo_d is cffimlo on the device

   size_t szdim = dim*(deg1)*sizeof(double);
   size_t sznvr = nvr*(deg1)*sizeof(double);
   size_t sznvr1 = (nvr-1)*(deg1)*sizeof(double);
   size_t sznvr2 = (nvr-2)*(deg1)*sizeof(double);
   size_t szcff = deg1*sizeof(double);

   hipMalloc((void**)&cffrehi_d,szcff);
   hipMalloc((void**)&cffremi_d,szcff);
   hipMalloc((void**)&cffrelo_d,szcff);
   hipMalloc((void**)&cffimhi_d,szcff);
   hipMalloc((void**)&cffimmi_d,szcff);
   hipMalloc((void**)&cffimlo_d,szcff);
   hipMalloc((void**)&inputrehi_d,szdim);
   hipMalloc((void**)&inputremi_d,szdim);
   hipMalloc((void**)&inputrelo_d,szdim);
   hipMalloc((void**)&inputimhi_d,szdim);
   hipMalloc((void**)&inputimmi_d,szdim);
   hipMalloc((void**)&inputimlo_d,szdim);
   hipMalloc((void**)&forwardrehi_d,sznvr);
   hipMalloc((void**)&forwardremi_d,sznvr);
   hipMalloc((void**)&forwardrelo_d,sznvr);
   hipMalloc((void**)&forwardimhi_d,sznvr);
   hipMalloc((void**)&forwardimmi_d,sznvr);
   hipMalloc((void**)&forwardimlo_d,sznvr);
   hipMalloc((void**)&backwardrehi_d,sznvr1);
   hipMalloc((void**)&backwardremi_d,sznvr1);
   hipMalloc((void**)&backwardrelo_d,sznvr1);
   hipMalloc((void**)&backwardimhi_d,sznvr1);
   hipMalloc((void**)&backwardimmi_d,sznvr1);
   hipMalloc((void**)&backwardimlo_d,sznvr1);
   hipMalloc((void**)&crossrehi_d,sznvr2);
   hipMalloc((void**)&crossremi_d,sznvr2);
   hipMalloc((void**)&crossrelo_d,sznvr2);
   hipMalloc((void**)&crossimhi_d,sznvr2);
   hipMalloc((void**)&crossimmi_d,sznvr2);
   hipMalloc((void**)&crossimlo_d,sznvr2);

   double *inputrehi_h = new double[dim*(deg1)];
   double *inputremi_h = new double[dim*(deg1)];
   double *inputrelo_h = new double[dim*(deg1)];
   double *inputimhi_h = new double[dim*(deg1)];
   double *inputimmi_h = new double[dim*(deg1)];
   double *inputimlo_h = new double[dim*(deg1)];
   int ix = 0;
   for(int i=0; i<dim; i++)
      for(int j=0; j<deg1; j++)
      {
         inputrehi_h[ix] = inputrehi[i][j];
         inputremi_h[ix] = inputremi[i][j];
         inputrelo_h[ix] = inputrelo[i][j];
         inputimhi_h[ix] = inputimhi[i][j];
         inputimmi_h[ix] = inputimmi[i][j];
         inputimlo_h[ix++] = inputimlo[i][j];
      }

   hipMemcpy(cffrehi_d,cffrehi,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffremi_d,cffremi,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffrelo_d,cffrelo,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffimhi_d,cffimhi,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffimmi_d,cffimmi,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffimlo_d,cffimlo,szcff,hipMemcpyHostToDevice);
   hipMemcpy(inputrehi_d,inputrehi_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputremi_d,inputremi_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputrelo_d,inputrelo_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputimhi_d,inputimhi_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputimmi_d,inputimmi_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputimlo_d,inputimlo_h,szdim,hipMemcpyHostToDevice);

   if(BS == deg1)
   {
      GPU_cmplx3_speel(BS,nvr,deg,idx,
         cffrehi_d,cffremi_d,cffrelo_d,cffimhi_d,cffimmi_d,cffimlo_d,
         inputrehi_d,inputremi_d,inputrelo_d,
         inputimhi_d,inputimmi_d,inputimlo_d,
         forwardrehi_d,forwardremi_d,forwardrelo_d,
         forwardimhi_d,forwardimmi_d,forwardimlo_d,
         backwardrehi_d,backwardremi_d,backwardrelo_d,
         backwardimhi_d,backwardimmi_d,backwardimlo_d,
         crossrehi_d,crossremi_d,crossrelo_d,
         crossimhi_d,crossimmi_d,crossimlo_d);
   }
   double *forwardrehi_h = new double[(deg1)*nvr];
   double *forwardremi_h = new double[(deg1)*nvr];
   double *forwardrelo_h = new double[(deg1)*nvr];
   double *forwardimhi_h = new double[(deg1)*nvr];
   double *forwardimmi_h = new double[(deg1)*nvr];
   double *forwardimlo_h = new double[(deg1)*nvr];
   double *backwardrehi_h = new double[(deg1)*(nvr-1)];
   double *backwardremi_h = new double[(deg1)*(nvr-1)];
   double *backwardrelo_h = new double[(deg1)*(nvr-1)];
   double *backwardimhi_h = new double[(deg1)*(nvr-1)];
   double *backwardimmi_h = new double[(deg1)*(nvr-1)];
   double *backwardimlo_h = new double[(deg1)*(nvr-1)];
   double *crossrehi_h = new double[(deg1)*(nvr-2)];
   double *crossremi_h = new double[(deg1)*(nvr-2)];
   double *crossrelo_h = new double[(deg1)*(nvr-2)];
   double *crossimhi_h = new double[(deg1)*(nvr-2)];
   double *crossimmi_h = new double[(deg1)*(nvr-2)];
   double *crossimlo_h = new double[(deg1)*(nvr-2)];
  
   hipMemcpy(forwardrehi_h,forwardrehi_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardremi_h,forwardremi_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardrelo_h,forwardrelo_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardimhi_h,forwardimhi_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardimmi_h,forwardimmi_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardimlo_h,forwardimlo_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(backwardrehi_h,backwardrehi_d,sznvr1,hipMemcpyDeviceToHost);
   hipMemcpy(backwardremi_h,backwardremi_d,sznvr1,hipMemcpyDeviceToHost);
   hipMemcpy(backwardrelo_h,backwardrelo_d,sznvr1,hipMemcpyDeviceToHost);
   hipMemcpy(backwardimhi_h,backwardimhi_d,sznvr1,hipMemcpyDeviceToHost);
   hipMemcpy(backwardimmi_h,backwardimmi_d,sznvr1,hipMemcpyDeviceToHost);
   hipMemcpy(backwardimlo_h,backwardimlo_d,sznvr1,hipMemcpyDeviceToHost);
   hipMemcpy(crossrehi_h,crossrehi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossremi_h,crossremi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossrelo_h,crossrelo_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossimhi_h,crossimhi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossimmi_h,crossimmi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossimlo_h,crossimlo_d,sznvr2,hipMemcpyDeviceToHost);

   int offset = (nvr-1)*deg1;
   for(int i=0; i<deg1; i++)   // assign value of the monomial
   {
      outputrehi[dim][i] = forwardrehi_h[offset+i];
      outputremi[dim][i] = forwardremi_h[offset+i];
      outputrelo[dim][i] = forwardrelo_h[offset+i];
      outputimhi[dim][i] = forwardimhi_h[offset+i];
      outputimmi[dim][i] = forwardimmi_h[offset+i];
      outputimlo[dim][i] = forwardimlo_h[offset+i];
   }
   ix = idx[nvr-1];
   offset = (nvr-2)*deg1;
   for(int i=0; i<deg1; i++)  // derivative with respect to x[n-1]
   {
      outputrehi[ix][i] = forwardrehi_h[offset+i];
      outputremi[ix][i] = forwardremi_h[offset+i];
      outputrelo[ix][i] = forwardrelo_h[offset+i];
      outputimhi[ix][i] = forwardimhi_h[offset+i];
      outputimmi[ix][i] = forwardimmi_h[offset+i];
      outputimlo[ix][i] = forwardimlo_h[offset+i];
   }
   ix = idx[0]; 
   offset = (nvr-2)*deg1;
   for(int i=0; i<deg1; i++)   // derivative with respect to x[0]
   {
      outputrehi[ix][i] = backwardrehi_h[offset+i];
      outputremi[ix][i] = backwardremi_h[offset+i];
      outputrelo[ix][i] = backwardrelo_h[offset+i];
      outputimhi[ix][i] = backwardimhi_h[offset+i];
      outputimmi[ix][i] = backwardimmi_h[offset+i];
      outputimlo[ix][i] = backwardimlo_h[offset+i];
   }
   for(int k=1; k<nvr-1; k++)  // derivative with respect to x[k]
   {
      ix = idx[k]; offset = (k-1)*deg1;
      for(int i=0; i<deg1; i++)
      {
         outputrehi[ix][i] = crossrehi_h[offset+i];
         outputremi[ix][i] = crossremi_h[offset+i];
         outputrelo[ix][i] = crossrelo_h[offset+i];
         outputimhi[ix][i] = crossimhi_h[offset+i];
         outputimmi[ix][i] = crossimmi_h[offset+i];
         outputimlo[ix][i] = crossimlo_h[offset+i];
      }
   }
}
