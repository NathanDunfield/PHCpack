#include "hip/hip_runtime.h"
// The file dbl5_monomials_kernels.cu defines the kernels specified
// in dbl5_monomials_kernels.h.

/* The algorithm to compute forward, backward, and cross products
 * (denoted respectively by arrays f, b, and c)
 * for a monomial cff*x[0]*x[1]* .. *x[n-1] goes as follows:
 *
 * f[0] := cff*x[0]
 * for i from 1 to n-1 do f[i] := f[i-1]*x[i]
 * if n > 2 then
 *    b[0] := x[n-1]*x[n-2]
 *    for i from 1 to n-3 do b[i] := b[i-1]*x[n-2-i]
 *    b[n-3] := b[n-3]*cff
 *    if n = 3 then
 *       c[0] = f[0]*x[2]
 *    else
 *       for i from 0 to n-4 do c[i] := f[i]*b[n-4-i]
 *       c[n-3] := f[n-3]*x[n-1]
 *
 * Compared to the evaluation and differentiation of a product of variables,
 * (without coefficient cff), two extra multiplications must be done,
 * but this is better than n+1 multiplications with cff afterwards. */

#include "double_double_gpufun.cu"
#include "penta_double_gpufun.cu"
#include "dbl5_convolutions_kernels.h"
#include "dbl5_monomials_kernels.h"

__device__ void dbl5_convolute
 ( double *xtb, double *xix, double *xmi, double *xrg, double *xpk,
   double *ytb, double *yix, double *ymi, double *yrg, double *ypk,
   double *ztb, double *zix, double *zmi, double *zrg, double *zpk,
   int dim, int k )
{
   double prdtb,prdix,prdmi,prdrg,prdpk;
   int idx;

   // zv[k] = xv[0]*yv[k];
   pdg_mul(xtb[0],xix[0],xmi[0],xrg[0],xpk[0],
           ytb[k],yix[k],ymi[k],yrg[k],ypk[k],
           &ztb[k],&zix[k],&zmi[k],&zrg[k],&zpk[k]);

   for(int i=1; i<=k; i++) // zv[k] = zv[k] + xv[i]*yv[k-i];
   {
      idx = k-i;
      pdg_mul(xtb[i],xix[i],xmi[i],xrg[i],xpk[i],
              ytb[idx],yix[idx],ymi[idx],yrg[idx],ypk[idx],
              &prdtb,&prdix,&prdmi,&prdrg,&prdpk);
      pdg_inc(&ztb[k],&zix[k],&zmi[k],&zrg[k],&zpk[k],
              prdtb,prdix,prdmi,prdrg,prdpk);
   }
}

__device__ void cmplx5_convolute
 ( double *xretb, double *xreix, double *xremi, double *xrerg, double *xrepk,
   double *ximtb, double *ximix, double *ximmi, double *ximrg, double *ximpk,
   double *yretb, double *yreix, double *yremi, double *yrerg, double *yrepk,
   double *yimtb, double *yimix, double *yimmi, double *yimrg, double *yimpk,
   double *zretb, double *zreix, double *zremi, double *zrerg, double *zrepk,
   double *zimtb, double *zimix, double *zimmi, double *zimrg, double *zimpk,
   int dim, int k )
{
   double xrtb,xitb,yrtb,yitb,zrtb,zitb,acctb;
   double xrix,xiix,yrix,yiix,zrix,ziix,accix;
   double xrmi,ximi,yrmi,yimi,zrmi,zimi,accmi;
   double xrrg,xirg,yrrg,yirg,zrrg,zirg,accrg;
   double xrpk,xipk,yrpk,yipk,zrpk,zipk,accpk;
   int idx;

   // z[k] = x[0]*y[k]
   xrtb = xretb[0]; xrix = xreix[0]; xrmi = xremi[0];
   xrrg = xrerg[0]; xrpk = xrepk[0];
   xitb = ximtb[0]; xiix = ximix[0]; ximi = ximmi[0];
   xirg = ximrg[0]; xipk = ximpk[0];
   yrtb = yretb[k]; yrix = yreix[k]; yrmi = yremi[k];
   yrrg = yrerg[k]; yrpk = yrepk[k];
   yitb = yimtb[k]; yiix = yimix[k]; yimi = yimmi[k];
   yirg = yimrg[k]; yipk = yimpk[k];

   pdg_mul(xrtb,xrix,xrmi,xrrg,xrpk,yrtb,yrix,yrmi,yrrg,yrpk,
           &zrtb,&zrix,&zrmi,&zrrg,&zrpk);         // zr = xr*yr
   pdg_mul(xitb,xiix,ximi,xirg,xipk,yitb,yiix,yimi,yirg,yipk,
           &acctb,&accix,&accmi,&accrg,&accpk);    // acc = xi*yi
   pdg_minus(&acctb,&accix,&accmi,&accrg,&accpk);
   pdg_inc(&zrtb,&zrix,&zrmi,&zrrg,&zrpk,
           acctb,accix,accmi,accrg,accpk);         // zr = xr*yr - xi*yi
   pdg_mul(xrtb,xrix,xrmi,xrrg,xrpk,yitb,yiix,yimi,yirg,yipk,
           &zitb,&ziix,&zimi,&zirg,&zipk);         // zi = xr*yi
   pdg_mul(xitb,xiix,ximi,xirg,xipk,yrtb,yrix,yrmi,yrrg,yrpk,
           &acctb,&accix,&accmi,&accrg,&accpk);    // acc = xi*yr
   pdg_inc(&zitb,&ziix,&zimi,&zirg,&zipk,
           acctb,accix,accmi,accrg,accpk);         // zr = xr*yr + xi*yi

   zretb[k] = zrtb; zreix[k] = zrix; zremi[k] = zrmi;
   zrerg[k] = zrrg; zrepk[k] = zrpk;
   zimtb[k] = zitb; zimix[k] = ziix; zimmi[k] = zimi;
   zimrg[k] = zirg; zimpk[k] = zipk;

   for(int i=1; i<=k; i++) // z[k] = z[k] + x[i]*y[k-i]
   {
      xrtb = xretb[i]; xrix = xreix[i]; xrmi = xremi[i];
      xrrg = xrerg[i]; xrpk = xrepk[i];
      xitb = ximtb[i]; xiix = ximix[i]; ximi = ximmi[i];
      xirg = ximrg[i]; xipk = ximpk[i];
      idx = k-i;
      yrtb = yretb[idx]; yrix = yreix[idx]; yrmi = yremi[idx];
      yrrg = yrerg[idx]; yrpk = yrepk[idx];
      yitb = yimtb[idx]; yiix = yimix[idx]; yimi = yimmi[idx];
      yirg = yimrg[idx]; yipk = yimpk[idx];

      pdg_mul(xrtb,xrix,xrmi,xrrg,xrpk,yrtb,yrix,yrmi,yrrg,yrpk,
              &zrtb,&zrix,&zrmi,&zrrg,&zrpk);        // zr = xr*yr
      pdg_mul(xitb,xiix,ximi,xirg,xipk,yitb,yiix,yimi,yirg,yipk,
              &acctb,&accix,&accmi,&accrg,&accpk);   // xi*yi
      pdg_minus(&acctb,&accix,&accmi,&accrg,&accpk);
      pdg_inc(&zrtb,&zrix,&zrmi,&zrrg,&zrpk,
              acctb,accix,accmi,accrg,accpk);        // zr = xr*yr - xi*yi
      pdg_mul(xrtb,xrix,xrmi,xrrg,xrpk,yitb,yiix,yimi,yirg,yipk,
              &zitb,&ziix,&zimi,&zirg,&zipk);        // zi = xr*yi
      pdg_mul(xitb,xiix,ximi,xirg,xipk,yrtb,yrix,yrmi,yrrg,yrpk,
              &acctb,&accix,&accmi,&accrg,&accpk);   // xi*yr
      pdg_inc(&zitb,&ziix,&zimi,&zirg,&zipk,
              acctb,accix,accmi,accrg,accpk);        // zr = xr*yr + xi*yi
      // zre[k] += zr; zim[k] += zi
      pdg_inc(&zretb[k],&zreix[k],&zremi[k],&zrerg[k],&zrepk[k],
              zrtb,zrix,zrmi,zrrg,zrpk);
      pdg_inc(&zimtb[k],&zimix[k],&zimmi[k],&zimrg[k],&zimpk[k],
              zitb,ziix,zimi,zirg,zipk);
   }
}

__global__ void GPU_dbl5_speel
 ( int nvr, int deg, int *idx,
   double *cfftb, double *cffix, double *cffmi, double *cffrg, double *cffpk,
   double *inputtb, double *inputix, double *inputmi,
   double *inputrg, double *inputpk,
   double *forwardtb, double *forwardix, double *forwardmi,
   double *forwardrg, double *forwardpk,
   double *backwardtb, double *backwardix, double *backwardmi,
   double *backwardrg, double *backwardpk,
   double *crosstb, double *crossix, double *crossmi,
   double *crossrg, double *crosspk )
{
   const int k = threadIdx.x;
   const int deg1 = deg+1;
   int ix1,ix2;

   __shared__ double xvtb[pd_shmemsize];
   __shared__ double xvix[pd_shmemsize];
   __shared__ double xvmi[pd_shmemsize];
   __shared__ double xvrg[pd_shmemsize];
   __shared__ double xvpk[pd_shmemsize];
   __shared__ double yvtb[pd_shmemsize];
   __shared__ double yvix[pd_shmemsize];
   __shared__ double yvmi[pd_shmemsize];
   __shared__ double yvrg[pd_shmemsize];
   __shared__ double yvpk[pd_shmemsize];
   __shared__ double zvtb[pd_shmemsize];
   __shared__ double zvix[pd_shmemsize];
   __shared__ double zvmi[pd_shmemsize];
   __shared__ double zvrg[pd_shmemsize];
   __shared__ double zvpk[pd_shmemsize];
  
   xvtb[k] = cfftb[k]; xvix[k] = cffix[k]; xvmi[k] = cffmi[k]; 
   xvrg[k] = cffrg[k]; xvpk[k] = cffpk[k];
   ix1 = idx[0]*deg1+k;
   yvtb[k] = inputtb[ix1]; yvix[k] = inputix[ix1]; yvmi[k] = inputmi[ix1];
   yvrg[k] = inputrg[ix1]; yvpk[k] = inputpk[ix1]; 
   __syncthreads();
   dbl5_convolute(xvtb,xvix,xvmi,xvrg,xvpk,
                  yvtb,yvix,yvmi,yvrg,yvpk,
                  zvtb,zvix,zvmi,zvrg,zvpk,deg1,k);
   __syncthreads();
   forwardtb[k] = zvtb[k]; forwardix[k] = zvix[k];
   forwardmi[k] = zvmi[k]; forwardrg[k] = zvrg[k];
   forwardpk[k] = zvpk[k];                            // f[0] = cff*x[0]

   for(int i=1; i<nvr; i++)
   {
      xvtb[k] = zvtb[k]; xvix[k] = zvix[k]; xvmi[k] = zvmi[k];
      xvrg[k] = zvrg[k]; xvpk[k] = zvpk[k];
      ix2 = idx[i]*deg1+k;
      yvtb[k] = inputtb[ix2]; yvix[k] = inputix[ix2]; yvmi[k] = inputmi[ix2];
      yvrg[k] = inputrg[ix2]; yvpk[k] = inputpk[ix2];
      __syncthreads();
      dbl5_convolute(xvtb,xvix,xvmi,xvrg,xvpk,
                     yvtb,yvix,yvmi,yvrg,yvpk,
                     zvtb,zvix,zvmi,zvrg,zvpk,deg1,k);
      __syncthreads();
      ix1 = i*deg1+k;
      forwardtb[ix1] = zvtb[k]; forwardix[ix1] = zvix[k]; 
      forwardmi[ix1] = zvmi[k]; forwardrg[ix1] = zvrg[k]; 
      forwardpk[ix1] = zvpk[k];                        // f[i] = f[i-1]*x[i]
   }
   if(nvr > 2)
   {
      ix1 = idx[nvr-1]*deg1+k;
      xvtb[k] = inputtb[ix1]; xvix[k] = inputix[ix1]; xvmi[k] = inputmi[ix1];
      xvrg[k] = inputrg[ix1]; xvpk[k] = inputpk[ix1];
      ix2 = idx[nvr-2]*deg1+k;
      yvtb[k] = inputtb[ix2]; yvix[k] = inputix[ix2]; yvmi[k] = inputmi[ix2];
      yvrg[k] = inputrg[ix2]; yvpk[k] = inputpk[ix2];
      __syncthreads();
      dbl5_convolute(xvtb,xvix,xvmi,xvrg,xvpk,
                     yvtb,yvix,yvmi,yvrg,yvpk,
                     zvtb,zvix,zvmi,zvrg,zvpk,deg1,k);
      __syncthreads();
      backwardtb[k] = zvtb[k]; backwardix[k] = zvix[k];
      backwardmi[k] = zvmi[k]; backwardrg[k] = zvrg[k];
      backwardpk[k] = zvpk[k];                       // b[0] = x[n-1]*x[n-2]
      for(int i=1; i<nvr-2; i++)
      {
         xvtb[k] = zvtb[k]; xvix[k] = zvix[k]; xvmi[k] = zvmi[k]; 
         xvrg[k] = zvrg[k]; xvpk[k] = zvpk[k];
         ix2 = idx[nvr-2-i]*deg1+k;
         yvtb[k] = inputtb[ix2]; yvix[k] = inputix[ix2];
         yvmi[k] = inputmi[ix2]; yvrg[k] = inputrg[ix2];
         yvpk[k] = inputpk[ix2];
         __syncthreads();
         dbl5_convolute(xvtb,xvix,xvmi,xvrg,xvpk,
                        yvtb,yvix,yvmi,yvrg,yvpk,
                        zvtb,zvix,zvmi,zvrg,zvpk,deg1,k);
         __syncthreads();
         ix1 = i*deg1+k;
         backwardtb[ix1] = zvtb[k]; backwardix[ix1] = zvix[k];
         backwardmi[ix1] = zvmi[k]; backwardrg[ix1] = zvrg[k];
         backwardpk[ix1] = zvpk[k];             // b[i] = b[i-1]*x[n-2-i]
      }
      xvtb[k] = zvtb[k];  xvix[k] = zvix[k];  xvmi[k] = zvmi[k];
      xvrg[k] = zvrg[k];  xvpk[k] = zvpk[k];
      yvtb[k] = cfftb[k]; yvix[k] = cffix[k]; yvmi[k] = cffmi[k];
      yvrg[k] = cffrg[k]; yvpk[k] = cffpk[k];
      __syncthreads();
      dbl5_convolute(xvtb,xvix,xvmi,xvrg,xvpk,
                     yvtb,yvix,yvmi,yvrg,yvpk,
                     zvtb,zvix,zvmi,zvrg,zvpk,deg1,k);
      __syncthreads();
      ix2 = (nvr-3)*deg1+k;
      backwardtb[ix2] = zvtb[k]; backwardix[ix2] = zvix[k];
      backwardmi[ix2] = zvmi[k]; backwardrg[ix2] = zvrg[k];
      backwardpk[ix2] = zvpk[k];                    // b[n-3] = b[n-3]*cff

      if(nvr == 3)
      {
         xvtb[k] = forwardtb[k]; xvix[k] = forwardix[k];
         xvmi[k] = forwardmi[k]; xvrg[k] = forwardrg[k];
         xvpk[k] = forwardpk[k];
         ix2 = idx[2]*deg1+k;
         yvtb[k] = inputtb[ix2]; yvix[k] = inputix[ix2];
         yvmi[k] = inputmi[ix2]; yvrg[k] = inputrg[ix2]; 
         yvpk[k] = inputpk[ix2];
         __syncthreads();
         dbl5_convolute(xvtb,xvix,xvmi,xvrg,xvpk,
                        yvtb,yvix,yvmi,yvrg,yvpk,
                        zvtb,zvix,zvmi,zvrg,zvpk,deg1,k);
         __syncthreads();
         crosstb[k] = zvtb[k]; crossix[k] = zvix[k];
         crossmi[k] = zvmi[k]; crossrg[k] = zvrg[k];
         crosspk[k] = zvpk[k];                      // c[0] = f[0]*x[2]
      }
      else
      {
         for(int i=0; i<nvr-3; i++)
         {
            ix1 = i*deg1+k; 
            xvtb[k] = forwardtb[ix1]; xvix[k] = forwardix[ix1];
            xvmi[k] = forwardmi[ix1]; xvrg[k] = forwardrg[ix1];
            xvpk[k] = forwardpk[ix1];
            ix2 = (nvr-4-i)*deg1+k;
            yvtb[k] = backwardtb[ix2]; yvix[k] = backwardix[ix2];
            yvmi[k] = backwardmi[ix2]; yvrg[k] = backwardrg[ix2];
            yvpk[k] = backwardpk[ix2];
            __syncthreads();
            dbl5_convolute(xvtb,xvix,xvmi,xvrg,xvpk,
                           yvtb,yvix,yvmi,yvrg,yvpk,
                           zvtb,zvix,zvmi,zvrg,zvpk,deg1,k);
            __syncthreads();
            crosstb[ix1] = zvtb[k]; crossix[ix1] = zvix[k];
            crossmi[ix1] = zvmi[k]; crossrg[ix1] = zvrg[k];
            crosspk[ix1] = zvpk[k];            // c[i] = f[i]*b[n-4-i]
         }
         ix1 = (nvr-3)*deg1+k;
         xvtb[k] = forwardtb[ix1]; xvix[k] = forwardix[ix1];
         xvmi[k] = forwardmi[ix1]; xvrg[k] = forwardrg[ix1];
         xvpk[k] = forwardpk[ix1];
         ix2 = idx[nvr-1]*deg1+k;
         yvtb[k] = inputtb[ix2]; yvix[k] = inputix[ix2];
         yvmi[k] = inputmi[ix2]; yvrg[k] = inputrg[ix2];
         yvpk[k] = inputpk[ix2];
         __syncthreads();
         dbl5_convolute(xvtb,xvix,xvmi,xvrg,xvpk,
                        yvtb,yvix,yvmi,yvrg,yvpk,
                        zvtb,zvix,zvmi,zvrg,zvpk,deg1,k);
         __syncthreads();
         crosstb[ix1] = zvtb[k]; crossix[ix1] = zvix[k];
         crossmi[ix1] = zvmi[k]; crossrg[ix1] = zvrg[k];
         crosspk[ix1] = zvpk[k];                   // c[n-3] = f[n-3]*x[n-1]
      }
   }
}

__global__ void GPU_cmplx5_speel
 ( int nvr, int deg, int *idx,
   double *cffretb, double *cffreix, double *cffremi,
   double *cffrerg, double *cffrepk,
   double *cffimtb, double *cffimix, double *cffimmi,
   double *cffimrg, double *cffimpk,
   double *inputretb, double *inputreix, double *inputremi,
   double *inputrerg, double *inputrepk,
   double *inputimtb, double *inputimix, double *inputimmi,
   double *inputimrg, double *inputimpk,
   double *forwardretb, double *forwardreix, double *forwardremi,
   double *forwardrerg, double *forwardrepk,
   double *forwardimtb, double *forwardimix, double *forwardimmi,
   double *forwardimrg, double *forwardimpk,
   double *backwardretb, double *backwardreix, double *backwardremi,
   double *backwardrerg, double *backwardrepk,
   double *backwardimtb, double *backwardimix, double *backwardimmi,
   double *backwardimrg, double *backwardimpk,
   double *crossretb, double *crossreix, double *crossremi,
   double *crossrerg, double *crossrepk,
   double *crossimtb, double *crossimix, double *crossimmi,
   double *crossimrg, double *crossimpk )
{
   const int k = threadIdx.x;
   const int deg1 = deg+1;
   int ix1,ix2;

   __shared__ double xvretb[pd_shmemsize];
   __shared__ double xvreix[pd_shmemsize];
   __shared__ double xvremi[pd_shmemsize];
   __shared__ double xvrerg[pd_shmemsize];
   __shared__ double xvrepk[pd_shmemsize];
   __shared__ double xvimtb[pd_shmemsize];
   __shared__ double xvimix[pd_shmemsize];
   __shared__ double xvimmi[pd_shmemsize];
   __shared__ double xvimrg[pd_shmemsize];
   __shared__ double xvimpk[pd_shmemsize];
   __shared__ double yvretb[pd_shmemsize];
   __shared__ double yvreix[pd_shmemsize];
   __shared__ double yvremi[pd_shmemsize];
   __shared__ double yvrerg[pd_shmemsize];
   __shared__ double yvrepk[pd_shmemsize];
   __shared__ double yvimtb[pd_shmemsize];
   __shared__ double yvimix[pd_shmemsize];
   __shared__ double yvimmi[pd_shmemsize];
   __shared__ double yvimrg[pd_shmemsize];
   __shared__ double yvimpk[pd_shmemsize];
   __shared__ double zvretb[pd_shmemsize];
   __shared__ double zvreix[pd_shmemsize];
   __shared__ double zvremi[pd_shmemsize];
   __shared__ double zvrerg[pd_shmemsize];
   __shared__ double zvrepk[pd_shmemsize];
   __shared__ double zvimtb[pd_shmemsize];
   __shared__ double zvimix[pd_shmemsize];
   __shared__ double zvimmi[pd_shmemsize];
   __shared__ double zvimrg[pd_shmemsize];
   __shared__ double zvimpk[pd_shmemsize];

   xvretb[k] = cffretb[k]; xvreix[k] = cffreix[k]; xvremi[k] = cffremi[k];
   xvrerg[k] = cffrerg[k]; xvrepk[k] = cffrepk[k];
   xvimtb[k] = cffimtb[k]; xvimix[k] = cffimix[k]; xvimmi[k] = cffimmi[k];
   xvimrg[k] = cffimrg[k]; xvimpk[k] = cffimpk[k];
   ix1 = idx[0]*deg1+k;
   yvretb[k] = inputretb[ix1]; yvreix[k] = inputreix[ix1];
   yvremi[k] = inputremi[ix1]; yvrerg[k] = inputrerg[ix1];
   yvrepk[k] = inputrepk[ix1];
   yvimtb[k] = inputimtb[ix1]; yvimix[k] = inputimix[ix1];
   yvimmi[k] = inputimmi[ix1]; yvimrg[k] = inputimrg[ix1];
   yvimpk[k] = inputimpk[ix1];
   __syncthreads();                                      // f[0] = cff*x[0] 
   cmplx5_convolute(xvretb,xvreix,xvremi,xvrerg,xvrepk,
                    xvimtb,xvimix,xvimmi,xvimrg,xvimpk,
                    yvretb,yvreix,yvremi,yvrerg,yvrepk,
                    yvimtb,yvimix,yvimmi,yvimrg,yvimpk,
                    zvretb,zvreix,zvremi,zvrerg,zvrepk,
                    zvimtb,zvimix,zvimmi,zvimrg,zvimpk,deg1,k);
   __syncthreads();
   forwardretb[k] = zvretb[k]; forwardreix[k] = zvreix[k];
   forwardremi[k] = zvremi[k]; forwardrerg[k] = zvrerg[k];
   forwardrepk[k] = zvrepk[k];
   forwardimtb[k] = zvimtb[k]; forwardimix[k] = zvimix[k];
   forwardimmi[k] = zvimmi[k]; forwardimrg[k] = zvimrg[k];
   forwardimpk[k] = zvimpk[k];

   for(int i=1; i<nvr; i++)
   {
      xvretb[k] = zvretb[k]; xvreix[k] = zvreix[k];
      xvremi[k] = zvremi[k]; xvrerg[k] = zvrerg[k];
      xvrepk[k] = zvrepk[k];
      xvimtb[k] = zvimtb[k]; xvimix[k] = zvimix[k];
      xvimmi[k] = zvimmi[k]; xvimrg[k] = zvimrg[k];
      xvimpk[k] = zvimpk[k];
      ix2 = idx[i]*deg1+k;
      yvretb[k] = inputretb[ix2]; yvreix[k] = inputreix[ix2];
      yvremi[k] = inputremi[ix2]; yvrerg[k] = inputrerg[ix2];
      yvrepk[k] = inputrepk[ix2];
      yvimtb[k] = inputimtb[ix2]; yvimix[k] = inputimix[ix2];
      yvimmi[k] = inputimmi[ix2]; yvimrg[k] = inputimrg[ix2];
      yvimpk[k] = inputimpk[ix2];
      __syncthreads();                                 // f[i] = f[i-i]*x[i]
      cmplx5_convolute(xvretb,xvreix,xvremi,xvrerg,xvrepk,
                       xvimtb,xvimix,xvimmi,xvimrg,xvimpk,
                       yvretb,yvreix,yvremi,yvrerg,yvrepk,
                       yvimtb,yvimix,yvimmi,yvimrg,yvimpk,
                       zvretb,zvreix,zvremi,zvrerg,zvrepk,
                       zvimtb,zvimix,zvimmi,zvimrg,zvimpk,deg1,k);
      __syncthreads();
      ix1 = i*deg1+k;                                   
      forwardretb[ix1] = zvretb[k]; forwardreix[ix1] = zvreix[k];
      forwardremi[ix1] = zvremi[k]; forwardrerg[ix1] = zvrerg[k];
      forwardrepk[ix1] = zvrepk[k];
      forwardimtb[ix1] = zvimtb[k]; forwardimix[ix1] = zvimix[k];
      forwardimmi[ix1] = zvimmi[k]; forwardimrg[ix1] = zvimrg[k];
      forwardimpk[ix1] = zvimpk[k]; 
   }
   if(nvr > 2)
   {
      ix1 = idx[nvr-1]*deg1+k;
      xvretb[k] = inputretb[ix1]; xvreix[k] = inputreix[ix1];
      xvremi[k] = inputremi[ix1]; xvrerg[k] = inputrerg[ix1];
      xvrepk[k] = inputrepk[ix1];
      xvimtb[k] = inputimtb[ix1]; xvimix[k] = inputimix[ix1];
      xvimmi[k] = inputimmi[ix1]; xvimrg[k] = inputimrg[ix1];
      xvimpk[k] = inputimpk[ix1];
      ix2 = idx[nvr-2]*deg1+k;
      yvretb[k] = inputretb[ix2]; yvreix[k] = inputreix[ix2];
      yvremi[k] = inputremi[ix2]; yvrerg[k] = inputrerg[ix2];
      yvrepk[k] = inputrepk[ix2];
      yvimtb[k] = inputimtb[ix2]; yvimix[k] = inputimix[ix2];
      yvimmi[k] = inputimmi[ix2]; yvimrg[k] = inputimrg[ix2];
      yvimpk[k] = inputimpk[ix2];
      __syncthreads();                               // b[0] = x[n-1]*x[n-2]
      cmplx5_convolute(xvretb,xvreix,xvremi,xvrerg,xvrepk,
                       xvimtb,xvimix,xvimmi,xvimrg,xvimpk,
                       yvretb,yvreix,yvremi,yvrerg,yvrepk,
                       yvimtb,yvimix,yvimmi,yvimrg,yvimpk,
                       zvretb,zvreix,zvremi,zvrerg,zvrepk,
                       zvimtb,zvimix,zvimmi,zvimrg,zvimpk,deg1,k);
      __syncthreads();
      backwardretb[k] = zvretb[k]; backwardreix[k] = zvreix[k];
      backwardremi[k] = zvremi[k]; backwardrerg[k] = zvrerg[k];
      backwardrepk[k] = zvrepk[k];
      backwardimtb[k] = zvimtb[k]; backwardimix[k] = zvimix[k];
      backwardimmi[k] = zvimmi[k]; backwardimrg[k] = zvimrg[k];
      backwardimpk[k] = zvimpk[k];

      for(int i=1; i<nvr-2; i++)
      {
         xvretb[k] = zvretb[k]; xvreix[k] = zvreix[k];
         xvremi[k] = zvremi[k]; xvrerg[k] = zvrerg[k];
         xvrepk[k] = zvrepk[k];
         xvimtb[k] = zvimtb[k]; xvimix[k] = zvimix[k];
         xvimmi[k] = zvimmi[k]; xvimrg[k] = zvimrg[k];
         xvimpk[k] = zvimpk[k];
         ix2 = idx[nvr-2-i]*deg1+k;
         yvretb[k] = inputretb[ix2]; yvreix[k] = inputreix[ix2];
         yvremi[k] = inputremi[ix2]; yvrerg[k] = inputrerg[ix2];
         yvrepk[k] = inputrepk[ix2];
         yvimtb[k] = inputimtb[ix2]; yvimix[k] = inputimix[ix2];
         yvimmi[k] = inputimmi[ix2]; yvimrg[k] = inputimrg[ix2];
         yvimpk[k] = inputimpk[ix2];
         __syncthreads();                           // b[i] = b[i]*x[n-2-i]
         cmplx5_convolute(xvretb,xvreix,xvremi,xvrerg,xvrepk,
                          xvimtb,xvimix,xvimmi,xvimrg,xvimpk,
                          yvretb,yvreix,yvremi,yvrerg,yvrepk,
                          yvimtb,yvimix,yvimmi,yvimrg,yvimpk,
                          zvretb,zvreix,zvremi,zvrerg,zvrepk,
                          zvimtb,zvimix,zvimmi,zvimrg,zvimpk,deg1,k);
         __syncthreads();
         ix1 = i*deg1+k;
         backwardretb[ix1] = zvretb[k]; backwardreix[ix1] = zvreix[k];
         backwardremi[ix1] = zvremi[k]; backwardrerg[ix1] = zvrerg[k];
         backwardrepk[ix1] = zvrepk[k];
         backwardimtb[ix1] = zvimtb[k]; backwardimix[ix1] = zvimix[k];
         backwardimmi[ix1] = zvimmi[k]; backwardimrg[ix1] = zvimrg[k];
         backwardimpk[ix1] = zvimpk[k];
      }
      xvretb[k] = zvretb[k]; xvreix[k] = zvreix[k]; xvremi[k] = zvremi[k];
      xvrerg[k] = zvrerg[k]; xvrepk[k] = zvrepk[k];
      xvimtb[k] = zvimtb[k]; xvimix[k] = zvimix[k]; xvimmi[k] = zvimmi[k];
      xvimrg[k] = zvimrg[k]; xvimpk[k] = zvimpk[k];
      yvretb[k] = cffretb[k]; yvreix[k] = cffreix[k]; yvremi[k] = cffremi[k];
      yvrerg[k] = cffrerg[k]; yvrepk[k] = cffrepk[k];
      yvimtb[k] = cffimtb[k]; yvimix[k] = cffimix[k]; yvimmi[k] = cffimmi[k];
      yvimrg[k] = cffimrg[k]; yvimpk[k] = cffimpk[k];
      __syncthreads();                               // b[n-3] = b[n-3]*cff
      cmplx5_convolute(xvretb,xvreix,xvremi,xvrerg,xvrepk,
                       xvimtb,xvimix,xvimmi,xvimrg,xvimpk,
                       yvretb,yvreix,yvremi,yvrerg,yvrepk,
                       yvimtb,yvimix,yvimmi,yvimrg,yvimpk,
                       zvretb,zvreix,zvremi,zvrerg,zvrepk,
                       zvimtb,zvimix,zvimmi,zvimrg,zvimpk,deg1,k);
      __syncthreads();
      ix1 = (nvr-3)*deg1+k;
      backwardretb[ix1] = zvretb[k]; backwardreix[ix1] = zvreix[k];
      backwardremi[ix1] = zvremi[k]; backwardrerg[ix1] = zvrerg[k];
      backwardrepk[ix1] = zvrepk[k];
      backwardimtb[ix1] = zvimtb[k]; backwardimix[ix1] = zvimix[k];
      backwardimmi[ix1] = zvimmi[k]; backwardimrg[ix1] = zvimrg[k];
      backwardimpk[ix1] = zvimpk[k];

      if(nvr == 3)
      {
         xvretb[k] = forwardretb[k]; xvreix[k] = forwardreix[k];
         xvremi[k] = forwardremi[k]; xvrerg[k] = forwardrerg[k];
         xvrepk[k] = forwardrepk[k];
         xvimtb[k] = forwardimtb[k]; xvimix[k] = forwardimix[k];
         xvimmi[k] = forwardimmi[k]; xvimrg[k] = forwardimrg[k];
         xvimpk[k] = forwardimpk[k];
         ix2 = idx[2]*deg1+k;
         yvretb[k] = inputretb[ix2]; yvreix[k] = inputreix[ix2];
         yvremi[k] = inputremi[ix2]; yvrerg[k] = inputrerg[ix2];
         yvrepk[k] = inputrepk[ix2];
         yvimtb[k] = inputimtb[ix2]; yvimix[k] = inputimix[ix2];
         yvimmi[k] = inputimmi[ix2]; yvimrg[k] = inputimrg[ix2];
         yvimpk[k] = inputimpk[ix2];
         __syncthreads();                               // c[0] = f[0]*x[2]
         cmplx5_convolute(xvretb,xvreix,xvremi,xvrerg,xvrepk,
                          xvimtb,xvimix,xvimmi,xvimrg,xvimpk,
                          yvretb,yvreix,yvremi,yvrerg,yvrepk,
                          yvimtb,yvimix,yvimmi,yvimrg,yvimpk,
                          zvretb,zvreix,zvremi,zvrerg,zvrepk,
                          zvimtb,zvimix,zvimmi,zvimrg,zvimpk,deg1,k);
         __syncthreads();
         crossretb[k] = zvretb[k]; crossreix[k] = zvreix[k];
         crossremi[k] = zvremi[k]; crossrerg[k] = zvrerg[k];
         crossrepk[k] = zvrepk[k];
         crossimtb[k] = zvimtb[k]; crossimix[k] = zvimix[k];
         crossimmi[k] = zvimmi[k]; crossimrg[k] = zvimrg[k];
         crossimpk[k] = zvimpk[k];
      }
      else
      {
         for(int i=0; i<nvr-3; i++)
         {
            ix1 = i*deg1+k;   
            xvretb[k] = forwardretb[ix1]; xvreix[k] = forwardreix[ix1];
            xvremi[k] = forwardremi[ix1]; xvrerg[k] = forwardrerg[ix1];
            xvrepk[k] = forwardrepk[ix1];
            xvimtb[k] = forwardimtb[ix1]; xvimix[k] = forwardimix[ix1];
            xvimmi[k] = forwardimmi[ix1]; xvimrg[k] = forwardimrg[ix1];
            xvimpk[k] = forwardimpk[ix1];
            ix2 = (nvr-4-i)*deg1+k;
            yvretb[k] = backwardretb[ix2]; yvreix[k] = backwardreix[ix2];
            yvremi[k] = backwardremi[ix2]; yvrerg[k] = backwardrerg[ix2];
            yvrepk[k] = backwardrepk[ix2];
            yvimtb[k] = backwardimtb[ix2]; yvimix[k] = backwardimix[ix2];
            yvimmi[k] = backwardimmi[ix2]; yvimrg[k] = backwardimrg[ix2];
            yvimpk[k] = backwardimpk[ix2];
            __syncthreads();                        // c[i] = f[i]*b[n-4-i]
            cmplx5_convolute(xvretb,xvreix,xvremi,xvrerg,xvrepk,
                             xvimtb,xvimix,xvimmi,xvimrg,xvimpk,
                             yvretb,yvreix,yvremi,yvrerg,yvrepk,
                             yvimtb,yvimix,yvimmi,yvimrg,yvimpk,
                             zvretb,zvreix,zvremi,zvrerg,zvrepk,
                             zvimtb,zvimix,zvimmi,zvimrg,zvimpk,deg1,k);
            __syncthreads();
            ix1 = i*deg1+k;
            crossretb[ix1] = zvretb[k]; crossreix[ix1] = zvreix[k];
            crossremi[ix1] = zvremi[k]; crossrerg[ix1] = zvrerg[k];
            crossrepk[ix1] = zvrepk[k];
            crossimtb[ix1] = zvimtb[k]; crossimix[ix1] = zvimix[k];
            crossimmi[ix1] = zvimmi[k]; crossimrg[ix1] = zvimrg[k];
            crossimpk[ix1] = zvimpk[k];
         }
         ix1 = (nvr-3)*deg1+k;
         xvretb[k] = forwardretb[ix1]; xvreix[k] = forwardreix[ix1];
         xvremi[k] = forwardremi[ix1]; xvrerg[k] = forwardrerg[ix1];
         xvrepk[k] = forwardrepk[ix1];
         xvimtb[k] = forwardimtb[ix1]; xvimix[k] = forwardimix[ix1];
         xvimmi[k] = forwardimmi[ix1]; xvimrg[k] = forwardimrg[ix1];
         xvimpk[k] = forwardimpk[ix1];
         ix2 = idx[nvr-1]*deg1+k;
         yvretb[k] = inputretb[ix2]; yvreix[k] = inputreix[ix2];
         yvremi[k] = inputremi[ix2]; yvrerg[k] = inputrerg[ix2];
         yvrepk[k] = inputrepk[ix2];
         yvimtb[k] = inputimtb[ix2]; yvimix[k] = inputimix[ix2];
         yvimmi[k] = inputimmi[ix2]; yvimrg[k] = inputimrg[ix2];
         yvimpk[k] = inputimpk[ix2];
         __syncthreads();                         // c[n-3] = f[n-3]*x[n-1]
         cmplx5_convolute(xvretb,xvreix,xvremi,xvrerg,xvrepk,
                          xvimtb,xvimix,xvimmi,xvimrg,xvimpk,
                          yvretb,yvreix,yvremi,yvrerg,yvrepk,
                          yvimtb,yvimix,yvimmi,yvimrg,yvimpk,
                          zvretb,zvreix,zvremi,zvrerg,zvrepk,
                          zvimtb,zvimix,zvimmi,zvimrg,zvimpk,deg1,k);
         __syncthreads();
         ix1 = (nvr-3)*deg1+k;
         crossretb[ix1] = zvretb[k]; crossreix[ix1] = zvreix[k];
         crossremi[ix1] = zvremi[k]; crossrerg[ix1] = zvrerg[k];
         crossrepk[ix1] = zvrepk[k];
         crossimtb[ix1] = zvimtb[k]; crossimix[ix1] = zvimix[k];
         crossimmi[ix1] = zvimmi[k]; crossimrg[ix1] = zvimrg[k];
         crossimpk[ix1] = zvimpk[k];
      }
   }
}

void GPU_dbl5_evaldiff
 ( int BS, int dim, int nvr, int deg, int *idx,
   double *cfftb, double *cffix, double *cffmi, double *cffrg, double *cffpk,
   double **inputtb, double **inputix, double **inputmi,
   double **inputrg, double **inputpk,
   double **outputtb, double **outputix, double **outputmi,
   double **outputrg, double **outputpk )
{
   const int deg1 = deg+1;      // length of all vectors
   double *inputtb_d;           // inputtb_d is inputtb on the device
   double *inputix_d;           // inputix_d is inputix on the device
   double *inputmi_d;           // inputmi_d is inputmi on the device
   double *inputrg_d;           // inputrg_d is inputrg on the device
   double *inputpk_d;           // inputpk_d is inputpk on the device
   double *forwardtb_d;         // highest forward products on the device
   double *forwardix_d;         // second highest forward products
   double *forwardmi_d;         // middle forward products
   double *forwardrg_d;         // second lowest forward products
   double *forwardpk_d;         // lowest forward products
   double *backwardtb_d;        // highest backward products on the device
   double *backwardix_d;        // second highest backward products
   double *backwardmi_d;        // middle backward products
   double *backwardrg_d;        // second lowest backward products
   double *backwardpk_d;        // lowest backward products
   double *crosstb_d;           // highest cross products on the device
   double *crossix_d;           // second highest cross products
   double *crossmi_d;           // middle cross products
   double *crossrg_d;           // second lowest cross products
   double *crosspk_d;           // lowest cross products
   double *cfftb_d;             // cfftb_d is cfftb on device
   double *cffix_d;             // cffix_d is cffix on device
   double *cffmi_d;             // cffmi_d is cffmi on device
   double *cffrg_d;             // cffrg_d is cffrg on device
   double *cffpk_d;             // cffpk_d is cffpk on device
   int *idx_d;                  // idx_d is idx on device

   size_t szcff = deg1*sizeof(double);
   size_t szdim = dim*(deg1)*sizeof(double);
   size_t sznvr = nvr*(deg1)*sizeof(double);
   size_t sznvr2 = (nvr-2)*(deg1)*sizeof(double);
   size_t szidx = nvr*sizeof(int);

   hipMalloc((void**)&idx_d,szidx);
   hipMalloc((void**)&cfftb_d,szcff);
   hipMalloc((void**)&cffix_d,szcff);
   hipMalloc((void**)&cffmi_d,szcff);
   hipMalloc((void**)&cffrg_d,szcff);
   hipMalloc((void**)&cffpk_d,szcff);
   hipMalloc((void**)&inputtb_d,szdim);
   hipMalloc((void**)&inputix_d,szdim);
   hipMalloc((void**)&inputmi_d,szdim);
   hipMalloc((void**)&inputrg_d,szdim);
   hipMalloc((void**)&inputpk_d,szdim);
   hipMalloc((void**)&forwardtb_d,sznvr);
   hipMalloc((void**)&forwardix_d,sznvr);
   hipMalloc((void**)&forwardmi_d,sznvr);
   hipMalloc((void**)&forwardrg_d,sznvr);
   hipMalloc((void**)&forwardpk_d,sznvr);
   hipMalloc((void**)&backwardtb_d,sznvr2);
   hipMalloc((void**)&backwardix_d,sznvr2);
   hipMalloc((void**)&backwardmi_d,sznvr2);
   hipMalloc((void**)&backwardrg_d,sznvr2);
   hipMalloc((void**)&backwardpk_d,sznvr2);
   hipMalloc((void**)&crosstb_d,sznvr2);
   hipMalloc((void**)&crossix_d,sznvr2);
   hipMalloc((void**)&crossmi_d,sznvr2);
   hipMalloc((void**)&crossrg_d,sznvr2);
   hipMalloc((void**)&crosspk_d,sznvr2);

   double *inputtb_h = new double[dim*(deg1)];
   double *inputix_h = new double[dim*(deg1)];
   double *inputmi_h = new double[dim*(deg1)];
   double *inputrg_h = new double[dim*(deg1)];
   double *inputpk_h = new double[dim*(deg1)];
   int ix = 0;
   for(int i=0; i<dim; i++)
      for(int j=0; j<deg1; j++)
      {
         inputtb_h[ix] = inputtb[i][j];
         inputix_h[ix] = inputix[i][j];
         inputmi_h[ix] = inputmi[i][j];
         inputrg_h[ix] = inputrg[i][j];
         inputpk_h[ix++] = inputpk[i][j];
      }

   hipMemcpy(idx_d,idx,szidx,hipMemcpyHostToDevice);
   hipMemcpy(cfftb_d,cfftb,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffix_d,cffix,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffmi_d,cffmi,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffrg_d,cffrg,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffpk_d,cffpk,szcff,hipMemcpyHostToDevice);
   hipMemcpy(inputtb_d,inputtb_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputix_d,inputix_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputmi_d,inputmi_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputrg_d,inputrg_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputpk_d,inputpk_h,szdim,hipMemcpyHostToDevice);

   if(BS == deg1)
   {
      GPU_dbl5_speel<<<1,BS>>>
         (nvr,deg,idx_d,cfftb_d,cffix_d,cffmi_d,cffrg_d,cffpk_d,
          inputtb_d,inputix_d,inputmi_d,inputrg_d,inputpk_d,
          forwardtb_d,forwardix_d,forwardmi_d,forwardrg_d,forwardpk_d,
          backwardtb_d,backwardix_d,backwardmi_d,backwardrg_d,backwardpk_d,
          crosstb_d,crossix_d,crossmi_d,crossrg_d,crosspk_d);
   }
   double *forwardtb_h = new double[(deg1)*nvr];
   double *forwardix_h = new double[(deg1)*nvr];
   double *forwardmi_h = new double[(deg1)*nvr];
   double *forwardrg_h = new double[(deg1)*nvr];
   double *forwardpk_h = new double[(deg1)*nvr];
   double *backwardtb_h = new double[(deg1)*(nvr-2)];
   double *backwardix_h = new double[(deg1)*(nvr-2)];
   double *backwardmi_h = new double[(deg1)*(nvr-2)];
   double *backwardrg_h = new double[(deg1)*(nvr-2)];
   double *backwardpk_h = new double[(deg1)*(nvr-2)];
   double *crosstb_h = new double[(deg1)*(nvr-2)];
   double *crossix_h = new double[(deg1)*(nvr-2)];
   double *crossmi_h = new double[(deg1)*(nvr-2)];
   double *crossrg_h = new double[(deg1)*(nvr-2)];
   double *crosspk_h = new double[(deg1)*(nvr-2)];
  
   hipMemcpy(forwardtb_h,forwardtb_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardix_h,forwardix_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardmi_h,forwardmi_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardrg_h,forwardrg_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardpk_h,forwardpk_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(backwardtb_h,backwardtb_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardix_h,backwardix_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardmi_h,backwardmi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardrg_h,backwardrg_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardpk_h,backwardpk_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crosstb_h,crosstb_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossix_h,crossix_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossmi_h,crossmi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossrg_h,crossrg_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crosspk_h,crosspk_d,sznvr2,hipMemcpyDeviceToHost);

   int offset = (nvr-1)*deg1;            // assign value of the monomial
   for(int i=0; i<deg1; i++)
   {
      outputtb[dim][i] = forwardtb_h[offset+i];
      outputix[dim][i] = forwardix_h[offset+i];
      outputmi[dim][i] = forwardmi_h[offset+i];
      outputrg[dim][i] = forwardrg_h[offset+i];
      outputpk[dim][i] = forwardpk_h[offset+i];
   }
   ix = idx[nvr-1];                      // derivative with respect to x[n-1]
   offset = (nvr-2)*deg1;
   for(int i=0; i<deg1; i++)
   {
      outputtb[ix][i] = forwardtb_h[offset+i];
      outputix[ix][i] = forwardix_h[offset+i];
      outputmi[ix][i] = forwardmi_h[offset+i];
      outputrg[ix][i] = forwardrg_h[offset+i];
      outputpk[ix][i] = forwardpk_h[offset+i];
   }
   ix = idx[0];                          // derivative with respect to x[0]
   offset = (nvr-3)*deg1;
   for(int i=0; i<deg1; i++)
   {
      outputtb[ix][i] = backwardtb_h[offset+i];
      outputix[ix][i] = backwardix_h[offset+i];
      outputmi[ix][i] = backwardmi_h[offset+i];
      outputrg[ix][i] = backwardrg_h[offset+i];
      outputpk[ix][i] = backwardpk_h[offset+i];
   }
   for(int k=1; k<nvr-1; k++)            // derivative with respect to x[k]
   {
      ix = idx[k]; offset = (k-1)*deg1;
      for(int i=0; i<deg1; i++)
      {
         outputtb[ix][i] = crosstb_h[offset+i];
         outputix[ix][i] = crossix_h[offset+i];
         outputmi[ix][i] = crossmi_h[offset+i];
         outputrg[ix][i] = crossrg_h[offset+i];
         outputpk[ix][i] = crosspk_h[offset+i];
      }
   }
}

void GPU_cmplx5_evaldiff
 ( int BS, int dim, int nvr, int deg, int *idx,
   double *cffretb, double *cffreix, double *cffremi,
   double *cffrerg, double *cffrepk,
   double *cffimtb, double *cffimix, double *cffimmi,
   double *cffimrg, double *cffimpk,
   double **inputretb, double **inputreix, double **inputremi,
   double **inputrerg, double **inputrepk,
   double **inputimtb, double **inputimix, double **inputimmi,
   double **inputimrg, double **inputimpk,
   double **outputretb, double **outputreix, double **outputremi,
   double **outputrerg, double **outputrepk,
   double **outputimtb, double **outputimix, double **outputimmi,
   double **outputimrg, double **outputimpk )
{
   const int deg1 = deg+1;          // length of all vectors
   double *inputretb_d;             // inputretb_d is inputretb on the device
   double *inputreix_d;             // inputreix_d is inputreix on the device
   double *inputremi_d;             // inputremi_d is inputremi on the device
   double *inputrerg_d;             // inputrerg_d is inputrerg on the device
   double *inputrepk_d;             // inputrepk_d is inputrepk on the device
   double *inputimtb_d;             // inputimtb_d is inputretb on the device
   double *inputimix_d;             // inputimix_d is inputreix on the device
   double *inputimmi_d;             // inputimmi_d is inputremi on the device
   double *inputimrg_d;             // inputimrg_d is inputrerg on the device
   double *inputimpk_d;             // inputimpk_d is inputrepk on the device
   double *forwardretb_d;
   double *forwardreix_d;
   double *forwardremi_d;
   double *forwardrerg_d;
   double *forwardrepk_d;
   double *forwardimtb_d;
   double *forwardimix_d;
   double *forwardimmi_d;
   double *forwardimrg_d;
   double *forwardimpk_d;
   double *backwardretb_d;
   double *backwardreix_d;
   double *backwardremi_d;
   double *backwardrerg_d;
   double *backwardrepk_d;
   double *backwardimtb_d;
   double *backwardimix_d;
   double *backwardimmi_d;
   double *backwardimrg_d;
   double *backwardimpk_d;
   double *crossretb_d;
   double *crossreix_d;
   double *crossremi_d;
   double *crossrerg_d;
   double *crossrepk_d;
   double *crossimtb_d;
   double *crossimix_d;
   double *crossimmi_d;
   double *crossimrg_d;
   double *crossimpk_d;
   double *cffretb_d;               // cffretb_d is cffretb on the device
   double *cffreix_d;               // cffreix_d is cffreix on the device
   double *cffremi_d;               // cffremi_d is cffremi on the device
   double *cffrerg_d;               // cffremi_d is cffrerg on the device
   double *cffrepk_d;               // cffrepk_d is cffrepk on the device
   double *cffimtb_d;               // cffimtb_d is cffimtb on the device
   double *cffimix_d;               // cffimix_d is cffimix on the device
   double *cffimmi_d;               // cffimmi_d is cffimmi on the device
   double *cffimrg_d;               // cffimrg_d is cffimrg on the device
   double *cffimpk_d;               // cffimpk_d is cffimpk on the device
   int *idx_d;                      // idx_d is idx on the device

   size_t szdim = dim*(deg1)*sizeof(double);
   size_t sznvr = nvr*(deg1)*sizeof(double);
   size_t sznvr2 = (nvr-2)*(deg1)*sizeof(double);
   size_t szidx = nvr*sizeof(int);
   size_t szcff = deg1*sizeof(double);

   hipMalloc((void**)&idx_d,szidx);
   hipMalloc((void**)&cffretb_d,szcff);
   hipMalloc((void**)&cffreix_d,szcff);
   hipMalloc((void**)&cffremi_d,szcff);
   hipMalloc((void**)&cffrerg_d,szcff);
   hipMalloc((void**)&cffrepk_d,szcff);
   hipMalloc((void**)&cffimtb_d,szcff);
   hipMalloc((void**)&cffimix_d,szcff);
   hipMalloc((void**)&cffimmi_d,szcff);
   hipMalloc((void**)&cffimrg_d,szcff);
   hipMalloc((void**)&cffimpk_d,szcff);
   hipMalloc((void**)&inputretb_d,szdim);
   hipMalloc((void**)&inputreix_d,szdim);
   hipMalloc((void**)&inputremi_d,szdim);
   hipMalloc((void**)&inputrerg_d,szdim);
   hipMalloc((void**)&inputrepk_d,szdim);
   hipMalloc((void**)&inputimtb_d,szdim);
   hipMalloc((void**)&inputimix_d,szdim);
   hipMalloc((void**)&inputimmi_d,szdim);
   hipMalloc((void**)&inputimrg_d,szdim);
   hipMalloc((void**)&inputimpk_d,szdim);
   hipMalloc((void**)&forwardretb_d,sznvr);
   hipMalloc((void**)&forwardreix_d,sznvr);
   hipMalloc((void**)&forwardremi_d,sznvr);
   hipMalloc((void**)&forwardrerg_d,sznvr);
   hipMalloc((void**)&forwardrepk_d,sznvr);
   hipMalloc((void**)&forwardimtb_d,sznvr);
   hipMalloc((void**)&forwardimix_d,sznvr);
   hipMalloc((void**)&forwardimmi_d,sznvr);
   hipMalloc((void**)&forwardimrg_d,sznvr);
   hipMalloc((void**)&forwardimpk_d,sznvr);
   hipMalloc((void**)&backwardretb_d,sznvr2);
   hipMalloc((void**)&backwardreix_d,sznvr2);
   hipMalloc((void**)&backwardremi_d,sznvr2);
   hipMalloc((void**)&backwardrerg_d,sznvr2);
   hipMalloc((void**)&backwardrepk_d,sznvr2);
   hipMalloc((void**)&backwardimtb_d,sznvr2);
   hipMalloc((void**)&backwardimix_d,sznvr2);
   hipMalloc((void**)&backwardimmi_d,sznvr2);
   hipMalloc((void**)&backwardimrg_d,sznvr2);
   hipMalloc((void**)&backwardimpk_d,sznvr2);
   hipMalloc((void**)&crossretb_d,sznvr2);
   hipMalloc((void**)&crossreix_d,sznvr2);
   hipMalloc((void**)&crossremi_d,sznvr2);
   hipMalloc((void**)&crossrerg_d,sznvr2);
   hipMalloc((void**)&crossrepk_d,sznvr2);
   hipMalloc((void**)&crossimtb_d,sznvr2);
   hipMalloc((void**)&crossimix_d,sznvr2);
   hipMalloc((void**)&crossimmi_d,sznvr2);
   hipMalloc((void**)&crossimrg_d,sznvr2);
   hipMalloc((void**)&crossimpk_d,sznvr2);

   double *inputretb_h = new double[dim*(deg1)];
   double *inputreix_h = new double[dim*(deg1)];
   double *inputremi_h = new double[dim*(deg1)];
   double *inputrerg_h = new double[dim*(deg1)];
   double *inputrepk_h = new double[dim*(deg1)];
   double *inputimtb_h = new double[dim*(deg1)];
   double *inputimix_h = new double[dim*(deg1)];
   double *inputimmi_h = new double[dim*(deg1)];
   double *inputimrg_h = new double[dim*(deg1)];
   double *inputimpk_h = new double[dim*(deg1)];
   int ix = 0;
   for(int i=0; i<dim; i++)
      for(int j=0; j<deg1; j++)
      {
         inputretb_h[ix] = inputretb[i][j];
         inputreix_h[ix] = inputreix[i][j];
         inputremi_h[ix] = inputremi[i][j];
         inputrerg_h[ix] = inputrerg[i][j];
         inputrepk_h[ix] = inputrepk[i][j];
         inputimtb_h[ix] = inputimtb[i][j];
         inputimix_h[ix] = inputimix[i][j];
         inputimmi_h[ix] = inputimmi[i][j];
         inputimrg_h[ix] = inputimrg[i][j];
         inputimpk_h[ix++] = inputimpk[i][j];
      }

   hipMemcpy(idx_d,idx,szidx,hipMemcpyHostToDevice);
   hipMemcpy(cffretb_d,cffretb,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffreix_d,cffreix,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffremi_d,cffremi,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffrerg_d,cffrerg,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffrepk_d,cffrepk,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffimtb_d,cffimtb,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffimix_d,cffimix,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffimmi_d,cffimmi,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffimrg_d,cffimrg,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffimpk_d,cffimpk,szcff,hipMemcpyHostToDevice);
   hipMemcpy(inputretb_d,inputretb_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputreix_d,inputreix_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputremi_d,inputremi_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputrerg_d,inputrerg_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputrepk_d,inputrepk_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputimtb_d,inputimtb_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputimix_d,inputimix_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputimmi_d,inputimmi_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputimrg_d,inputimrg_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputimpk_d,inputimpk_h,szdim,hipMemcpyHostToDevice);

   if(BS == deg1)
   {
      GPU_cmplx5_speel<<<1,BS>>>(nvr,deg,idx_d,
         cffretb_d,cffreix_d,cffremi_d,cffrerg_d,cffrepk_d,
         cffimtb_d,cffimix_d,cffimmi_d,cffimrg_d,cffimpk_d,
         inputretb_d,inputreix_d,inputremi_d,inputrerg_d,inputrepk_d,
         inputimtb_d,inputimix_d,inputimmi_d,inputimrg_d,inputimpk_d,
         forwardretb_d,forwardreix_d,forwardremi_d,
         forwardrerg_d,forwardrepk_d,
         forwardimtb_d,forwardimix_d,forwardimmi_d,
         forwardimrg_d,forwardimpk_d,
         backwardretb_d,backwardreix_d,backwardremi_d,
         backwardrerg_d,backwardrepk_d,
         backwardimtb_d,backwardimix_d,backwardimmi_d,
         backwardimrg_d,backwardimpk_d,
         crossretb_d,crossreix_d,crossremi_d,crossrerg_d,crossrepk_d,
         crossimtb_d,crossimix_d,crossimmi_d,crossimrg_d,crossimpk_d);
   }
   double *forwardretb_h = new double[(deg1)*nvr];
   double *forwardreix_h = new double[(deg1)*nvr];
   double *forwardremi_h = new double[(deg1)*nvr];
   double *forwardrerg_h = new double[(deg1)*nvr];
   double *forwardrepk_h = new double[(deg1)*nvr];
   double *forwardimtb_h = new double[(deg1)*nvr];
   double *forwardimix_h = new double[(deg1)*nvr];
   double *forwardimmi_h = new double[(deg1)*nvr];
   double *forwardimrg_h = new double[(deg1)*nvr];
   double *forwardimpk_h = new double[(deg1)*nvr];
   double *backwardretb_h = new double[(deg1)*(nvr-2)];
   double *backwardreix_h = new double[(deg1)*(nvr-2)];
   double *backwardremi_h = new double[(deg1)*(nvr-2)];
   double *backwardrerg_h = new double[(deg1)*(nvr-2)];
   double *backwardrepk_h = new double[(deg1)*(nvr-2)];
   double *backwardimtb_h = new double[(deg1)*(nvr-2)];
   double *backwardimix_h = new double[(deg1)*(nvr-2)];
   double *backwardimmi_h = new double[(deg1)*(nvr-2)];
   double *backwardimrg_h = new double[(deg1)*(nvr-2)];
   double *backwardimpk_h = new double[(deg1)*(nvr-2)];
   double *crossretb_h = new double[(deg1)*(nvr-2)];
   double *crossreix_h = new double[(deg1)*(nvr-2)];
   double *crossremi_h = new double[(deg1)*(nvr-2)];
   double *crossrerg_h = new double[(deg1)*(nvr-2)];
   double *crossrepk_h = new double[(deg1)*(nvr-2)];
   double *crossimtb_h = new double[(deg1)*(nvr-2)];
   double *crossimix_h = new double[(deg1)*(nvr-2)];
   double *crossimmi_h = new double[(deg1)*(nvr-2)];
   double *crossimrg_h = new double[(deg1)*(nvr-2)];
   double *crossimpk_h = new double[(deg1)*(nvr-2)];
  
   hipMemcpy(forwardretb_h,forwardretb_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardreix_h,forwardreix_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardremi_h,forwardremi_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardrerg_h,forwardrerg_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardrepk_h,forwardrepk_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardimtb_h,forwardimtb_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardimix_h,forwardimix_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardimmi_h,forwardimmi_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardimrg_h,forwardimrg_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardimpk_h,forwardimpk_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(backwardretb_h,backwardretb_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardreix_h,backwardreix_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardremi_h,backwardremi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardrerg_h,backwardrerg_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardrepk_h,backwardrepk_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardimtb_h,backwardimtb_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardimix_h,backwardimix_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardimmi_h,backwardimmi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardimrg_h,backwardimrg_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardimpk_h,backwardimpk_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossretb_h,crossretb_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossreix_h,crossreix_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossremi_h,crossremi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossrerg_h,crossrerg_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossrepk_h,crossrepk_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossimtb_h,crossimtb_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossimix_h,crossimix_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossimmi_h,crossimmi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossimrg_h,crossimrg_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossimpk_h,crossimpk_d,sznvr2,hipMemcpyDeviceToHost);

   int offset = (nvr-1)*deg1;
   for(int i=0; i<deg1; i++)   // assign value of the monomial
   {
      outputretb[dim][i] = forwardretb_h[offset+i];
      outputreix[dim][i] = forwardreix_h[offset+i];
      outputremi[dim][i] = forwardremi_h[offset+i];
      outputrerg[dim][i] = forwardrerg_h[offset+i];
      outputrepk[dim][i] = forwardrepk_h[offset+i];
      outputimtb[dim][i] = forwardimtb_h[offset+i];
      outputimix[dim][i] = forwardimix_h[offset+i];
      outputimmi[dim][i] = forwardimmi_h[offset+i];
      outputimrg[dim][i] = forwardimrg_h[offset+i];
      outputimpk[dim][i] = forwardimpk_h[offset+i];
   }
   ix = idx[nvr-1];
   offset = (nvr-2)*deg1;
   for(int i=0; i<deg1; i++)  // derivative with respect to x[n-1]
   {
      outputretb[ix][i] = forwardretb_h[offset+i];
      outputreix[ix][i] = forwardreix_h[offset+i];
      outputremi[ix][i] = forwardremi_h[offset+i];
      outputrerg[ix][i] = forwardrerg_h[offset+i];
      outputrepk[ix][i] = forwardrepk_h[offset+i];
      outputimtb[ix][i] = forwardimtb_h[offset+i];
      outputimix[ix][i] = forwardimix_h[offset+i];
      outputimmi[ix][i] = forwardimmi_h[offset+i];
      outputimrg[ix][i] = forwardimrg_h[offset+i];
      outputimpk[ix][i] = forwardimpk_h[offset+i];
   }
   ix = idx[0]; 
   offset = (nvr-3)*deg1;
   for(int i=0; i<deg1; i++)   // derivative with respect to x[0]
   {
      outputretb[ix][i] = backwardretb_h[offset+i];
      outputreix[ix][i] = backwardreix_h[offset+i];
      outputremi[ix][i] = backwardremi_h[offset+i];
      outputrerg[ix][i] = backwardrerg_h[offset+i];
      outputrepk[ix][i] = backwardrepk_h[offset+i];
      outputimtb[ix][i] = backwardimtb_h[offset+i];
      outputimix[ix][i] = backwardimix_h[offset+i];
      outputimmi[ix][i] = backwardimmi_h[offset+i];
      outputimrg[ix][i] = backwardimrg_h[offset+i];
      outputimpk[ix][i] = backwardimpk_h[offset+i];
   }
   for(int k=1; k<nvr-1; k++)  // derivative with respect to x[k]
   {
      ix = idx[k]; offset = (k-1)*deg1;
      for(int i=0; i<deg1; i++)
      {
         outputretb[ix][i] = crossretb_h[offset+i];
         outputreix[ix][i] = crossreix_h[offset+i];
         outputremi[ix][i] = crossremi_h[offset+i];
         outputrerg[ix][i] = crossrerg_h[offset+i];
         outputrepk[ix][i] = crossrepk_h[offset+i];
         outputimtb[ix][i] = crossimtb_h[offset+i];
         outputimix[ix][i] = crossimix_h[offset+i];
         outputimmi[ix][i] = crossimmi_h[offset+i];
         outputimrg[ix][i] = crossimrg_h[offset+i];
         outputimpk[ix][i] = crossimpk_h[offset+i];
      }
   }
}
