#include "hip/hip_runtime.h"
// The file dbl2_monomials_kernels.cu defines the kernels with prototypes
// in dbl2_monomials_kernels.h.

/* The algorithm to compute forward, backward, and cross products
 * (denoted respectively by arrays f, b, and c)
 * for a monomial cff*x[0]*x[1]* .. *x[n-1] goes as follows:
 *
 * f[0] := cff*x[0]
 * for i from 1 to n-1 do f[i] := f[i-1]*x[i]
 * if n > 2 then
 *    b[0] := x[n-1]*x[n-2]
 *    for i from 1 to n-3 do b[i] := b[i-1]*x[n-2-i]
 *    b[n-3] := b[n-3]*cff
 *    if n = 3 then
 *       c[0] = f[0]*x[2]
 *    else
 *       for i from 0 to n-4 do c[i] := f[i]*b[n-4-i]
 *       c[n-3] := f[n-3]*x[n-1]
 *
 * Compared to the evaluation and differentiation of a product of variables,
 * (without coefficient cff), two extra multiplications must be done,
 * but this is better than n+1 multiplications with cff afterwards. */

#include "double_double_gpufun.cu"
#include "dbl2_convolutions_kernels.h"
#include "dbl2_monomials_kernels.h"

__device__ void dbl2_convolute
 ( double *xhi, double *xlo, double *yhi, double *ylo,
   double *zhi, double *zlo, int dim, int k )
{
   double prdhi,prdlo;

   // zv[k] = xv[0]*yv[k];
   ddg_mul(xhi[0],xlo[0],yhi[k],ylo[k],&zhi[k],&zlo[k]);

   for(int i=1; i<=k; i++) // zv[k] = zv[k] + xv[i]*yv[k-i];
   {
      ddg_mul(xhi[i],xlo[i],yhi[k-i],ylo[k-i],&prdhi,&prdlo);
      ddg_inc(&zhi[k],&zlo[k],prdhi,prdlo);
   }
}

__device__ void cmplx2_convolute
 ( double *xrehi, double *xrelo, double *ximhi, double *ximlo,
   double *yrehi, double *yrelo, double *yimhi, double *yimlo,
   double *zrehi, double *zrelo, double *zimhi, double *zimlo,
   int dim, int k )
{
   double xrhi,xihi,yrhi,yihi,zrhi,zihi,acchi;
   double xrlo,xilo,yrlo,yilo,zrlo,zilo,acclo;

   // z[k] = x[0]*y[k]
   xrhi = xrehi[0]; xrlo = xrelo[0]; xihi = ximhi[0]; xilo = ximlo[0];
   yrhi = yrehi[k]; yrlo = yrelo[k]; yihi = yimhi[k]; yilo = yimlo[k];

   ddg_mul(xrhi,xrlo,yrhi,yrlo,&zrhi,&zrlo);   // zr = xr*yr
   ddg_mul(xihi,xilo,yihi,yilo,&acchi,&acclo); // acc = xi*yi
   ddg_dec(&zrhi,&zrlo,acchi,acclo);           // zr = xr*yr - xi*yi
   ddg_mul(xrhi,xrlo,yihi,yilo,&zihi,&zilo);   // zi = xr*yi
   ddg_mul(xihi,xilo,yrhi,yrlo,&acchi,&acclo); // acc = xi*yr
   ddg_inc(&zihi,&zilo,acchi,acclo);           // zr = xr*yr + xi*yi

   zrehi[k] = zrhi; zrelo[k] = zrlo;
   zimhi[k] = zihi; zimlo[k] = zilo;

   for(int i=1; i<=k; i++) // z[k] = z[k] + x[i]*y[k-i]
   {
      xrhi = xrehi[i]; xrlo = xrelo[i];
      xihi = ximhi[i]; xilo = ximlo[i];
      yrhi = yrehi[k-i]; yrlo = yrelo[k-i];
      yihi = yimhi[k-i]; yilo = yimlo[k-i];

      ddg_mul(xrhi,xrlo,yrhi,yrlo,&zrhi,&zrlo);   // zr = xr*yr
      ddg_mul(xihi,xilo,yihi,yilo,&acchi,&acclo); // acc = xi*yi
      ddg_dec(&zrhi,&zrlo,acchi,acclo);           // zr = xr*yr - xi*yi
      ddg_mul(xrhi,xrlo,yihi,yilo,&zihi,&zilo);   // zi = xr*yi
      ddg_mul(xihi,xilo,yrhi,yrlo,&acchi,&acclo); // acc = xi*yr
      ddg_inc(&zihi,&zilo,acchi,acclo);           // zr = xr*yr + xi*yi

      ddg_inc(&zrehi[k],&zrelo[k],zrhi,zrlo);     // zvre[k] += zr;
      ddg_inc(&zimhi[k],&zimlo[k],zihi,zilo);     // zvim[k] += zi;
   }
}

__global__ void GPU_dbl2_speel
 ( int nvr, int deg, int *idx, double *cffhi, double *cfflo, double *inputhi,
   double *inputlo, double *forwardhi, double *forwardlo, double *backwardhi,
   double *backwardlo, double *crosshi, double *crosslo )
{
   const int k = threadIdx.x;
   const int deg1 = deg+1;
   int ix1,ix2;

   __shared__ double xvhi[dd_shmemsize];
   __shared__ double xvlo[dd_shmemsize];
   __shared__ double yvhi[dd_shmemsize];
   __shared__ double yvlo[dd_shmemsize];
   __shared__ double zvhi[dd_shmemsize];
   __shared__ double zvlo[dd_shmemsize];
  
   xvhi[k] = cffhi[k]; xvlo[k] = cfflo[k];
   ix1 = idx[0]*deg1+k;
   yvhi[k] = inputhi[ix1]; yvlo[k] = inputlo[ix1]; 
   __syncthreads(); dbl2_convolute(xvhi,xvlo,yvhi,yvlo,zvhi,zvlo,deg1,k);
   forwardhi[k] = zvhi[k];
   forwardlo[k] = zvlo[k];                            // f[0] = cff*x[0]

   for(int i=1; i<nvr; i++)
   {
      xvhi[k] = zvhi[k]; xvlo[k] = zvlo[k];
      ix2 = idx[i]*deg1+k;
      yvhi[k] = inputhi[ix2]; yvlo[k] = inputlo[ix2];
      __syncthreads(); dbl2_convolute(xvhi,xvlo,yvhi,yvlo,zvhi,zvlo,deg1,k);
      forwardhi[i*deg1+k] = zvhi[k]; 
      forwardlo[i*deg1+k] = zvlo[k];                  // f[i] = f[i-1]*x[i]
   }
   if(nvr > 2)
   {
      ix1 = idx[nvr-1]*deg1+k;
      xvhi[k] = inputhi[ix1]; xvlo[k] = inputlo[ix1];
      ix2 = idx[nvr-2]*deg1+k;
      yvhi[k] = inputhi[ix2]; yvlo[k] = inputlo[ix2];
      __syncthreads(); dbl2_convolute(xvhi,xvlo,yvhi,yvlo,zvhi,zvlo,deg1,k);
      backwardhi[k] = zvhi[k];
      backwardlo[k] = zvlo[k];                       // b[0] = x[n-1]*x[n-2]
      for(int i=1; i<nvr-2; i++)
      {
         xvhi[k] = zvhi[k]; xvlo[k] = zvlo[k];
         ix2 = idx[nvr-2-i]*deg1+k;
         yvhi[k] = inputhi[ix2]; yvlo[k] = inputlo[ix2];
         __syncthreads();
         dbl2_convolute(xvhi,xvlo,yvhi,yvlo,zvhi,zvlo,deg1,k);
         backwardhi[i*deg1+k] = zvhi[k];
         backwardlo[i*deg1+k] = zvlo[k];             // b[i] = b[i-1]*x[n-2-i]
      }
      xvhi[k] = zvhi[k];  xvlo[k] = zvlo[k];
      yvhi[k] = cffhi[k]; yvlo[k] = cfflo[k];
      __syncthreads(); dbl2_convolute(xvhi,xvlo,yvhi,yvlo,zvhi,zvlo,deg1,k);
      ix2 = (nvr-3)*deg1+k;
      backwardhi[ix2] = zvhi[k];
      backwardlo[ix2] = zvlo[k];                    // b[n-3] = b[n-3]*cff

      if(nvr == 3)
      {
         xvhi[k] = forwardhi[k]; xvlo[k] = forwardlo[k];
         ix2 = idx[2]*deg1+k;
         yvhi[k] = inputhi[ix2]; yvlo[k] = inputlo[ix2];
         __syncthreads(); dbl2_convolute(xvhi,xvlo,yvhi,yvlo,zvhi,zvlo,deg1,k);
         crosshi[k] = zvhi[k];
         crosslo[k] = zvlo[k];                      // c[0] = f[0]*x[2]
      }
      else
      {
         for(int i=0; i<nvr-3; i++)
         {
            ix1 = i*deg1+k; 
            xvhi[k] = forwardhi[ix1]; xvlo[k] = forwardlo[ix1];
            ix2 = (nvr-4-i)*deg1+k;
            yvhi[k] = backwardhi[ix2]; yvlo[k] = backwardlo[ix2];
            __syncthreads();
            dbl2_convolute(xvhi,xvlo,yvhi,yvlo,zvhi,zvlo,deg1,k);
            crosshi[i*deg1+k] = zvhi[k];
            crosslo[i*deg1+k] = zvlo[k];            // c[i] = f[i]*b[n-4-i]
         }
         ix1 = (nvr-3)*deg1+k;
         xvhi[k] = forwardhi[ix1]; xvlo[k] = forwardlo[ix1];
         ix2 = idx[nvr-1]*deg1+k;
         yvhi[k] = inputhi[ix2]; yvlo[k] = inputlo[ix2];
         __syncthreads();
         dbl2_convolute(xvhi,xvlo,yvhi,yvlo,zvhi,zvlo,deg1,k);
         crosshi[(nvr-3)*deg1+k] = zvhi[k];
         crosslo[(nvr-3)*deg1+k] = zvlo[k];         // c[n-3] = f[n-3]*x[n-1]
      }
   }
}

__global__ void GPU_cmplx2_speel
 ( int nvr, int deg, int *idx,
   double *cffrehi, double *cffrelo, double *cffimhi, double *cffimlo,
   double *inputrehi, double *inputrelo, double *inputimhi, double *inputimlo,
   double *forwardrehi, double *forwardrelo, double *forwardimhi,
   double *forwardimlo, double *backwardrehi, double *backwardrelo,
   double *backwardimhi, double *backwardimlo, double *crossrehi,
   double *crossrelo, double *crossimhi, double *crossimlo )
{
   const int k = threadIdx.x;
   const int deg1 = deg+1;
   int ix1,ix2;

   __shared__ double xvrehi[dd_shmemsize];
   __shared__ double xvrelo[dd_shmemsize];
   __shared__ double xvimhi[dd_shmemsize];
   __shared__ double xvimlo[dd_shmemsize];
   __shared__ double yvrehi[dd_shmemsize];
   __shared__ double yvrelo[dd_shmemsize];
   __shared__ double yvimhi[dd_shmemsize];
   __shared__ double yvimlo[dd_shmemsize];
   __shared__ double zvrehi[dd_shmemsize];
   __shared__ double zvrelo[dd_shmemsize];
   __shared__ double zvimhi[dd_shmemsize];
   __shared__ double zvimlo[dd_shmemsize];

   xvrehi[k] = cffrehi[k]; xvrelo[k] = cffrelo[k];
   xvimhi[k] = cffimhi[k]; xvimlo[k] = cffimlo[k];
   ix1 = idx[0]*deg1+k;
   yvrehi[k] = inputrehi[ix1]; yvrelo[k] = inputrelo[ix1];
   yvimhi[k] = inputimhi[ix1]; yvimlo[k] = inputimlo[ix1];
   __syncthreads();                                       // f[0] = cff*x[0] 
   cmplx2_convolute(xvrehi,xvrelo,xvimhi,xvimlo,yvrehi,yvrelo,yvimhi,yvimlo,
                    zvrehi,zvrelo,zvimhi,zvimlo,deg1,k);
   forwardrehi[k] = zvrehi[k]; forwardrelo[k] = zvrelo[k];
   forwardimhi[k] = zvimhi[k]; forwardimlo[k] = zvimlo[k];

   for(int i=1; i<nvr; i++)
   {
      xvrehi[k] = zvrehi[k]; xvrelo[k] = zvrelo[k];
      xvimhi[k] = zvimhi[k]; xvimlo[k] = zvimlo[k];
      ix2 = idx[i]*deg1+k;
      yvrehi[k] = inputrehi[ix2]; yvrelo[k] = inputrelo[ix2];
      yvimhi[k] = inputimhi[ix2]; yvimlo[k] = inputimlo[ix2];
      __syncthreads();                                 // f[i] = f[i-i]*x[i]
      cmplx2_convolute(xvrehi,xvrelo,xvimhi,xvimlo,
                       yvrehi,yvrelo,yvimhi,yvimlo,
                       zvrehi,zvrelo,zvimhi,zvimlo,deg1,k);
      ix1 = i*deg1+k;                                   
      forwardrehi[ix1] = zvrehi[k]; forwardrelo[ix1] = zvrelo[k];
      forwardimhi[ix1] = zvimhi[k]; forwardimlo[ix1] = zvimlo[k]; 
   }
   if(nvr > 2)
   {
      ix1 = idx[nvr-1]*deg1+k;
      xvrehi[k] = inputrehi[ix1]; xvrelo[k] = inputrelo[ix1];
      xvimhi[k] = inputimhi[ix1]; xvimlo[k] = inputimlo[ix1];
      ix2 = idx[nvr-2]*deg1+k;
      yvrehi[k] = inputrehi[ix2]; yvrelo[k] = inputrelo[ix2];
      yvimhi[k] = inputimhi[ix2]; yvimlo[k] = inputimlo[ix2];
      __syncthreads();                               // b[0] = x[n-1]*x[n-2]
      cmplx2_convolute(xvrehi,xvrelo,xvimhi,xvimlo,
                       yvrehi,yvrelo,yvimhi,yvimlo,
                       zvrehi,zvrelo,zvimhi,zvimlo,deg1,k);
      backwardrehi[k] = zvrehi[k]; backwardrelo[k] = zvrelo[k];
      backwardimhi[k] = zvimhi[k]; backwardimlo[k] = zvimlo[k];

      for(int i=1; i<nvr-2; i++)
      {
         xvrehi[k] = zvrehi[k]; xvrelo[k] = zvrelo[k];
         xvimhi[k] = zvimhi[k]; xvimlo[k] = zvimlo[k];
         ix2 = idx[nvr-2-i]*deg1+k;
         yvrehi[k] = inputrehi[ix2]; yvrelo[k] = inputrelo[ix2];
         yvimhi[k] = inputimhi[ix2]; yvimlo[k] = inputimlo[ix2];
         __syncthreads();                           // b[i] = b[i]*x[n-2-i]
         cmplx2_convolute(xvrehi,xvrelo,xvimhi,xvimlo,
                          yvrehi,yvrelo,yvimhi,yvimlo,
                          zvrehi,zvrelo,zvimhi,zvimlo,deg1,k);
         ix1 = i*deg1+k;
         backwardrehi[ix1] = zvrehi[k]; backwardrelo[ix1] = zvrelo[k];
         backwardimhi[ix1] = zvimhi[k]; backwardimlo[ix1] = zvimlo[k];
      }
      xvrehi[k] = zvrehi[k]; xvrelo[k] = zvrelo[k];
      xvimhi[k] = zvimhi[k]; xvimlo[k] = zvimlo[k];
      yvrehi[k] = cffrehi[k]; yvrelo[k] = cffrelo[k];
      yvimhi[k] = cffimhi[k]; yvimlo[k] = cffimlo[k];
      __syncthreads();                               // b[n-3] = b[n-3]*cff
      cmplx2_convolute(xvrehi,xvrelo,xvimhi,xvimlo,
                       yvrehi,yvrelo,yvimhi,yvimlo,
                       zvrehi,zvrelo,zvimhi,zvimlo,deg1,k);
      ix1 = (nvr-3)*deg1+k;
      backwardrehi[ix1] = zvrehi[k]; backwardrelo[ix1] = zvrelo[k];
      backwardimhi[ix1] = zvimhi[k]; backwardimlo[ix1] = zvimlo[k];

      if(nvr == 3)
      {
         xvrehi[k] = forwardrehi[k]; xvrelo[k] = forwardrelo[k];
         xvimhi[k] = forwardimhi[k]; xvimlo[k] = forwardimlo[k];
         ix2 = idx[2]*deg1+k;
         yvrehi[k] = inputrehi[ix2]; yvrelo[k] = inputrelo[ix2];
         yvimhi[k] = inputimhi[ix2]; yvimlo[k] = inputimlo[ix2];
         __syncthreads();                               // c[0] = f[0]*x[2]
         cmplx2_convolute(xvrehi,xvrelo,xvimhi,xvimlo,
                          yvrehi,yvrelo,yvimhi,yvimlo,
                          zvrehi,zvrelo,zvimhi,zvimlo,deg1,k);
         crossrehi[k] = zvrehi[k]; crossrelo[k] = zvrelo[k];
         crossimhi[k] = zvimhi[k]; crossimlo[k] = zvimlo[k];
      }
      else
      {
         for(int i=0; i<nvr-3; i++)
         {
            ix1 = i*deg1+k;   
            xvrehi[k] = forwardrehi[ix1]; xvrelo[k] = forwardrelo[ix1];
            xvimhi[k] = forwardimhi[ix1]; xvimlo[k] = forwardimlo[ix1];
            ix2 = (nvr-4-i)*deg1+k;
            yvrehi[k] = backwardrehi[ix2]; yvrelo[k] = backwardrelo[ix2];
            yvimhi[k] = backwardimhi[ix2]; yvimlo[k] = backwardimlo[ix2];
            __syncthreads();                        // c[i] = f[i]*b[n-4-i]
            cmplx2_convolute(xvrehi,xvrelo,xvimhi,xvimlo,
                             yvrehi,yvrelo,yvimhi,yvimlo,
                             zvrehi,zvrelo,zvimhi,zvimlo,deg1,k);
            ix1 = i*deg1+k;
            crossrehi[ix1] = zvrehi[k]; crossrelo[ix1] = zvrelo[k];
            crossimhi[ix1] = zvimhi[k]; crossimlo[ix1] = zvimlo[k];
         }
         ix1 = (nvr-3)*deg1+k;
         xvrehi[k] = forwardrehi[ix1]; xvrelo[k] = forwardrelo[ix1];
         xvimhi[k] = forwardimhi[ix1]; xvimlo[k] = forwardimlo[ix1];
         ix2 = idx[nvr-1]*deg1+k;
         yvrehi[k] = inputrehi[ix2]; yvrelo[k] = inputrelo[ix2];
         yvimhi[k] = inputimhi[ix2]; yvimlo[k] = inputimlo[ix2];
         __syncthreads();                         // c[n-3] = f[n-3]*x[n-1]
         cmplx2_convolute(xvrehi,xvrelo,xvimhi,xvimlo,
                          yvrehi,yvrelo,yvimhi,yvimlo,
                          zvrehi,zvrelo,zvimhi,zvimlo,deg1,k);
         ix1 = (nvr-3)*deg1+k;
         crossrehi[ix1] = zvrehi[k]; crossrelo[ix1] = zvrelo[k];
         crossimhi[ix1] = zvimhi[k]; crossimlo[ix1] = zvimlo[k];
      }
   }
}

void GPU_dbl2_evaldiff
 ( int BS, int dim, int nvr, int deg, int *idx, double *cffhi, double *cfflo,
   double **inputhi, double **inputlo, double **outputhi, double **outputlo )
{
   const int deg1 = deg+1;            // length of all vectors
   double *inputhi_d;                 // inputhi_d is input on the device
   double *inputlo_d;                 // inputlo_d is input on the device
   double *forwardhi_d;               // high forward products on the device
   double *forwardlo_d;               // low forward products on the device
   double *backwardhi_d;              // high backward products on the device
   double *backwardlo_d;              // low backward products on the device
   double *crosshi_d;                 // high cross products on the device
   double *crosslo_d;                 // low cross products on the device
   double *cffhi_d;                   // cffhi_d is cffhi on device
   double *cfflo_d;                   // cfflo_d is cfflo on device
   int *idx_d;                        // idx_d is idx on device

   size_t szcff = deg1*sizeof(double);
   size_t szdim = dim*(deg1)*sizeof(double);
   size_t sznvr = nvr*(deg1)*sizeof(double);
   size_t sznvr2 = (nvr-2)*(deg1)*sizeof(double);
   size_t szidx = nvr*sizeof(int);

   hipMalloc((void**)&idx_d,szidx);
   hipMalloc((void**)&cffhi_d,szcff);
   hipMalloc((void**)&cfflo_d,szcff);
   hipMalloc((void**)&inputhi_d,szdim);
   hipMalloc((void**)&inputlo_d,szdim);
   hipMalloc((void**)&forwardhi_d,sznvr);
   hipMalloc((void**)&forwardlo_d,sznvr);
   hipMalloc((void**)&backwardhi_d,sznvr2);
   hipMalloc((void**)&backwardlo_d,sznvr2);
   hipMalloc((void**)&crosshi_d,sznvr2);
   hipMalloc((void**)&crosslo_d,sznvr2);

   double *inputhi_h = new double[dim*(deg1)];
   double *inputlo_h = new double[dim*(deg1)];
   int ix = 0;
   for(int i=0; i<dim; i++)
      for(int j=0; j<deg1; j++)
      {
         inputhi_h[ix] = inputhi[i][j];
         inputlo_h[ix++] = inputlo[i][j];
      }

   hipMemcpy(idx_d,idx,szidx,hipMemcpyHostToDevice);
   hipMemcpy(cffhi_d,cffhi,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cfflo_d,cfflo,szcff,hipMemcpyHostToDevice);
   hipMemcpy(inputhi_d,inputhi_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputlo_d,inputlo_h,szdim,hipMemcpyHostToDevice);

   if(BS == deg1)
   {
      GPU_dbl2_speel<<<1,BS>>>
         (nvr,deg,idx_d,cffhi_d,cfflo_d,inputhi_d,inputlo_d,forwardhi_d,
          forwardlo_d,backwardhi_d,backwardlo_d,crosshi_d,crosslo_d);
   }
   double *forwardhi_h = new double[(deg1)*nvr];
   double *forwardlo_h = new double[(deg1)*nvr];
   double *backwardhi_h = new double[(deg1)*(nvr-2)];
   double *backwardlo_h = new double[(deg1)*(nvr-2)];
   double *crosshi_h = new double[(deg1)*(nvr-2)];
   double *crosslo_h = new double[(deg1)*(nvr-2)];
  
   hipMemcpy(forwardhi_h,forwardhi_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardlo_h,forwardlo_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(backwardhi_h,backwardhi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardlo_h,backwardlo_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crosshi_h,crosshi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crosslo_h,crosslo_d,sznvr2,hipMemcpyDeviceToHost);

   int offset = (nvr-1)*deg1;            // assign value of the monomial
   for(int i=0; i<deg1; i++)
   {
      outputhi[dim][i] = forwardhi_h[offset+i];
      outputlo[dim][i] = forwardlo_h[offset+i];
   }
   ix = idx[nvr-1];                      // derivative with respect to x[n-1]
   offset = (nvr-2)*deg1;
   for(int i=0; i<deg1; i++)
   {
      outputhi[ix][i] = forwardhi_h[offset+i];
      outputlo[ix][i] = forwardlo_h[offset+i];
   }
   ix = idx[0];                          // derivative with respect to x[0]
   offset = (nvr-3)*deg1;
   for(int i=0; i<deg1; i++)
   {
      outputhi[ix][i] = backwardhi_h[offset+i];
      outputlo[ix][i] = backwardlo_h[offset+i];
   }
   for(int k=1; k<nvr-1; k++)            // derivative with respect to x[k]
   {
      ix = idx[k]; offset = (k-1)*deg1;
      for(int i=0; i<deg1; i++)
      {
         outputhi[ix][i] = crosshi_h[offset+i];
         outputlo[ix][i] = crosslo_h[offset+i];
      }
   }
}

void GPU_cmplx2_evaldiff
 ( int BS, int dim, int nvr, int deg, int *idx,
   double *cffrehi, double *cffrelo, double *cffimhi, double *cffimlo,
   double **inputrehi, double **inputrelo, double **inputimhi,
   double **inputimlo, double **outputrehi, double **outputrelo,
   double **outputimhi, double **outputimlo )
{
   const int deg1 = deg+1;          // length of all vectors
   double *inputrehi_d;             // inputrehi_d is inputrehi on the device
   double *inputrelo_d;             // inputrelo_d is inputrelo on the device
   double *inputimhi_d;             // inputimhi_d is inputrehi on the device
   double *inputimlo_d;             // inputimlo_d is inputrelo on the device
   double *forwardrehi_d;
   double *forwardrelo_d;
   double *forwardimhi_d;
   double *forwardimlo_d;
   double *backwardrehi_d;
   double *backwardrelo_d;
   double *backwardimhi_d;
   double *backwardimlo_d;
   double *crossrehi_d;
   double *crossrelo_d;
   double *crossimhi_d;
   double *crossimlo_d;
   double *cffrehi_d;               // cffrehi_d is cffrehi on the device
   double *cffrelo_d;               // cffrelo_d is cffrelo on the device
   double *cffimhi_d;               // cffimhi_d is cffimhi on the device
   double *cffimlo_d;               // cffimlo_d is cffimlo on the device
   int *idx_d;                      // idx_d is idx on the device

   size_t szdim = dim*(deg1)*sizeof(double);
   size_t sznvr = nvr*(deg1)*sizeof(double);
   size_t sznvr2 = (nvr-2)*(deg1)*sizeof(double);
   size_t szidx = nvr*sizeof(int);
   size_t szcff = deg1*sizeof(double);

   hipMalloc((void**)&idx_d,szidx);
   hipMalloc((void**)&cffrehi_d,szcff);
   hipMalloc((void**)&cffrelo_d,szcff);
   hipMalloc((void**)&cffimhi_d,szcff);
   hipMalloc((void**)&cffimlo_d,szcff);
   hipMalloc((void**)&inputrehi_d,szdim);
   hipMalloc((void**)&inputrelo_d,szdim);
   hipMalloc((void**)&inputimhi_d,szdim);
   hipMalloc((void**)&inputimlo_d,szdim);
   hipMalloc((void**)&forwardrehi_d,sznvr);
   hipMalloc((void**)&forwardrelo_d,sznvr);
   hipMalloc((void**)&forwardimhi_d,sznvr);
   hipMalloc((void**)&forwardimlo_d,sznvr);
   hipMalloc((void**)&backwardrehi_d,sznvr2);
   hipMalloc((void**)&backwardrelo_d,sznvr2);
   hipMalloc((void**)&backwardimhi_d,sznvr2);
   hipMalloc((void**)&backwardimlo_d,sznvr2);
   hipMalloc((void**)&crossrehi_d,sznvr2);
   hipMalloc((void**)&crossrelo_d,sznvr2);
   hipMalloc((void**)&crossimhi_d,sznvr2);
   hipMalloc((void**)&crossimlo_d,sznvr2);

   double *inputrehi_h = new double[dim*(deg1)];
   double *inputrelo_h = new double[dim*(deg1)];
   double *inputimhi_h = new double[dim*(deg1)];
   double *inputimlo_h = new double[dim*(deg1)];
   int ix = 0;
   for(int i=0; i<dim; i++)
      for(int j=0; j<deg1; j++)
      {
         inputrehi_h[ix] = inputrehi[i][j];
         inputrelo_h[ix] = inputrelo[i][j];
         inputimhi_h[ix] = inputimhi[i][j];
         inputimlo_h[ix++] = inputimlo[i][j];
      }

   hipMemcpy(idx_d,idx,szidx,hipMemcpyHostToDevice);
   hipMemcpy(cffrehi_d,cffrehi,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffrelo_d,cffrelo,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffimhi_d,cffimhi,szcff,hipMemcpyHostToDevice);
   hipMemcpy(cffimlo_d,cffimlo,szcff,hipMemcpyHostToDevice);
   hipMemcpy(inputrehi_d,inputrehi_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputrelo_d,inputrelo_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputimhi_d,inputimhi_h,szdim,hipMemcpyHostToDevice);
   hipMemcpy(inputimlo_d,inputimlo_h,szdim,hipMemcpyHostToDevice);

   if(BS == deg1)
   {
      GPU_cmplx2_speel<<<1,BS>>>
         (nvr,deg,idx_d,cffrehi_d,cffrelo_d,cffimhi_d,cffimlo_d,
          inputrehi_d,inputrelo_d,inputimhi_d,inputimlo_d,
          forwardrehi_d,forwardrelo_d,forwardimhi_d,forwardimlo_d,
          backwardrehi_d,backwardrelo_d,backwardimhi_d,backwardimlo_d,
          crossrehi_d,crossrelo_d,crossimhi_d,crossimlo_d);
   }
   double *forwardrehi_h = new double[(deg1)*nvr];
   double *forwardrelo_h = new double[(deg1)*nvr];
   double *forwardimhi_h = new double[(deg1)*nvr];
   double *forwardimlo_h = new double[(deg1)*nvr];
   double *backwardrehi_h = new double[(deg1)*(nvr-2)];
   double *backwardrelo_h = new double[(deg1)*(nvr-2)];
   double *backwardimhi_h = new double[(deg1)*(nvr-2)];
   double *backwardimlo_h = new double[(deg1)*(nvr-2)];
   double *crossrehi_h = new double[(deg1)*(nvr-2)];
   double *crossrelo_h = new double[(deg1)*(nvr-2)];
   double *crossimhi_h = new double[(deg1)*(nvr-2)];
   double *crossimlo_h = new double[(deg1)*(nvr-2)];
  
   hipMemcpy(forwardrehi_h,forwardrehi_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardrelo_h,forwardrelo_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardimhi_h,forwardimhi_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(forwardimlo_h,forwardimlo_d,sznvr,hipMemcpyDeviceToHost);
   hipMemcpy(backwardrehi_h,backwardrehi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardrelo_h,backwardrelo_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardimhi_h,backwardimhi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(backwardimlo_h,backwardimlo_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossrehi_h,crossrehi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossrelo_h,crossrelo_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossimhi_h,crossimhi_d,sznvr2,hipMemcpyDeviceToHost);
   hipMemcpy(crossimlo_h,crossimlo_d,sznvr2,hipMemcpyDeviceToHost);

   int offset = (nvr-1)*deg1;
   for(int i=0; i<deg1; i++)   // assign value of the monomial
   {
      outputrehi[dim][i] = forwardrehi_h[offset+i];
      outputrelo[dim][i] = forwardrelo_h[offset+i];
      outputimhi[dim][i] = forwardimhi_h[offset+i];
      outputimlo[dim][i] = forwardimlo_h[offset+i];
   }
   ix = idx[nvr-1];
   offset = (nvr-2)*deg1;
   for(int i=0; i<deg1; i++)  // derivative with respect to x[n-1]
   {
      outputrehi[ix][i] = forwardrehi_h[offset+i];
      outputrelo[ix][i] = forwardrelo_h[offset+i];
      outputimhi[ix][i] = forwardimhi_h[offset+i];
      outputimlo[ix][i] = forwardimlo_h[offset+i];
   }
   ix = idx[0]; 
   offset = (nvr-3)*deg1;
   for(int i=0; i<deg1; i++)   // derivative with respect to x[0]
   {
      outputrehi[ix][i] = backwardrehi_h[offset+i];
      outputrelo[ix][i] = backwardrelo_h[offset+i];
      outputimhi[ix][i] = backwardimhi_h[offset+i];
      outputimlo[ix][i] = backwardimlo_h[offset+i];
   }
   for(int k=1; k<nvr-1; k++)  // derivative with respect to x[k]
   {
      ix = idx[k]; offset = (k-1)*deg1;
      for(int i=0; i<deg1; i++)
      {
         outputrehi[ix][i] = crossrehi_h[offset+i];
         outputrelo[ix][i] = crossrelo_h[offset+i];
         outputimhi[ix][i] = crossimhi_h[offset+i];
         outputimlo[ix][i] = crossimlo_h[offset+i];
      }
   }
}
