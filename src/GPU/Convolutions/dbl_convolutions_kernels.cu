#include "hip/hip_runtime.h"
// The file dbl_convolutions_kernels.cu defines kernels with prototypes
// in dbl_convolution_kernels.h.

#include "dbl_convolutions_kernels.h"

__global__ void dbl_convolute
 ( double *x, double *y, double *z, int dim )
{
   int k = threadIdx.x;                 // thread k computes z[k]

   __shared__ double xv[d_shmemsize];
   __shared__ double yv[d_shmemsize];
   __shared__ double zv[d_shmemsize];

   xv[k] = x[k];
   yv[k] = y[k];

   zv[k] = xv[0]*yv[k];

   for(int i=1; i<=k; i++) zv[k] = zv[k] + xv[i]*yv[k-i];

   __syncthreads();

   z[k] = zv[k];
}

__global__ void cmplx_convolute
 ( double *xre, double *xim, double *yre, double *yim,
   double *zre, double *zim, int dim )
{
   int k = threadIdx.x;       // thread k computes zre[k] and zim[k]

   __shared__ double xvre[d_shmemsize];
   __shared__ double xvim[d_shmemsize];
   __shared__ double yvre[d_shmemsize];
   __shared__ double yvim[d_shmemsize];
   __shared__ double zvre[d_shmemsize];
   __shared__ double zvim[d_shmemsize];

   double xr,xi,yr,yi,zr,zi;

   xvre[k] = xre[k]; xvim[k] = xim[k];
   yvre[k] = yre[k]; yvim[k] = yim[k];

   xr = xvre[0]; xi = xvim[0];    // z[k] = x[0]*y[k]
   yr = yvre[k]; yi = yvim[k];
   zr = xr*yr - xi*yi;
   zi = xr*yi + xi*yr;
   zvre[k] = zr;
   zvim[k] = zi;

   for(int i=1; i<=k; i++) // z[k] = z[k] + x[i]*y[k-i]
   {
      xr = xvre[i];   xi = xvim[i];
      yr = yvre[k-i]; yi = yvim[k-i];
      zr = xr*yr - xi*yi;
      zi = xr*yi + xi*yr;
      zvre[k] += zr;
      zvim[k] += zi;
   }
   __syncthreads();

   zre[k] = zvre[k];
   zim[k] = zvim[k];
}

void GPU_dbl_product
 ( double *x_h, double *y_h, double *z_h, int deg, int freq, int BS )
{
   const int dim = deg+1;            // length of all vectors
   double* x_d;                      // x_d is x_h on the device
   double* y_d;                      // y_d is y_h on the device
   double* z_d;                      // z_d is z_h on the device
   size_t size = dim*sizeof(double); // number of bytes for each vector

   hipMalloc((void**)&x_d,size);
   hipMalloc((void**)&y_d,size);
   hipMalloc((void**)&z_d,size);
   hipMemcpy(x_d,x_h,size,hipMemcpyHostToDevice);
   hipMemcpy(y_d,y_h,size,hipMemcpyHostToDevice);
   hipMemcpy(z_d,z_h,size,hipMemcpyHostToDevice);

   if(dim == BS)
   {
      for(int i=0; i<freq; i++)
         dbl_convolute<<<1,BS>>>(x_d,y_d,z_d,dim);
   }

   hipMemcpy(z_h,z_d,size,hipMemcpyDeviceToHost);
}

void GPU_cmplx_product
 ( double *xre_h, double *xim_h, double *yre_h, double *yim_h,
   double *zre_h, double *zim_h, int deg, int freq, int BS )
{
   const int dim = deg+1;            // length of all vectors
   double* xre_d;                    // xre_d is xre_h on the device
   double* xim_d;                    // xim_d is xim_h on the device
   double* yre_d;                    // yre_d is yre_h on the device
   double* yim_d;                    // yim_d is yim_h on the device
   double* zre_d;                    // zre_d is zre_h on the device
   double* zim_d;                    // zim_d is zim_h on the device
   size_t size = dim*sizeof(double); // number of bytes for each vector

   hipMalloc((void**)&xre_d,size);
   hipMalloc((void**)&xim_d,size);
   hipMalloc((void**)&yre_d,size);
   hipMalloc((void**)&yim_d,size);
   hipMalloc((void**)&zre_d,size);
   hipMalloc((void**)&zim_d,size);
   hipMemcpy(xre_d,xre_h,size,hipMemcpyHostToDevice);
   hipMemcpy(xim_d,xim_h,size,hipMemcpyHostToDevice);
   hipMemcpy(yre_d,yre_h,size,hipMemcpyHostToDevice);
   hipMemcpy(yim_d,yim_h,size,hipMemcpyHostToDevice);
   hipMemcpy(zre_d,zre_h,size,hipMemcpyHostToDevice);
   hipMemcpy(zim_d,zim_h,size,hipMemcpyHostToDevice);

   if(dim == BS)
   {
      for(int i=0; i<freq; i++)
         cmplx_convolute<<<1,BS>>>(xre_d,xim_d,yre_d,yim_d,zre_d,zim_d,dim);
   }

   hipMemcpy(zre_h,zre_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(zim_h,zim_d,size,hipMemcpyDeviceToHost);
}
