#include "hip/hip_runtime.h"
// Defines code of the functions in cmplx2_norm_kernels.h,
// to compute the 2-norm and normalize a complex vector,
// in double double precision,
// for vectors of small, medium, and large size.

#include <iostream>
#include <cmath>
#include <assert.h>
#include <cstdio>
#include "double_double_gpufun.cu"
#include "cmplx2_norm_kernels.h"

using namespace std;

__global__ void small_normalize_vector
 ( double* vrehi, double* vrelo, double* vimhi, double* vimlo,
   int dim, int dimLog2, double* normhi, double* normlo )
{
   int j = threadIdx.x;

   __shared__ double shvrehi[dd_shmemsize];
   __shared__ double shvrelo[dd_shmemsize];
   __shared__ double shvimhi[dd_shmemsize];
   __shared__ double shvimlo[dd_shmemsize];
   __shared__ double prdhi[dd_shmemsize];
   __shared__ double prdlo[dd_shmemsize];
   __shared__ double sumhi[dd_shmemsize];
   __shared__ double sumlo[dd_shmemsize];

   shvrehi[j] = vrehi[j]; // reading real parts into shared memory
   shvrelo[j] = vrelo[j];
   shvimhi[j] = vimhi[j]; // reading imaginary parts into shared memory
   shvimlo[j] = vimlo[j];

   ddg_sqr(shvrehi[j],shvrelo[j],&sumhi[j],&sumlo[j]);
   ddg_sqr(shvimhi[j],shvimlo[j],&prdhi[j],&prdlo[j]);
   ddg_inc(&sumhi[j],&sumlo[j],prdhi[j],prdlo[j]);

   __syncthreads();
   int powTwo = 1;                          // sum reduction
   for(int k=0; k < dimLog2; k++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < dim)
            ddg_inc(&sumhi[j],&sumlo[j],sumhi[j+powTwo],sumlo[j+powTwo]);
      powTwo = powTwo*2;
      __syncthreads();
   }
   // thread 0 computes the sqrt of the inner product, others wait
   if(j == 0) ddg_sqrt(sumhi[0],sumlo[0],&sumhi[0],&sumlo[0]); 
   if(j == 0)
   {
      *normhi = sumhi[0];
      *normlo = sumlo[0];
   }
   __syncthreads();
   ddg_div(shvrehi[j],shvrelo[j],sumhi[0],sumlo[0],&vrehi[j],&vrelo[j]);
   ddg_div(shvimhi[j],shvimlo[j],sumhi[0],sumlo[0],&vimhi[j],&vimlo[j]);
}

__global__ void medium_normalize_vector
 ( double* vrehi, double* vrelo, double* vimhi, double* vimlo,
   int dim, int rnd, int rndLog2, int BS, int BSLog2,
   double* normhi, double* normlo )
{
   int j = threadIdx.x;
   int powTwo;
   int vBSind = 0;

   __shared__ double shvrehi[dd_shmemsize];
   __shared__ double shvrelo[dd_shmemsize];
   __shared__ double shvimhi[dd_shmemsize];
   __shared__ double shvimlo[dd_shmemsize];
   __shared__ double prdhi[dd_shmemsize];
   __shared__ double prdlo[dd_shmemsize];
   __shared__ double acchi[dd_shmemsize];
   __shared__ double acclo[dd_shmemsize];
   __shared__ double sumshi[maxrounds];
   __shared__ double sumslo[maxrounds];

   for(int i=0; i<rnd; i++)
   {
      if(vBSind+j >= dim)       // at last round some threads are excluded
      {
         acchi[j] = 0.0;
         acclo[j] = 0.0;
      }
      else
      {
         shvrehi[j] = vrehi[vBSind+j];  // reading into shared memory
         shvrelo[j] = vrelo[vBSind+j];
         shvimhi[j] = vimhi[vBSind+j]; 
         shvimlo[j] = vimlo[vBSind+j]; 
         ddg_sqr(shvrehi[j],shvrelo[j],&acchi[j],&acclo[j]);
         ddg_sqr(shvimhi[j],shvimlo[j],&prdhi[j],&prdlo[j]);
         ddg_inc(&acchi[j],&acclo[j],prdhi[j],prdlo[j]);
      }
      __syncthreads();
      powTwo = 1;                          // sum reduction
      for(int k=0; k < BSLog2; k++)
      {
         if((j%(powTwo*2)) == 0)
            if(j+powTwo < BS)
               ddg_inc(&acchi[j],&acclo[j],acchi[j+powTwo],acclo[j+powTwo]);
         powTwo = powTwo*2;
         __syncthreads();
      }
      // thread 0 copies the sum of this round in sums[i], the others wait
      if(j == 0)
      {
         sumshi[i] = acchi[0]; 
         sumslo[i] = acclo[0]; 
      }
      __syncthreads();
      vBSind = vBSind + BS;
   }
   powTwo = 1;                          // sum reduction
   for(int k=0; k < rndLog2; k++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < rnd)
            ddg_inc(&sumshi[j],&sumslo[j],sumshi[j+powTwo],sumslo[j+powTwo]);
      powTwo = powTwo*2;
      __syncthreads();
   }
   if(j == 0)
   {
      ddg_sqrt(sumshi[0],sumslo[0],&sumshi[0],&sumslo[0]);
      *normhi = sumshi[0];
      *normlo = sumslo[0];
   }
   __syncthreads();
   vBSind = 0;
   for(int i=0; i<rnd; i++)
   {
      if(vBSind+j < dim)
      {
         shvrehi[j] = vrehi[vBSind+j];       // read into shared memory
         shvrelo[j] = vrelo[vBSind+j];
         shvimhi[j] = vimhi[vBSind+j];
         shvimlo[j] = vimlo[vBSind+j];
         // normalize vector
         ddg_div(shvrehi[j],shvrelo[j],sumshi[0],sumslo[0],
                 &vrehi[vBSind+j],&vrelo[vBSind+j]);
         ddg_div(shvimhi[j],shvimlo[j],sumshi[0],sumslo[0],
                 &vimhi[vBSind+j],&vimlo[vBSind+j]);
      }
      __syncthreads();
      vBSind = vBSind + BS;
   }
}

__global__ void large_sum_the_squares
 ( double* vrehi, double* vrelo, double* vimhi, double* vimlo,
   int dim, double* sumshi, double* sumslo, int BS, int BSLog2 )
{
   const int i = blockIdx.x;
   const int j = threadIdx.x;
   const int k = i*BS + j;

   __shared__ double shvrehi[dd_shmemsize];
   __shared__ double shvrelo[dd_shmemsize];
   __shared__ double shvimhi[dd_shmemsize];
   __shared__ double shvimlo[dd_shmemsize];
   __shared__ double prdhi[dd_shmemsize];
   __shared__ double prdlo[dd_shmemsize];
   __shared__ double acchi[dd_shmemsize];
   __shared__ double acclo[dd_shmemsize];

   shvrehi[j] = vrehi[k];
   shvrelo[j] = vrelo[k];
   shvimhi[j] = vimhi[k];
   shvimlo[j] = vimlo[k];

   ddg_sqr(shvrehi[j],shvrelo[j],&acchi[j],&acclo[j]);
   ddg_sqr(shvimhi[j],shvimlo[j],&prdhi[j],&prdlo[j]);
   ddg_inc(&acchi[j],&acclo[j],prdhi[j],prdlo[j]);

   __syncthreads();

   int powTwo = 1;                          // sum reduction
   for(int L=0; L < BSLog2; L++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < BS)
            ddg_inc(&acchi[j],&acclo[j],acchi[j+powTwo],acclo[j+powTwo]);
      powTwo = powTwo*2;

      __syncthreads();
   }
   if(j == 0)                               // thread 0 writes the sum
   {
      sumshi[i] = acchi[0];
      sumslo[i] = acclo[0];
   }
}

__global__ void large_normalize_vector
 ( double* vrehi, double* vrelo, double* vimhi, double* vimlo,
   int dim, double* sumshi, double* sumslo, int nbsums, int nbsumsLog2,
   int BS, double* normhi, double* normlo )
{
   const int i = blockIdx.x;
   const int j = threadIdx.x;
   const int k = i*BS + j;

   __shared__ double shvrehi[dd_shmemsize];
   __shared__ double shvrelo[dd_shmemsize];
   __shared__ double shvimhi[dd_shmemsize];
   __shared__ double shvimlo[dd_shmemsize];

   if(j < nbsums)
   {
      shvrehi[j] = sumshi[j];
      shvrelo[j] = sumslo[j];
   }

   __syncthreads();

   int powTwo = 1;                          // sum reduction
   for(int L=0; L < nbsumsLog2; L++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < nbsums)
            ddg_inc(&shvrehi[j],&shvrelo[j],
                    shvrehi[j+powTwo],shvrelo[j+powTwo]);
      powTwo = powTwo*2;

      __syncthreads();
   }
   __syncthreads();                      // every thread 0 of all blocks
   if(j == 0)                            // compute the 2-norm and assigns
   {                                     // to the output parameter
      ddg_sqrt(shvrehi[0],shvrelo[0],normhi,normlo);
   }
   __syncthreads();                    

   if(k < dim)
   {
      shvrehi[j] = vrehi[k];
      shvrelo[j] = vrelo[k];
      shvimhi[j] = vimhi[k];
      shvimlo[j] = vimlo[k];

      ddg_div(shvrehi[j],shvrelo[j],*normhi,*normlo,&shvrehi[j],&shvrelo[j]);
      ddg_div(shvimhi[j],shvimlo[j],*normhi,*normlo,&shvimhi[j],&shvimlo[j]);

      vrehi[k] = shvrehi[j];
      vrelo[k] = shvrelo[j];
      vimhi[k] = shvimhi[j];
      vimlo[k] = shvimlo[j];
   }
}

void GPU_norm
 ( double* vrehi_h, double* vrelo_h, double* vimhi_h, double* vimlo_h,
   int dim, int freq, int BS, double* normhi, double* normlo, int blocked )
{
   int BSLog2 = ceil(log2((double) BS)); // ceil for sum reduction

   double* vrehi_d;                      // high real parts on device
   double* vrelo_d;                      // low real parts on device
   double* vimhi_d;                      // high imaginary parts on device
   double* vimlo_d;                      // low imaginary parts on device
   size_t size = dim*sizeof(double);
   hipMalloc((void**)&vrehi_d,size);
   hipMalloc((void**)&vrelo_d,size);
   hipMalloc((void**)&vimhi_d,size);
   hipMalloc((void**)&vimlo_d,size);
   hipMemcpy(vrehi_d,vrehi_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrelo_d,vrelo_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimhi_d,vimhi_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimlo_d,vimlo_h,size,hipMemcpyHostToDevice);
   double* normhi_d;
   double* normlo_d;
   hipMalloc((void**)&normhi_d,sizeof(double));
   hipMalloc((void**)&normlo_d,sizeof(double));

   if(dim == BS)
   {
      for(int i=0; i<freq; i++)
         small_normalize_vector<<<1,BS>>>
            (vrehi_d,vrelo_d,vimhi_d,vimlo_d,dim,BSLog2,normhi_d,normlo_d);
   }
   else if(blocked == 0)
   {
      int rf = ceil(((double) dim)/BS);
      int rfLog2 = ceil(log2((double) rf));
      for(int i=0; i<freq; i++)
         medium_normalize_vector<<<1,BS>>>
            (vrehi_d,vrelo_d,vimhi_d,vimlo_d,dim,rf,rfLog2,
             BS,BSLog2,normhi_d,normlo_d);
   }
   else
   {
      const int nblocks = dim/BS;
      const int nblocksLog2 = ceil(log2((double) nblocks));
      double* sumshi_d; // high parts of sums of squares for each block
      double* sumslo_d; // low parts of sums of squares for each block
      size_t sums_size = nblocks*sizeof(double);
      hipMalloc((void**)&sumshi_d,sums_size);
      hipMalloc((void**)&sumslo_d,sums_size);
      for(int i=0; i<freq; i++)
      {
         large_sum_the_squares<<<nblocks,BS>>>
            (vrehi_d,vrelo_d,vimhi_d,vimlo_d,dim,sumshi_d,sumslo_d,BS,BSLog2);
         large_normalize_vector<<<nblocks,BS>>>
            (vrehi_d,vrelo_d,vimhi_d,vimlo_d,dim,sumshi_d,sumslo_d,
             nblocks,nblocksLog2,BS,normhi_d,normlo_d);
      }
   }
   hipMemcpy(vrehi_h,vrehi_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrelo_h,vrelo_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimhi_h,vimhi_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimlo_h,vimlo_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(normhi,normhi_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normlo,normlo_d,sizeof(double),hipMemcpyDeviceToHost);
}
