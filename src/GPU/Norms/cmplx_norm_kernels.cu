#include "hip/hip_runtime.h"
// Defines code of the functions in cmplx_norm_kernels.h,
// to compute the 2-norm and normalize a complex vector in double precision,
// for small and large vectors.

#include <iostream>
#include <cmath>
#include <assert.h>
#include <cstdio>
#include "cmplx_norm_kernels.h"

using namespace std;

__global__ void small_normalize_vector
 ( double* vre, double* vim, int dim, int dimLog2, double* twonorm )
{
   int j = threadIdx.x;
   __shared__ double shvre[d_shmemsize];
   __shared__ double shvim[d_shmemsize];
   __shared__ double prd[d_shmemsize];
   shvre[j] = vre[j];    // reading real parts into shared memory
   shvim[j] = vim[j];    // reading imaginary parts into shared memory
   prd[j] = shvre[j]*shvre[j] + shvim[j]*shvim[j];
   __syncthreads();
   int powTwo = 1;                          // sum reduction
   for(int k=0; k < dimLog2; k++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < dim) prd[j] = prd[j] + prd[j+powTwo];
      powTwo = powTwo*2;
      __syncthreads();
   }
   // thread 0 computes the sqrt of the inner product, others wait
   if(j == 0) prd[0] = sqrt(prd[0]); 
   if(j == 0) *twonorm = prd[0];
   __syncthreads();
   vre[j] = shvre[j]/prd[0];
   vim[j] = shvim[j]/prd[0];
}

__global__ void medium_normalize_vector
 ( double* vre, double* vim, int dim, int rnd, int rndLog2,
   int BS, int BSLog2, double* twonorm )
{
   int j = threadIdx.x;
   int powTwo;
   int vBSind = 0;

   __shared__ double shvre[d_shmemsize];
   __shared__ double shvim[d_shmemsize];
   __shared__ double prd[d_shmemsize];
   __shared__ double sums[maxrounds];

   for(int i=0; i<rnd; i++)
   {
      if(vBSind+j >= dim)       // at last round some threads are excluded
         prd[j] = 0.0;
      else
      {
         shvre[j] = vre[vBSind+j];  // reading of vector into shared memory
         shvim[j] = vim[vBSind+j]; 
         prd[j] = shvre[j]*shvre[j] + shvim[j]*shvim[j];
      }
      __syncthreads();
      powTwo = 1;                          // sum reduction
      for(int k=0; k < BSLog2; k++)
      {
         if((j%(powTwo*2)) == 0)
            if(j+powTwo < BS)
               prd[j] = prd[j] + prd[j+powTwo];
         powTwo = powTwo*2;
         __syncthreads();
      }
      // thread 0 copies the sum of this round in sums[i], the others wait
      if(j == 0) sums[i] = prd[0]; 
      __syncthreads();
      vBSind = vBSind + BS;
   }
   powTwo = 1;                          // sum reduction
   for(int k=0; k < rndLog2; k++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < rnd)
            sums[j] = sums[j] + sums[j+powTwo];
      powTwo = powTwo*2;
      __syncthreads();
   }
   if(j == 0) sums[0] = sqrt(sums[0]);
   if(j == 0) *twonorm = sums[0];
   __syncthreads();
   vBSind = 0;
   for(int i=0; i<rnd; i++)
   {
      if(vBSind+j < dim)
      {
         shvre[j] = vre[vBSind+j];           // read into shared memory
         shvim[j] = vim[vBSind+j];
         vre[vBSind+j] = shvre[j]/sums[0];   // normalize vector
         vim[vBSind+j] = shvim[j]/sums[0];
      }
      __syncthreads();
      vBSind = vBSind + BS;
   }
}

void GPU_norm
 ( double* vre_h, double* vim_h, int dim, int freq, int BS, double* twonorm )
{
   int BSLog2 = ceil(log2((double) BS)); // ceil for sum reduction

   double* vre_d;                   // allocate for real parts on device
   double* vim_d;                   // allocate for imaginary parts on device
   size_t size = dim*sizeof(double);
   hipMalloc((void**)&vre_d,size);
   hipMalloc((void**)&vim_d,size);
   hipMemcpy(vre_d,vre_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vim_d,vim_h,size,hipMemcpyHostToDevice);
   double* twonorm_d;
   hipMalloc((void**)&twonorm_d,sizeof(double));

   if(dim == BS)
      for(int i=0; i<freq; i++)
         small_normalize_vector<<<1,BS>>>(vre_d,vim_d,dim,BSLog2,twonorm_d);
   else
   {
      int rf = ceil(((double) dim)/BS);
      int rfLog2 = ceil(log2((double) rf));
      for(int i=0; i<freq; i++)
         medium_normalize_vector<<<1,BS>>>
            (vre_d,vim_d,dim,rf,rfLog2,BS,BSLog2,twonorm_d);
   }

   hipMemcpy(vre_h,vre_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vim_h,vim_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(twonorm,twonorm_d,sizeof(double),hipMemcpyDeviceToHost);
}
