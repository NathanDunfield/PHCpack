#include "hip/hip_runtime.h"
// This file contains the definition for the functions in dbl3_norm_kernels.h,
// to compute the 2-norm and normalize a triple double precision vector,
// for small, medium, and large vectors.

#include "double_double_gpufun.cu"
#include "triple_double_gpufun.cu"
#include "dbl3_norm_kernels.h"

__global__ void small_normalize_vector
 ( double *vhi, double *vmi, double *vlo, int dim, int dimLog2,
   double *normhi, double *normmi, double *normlo )
{
   int j = threadIdx.x;
   __shared__ double shvhi[td_shmemsize];
   __shared__ double shvmi[td_shmemsize];
   __shared__ double shvlo[td_shmemsize];
   __shared__ double prdhi[td_shmemsize];
   __shared__ double prdmi[td_shmemsize];
   __shared__ double prdlo[td_shmemsize];
   shvhi[j] = vhi[j];    // reading of vector into shared memory
   shvmi[j] = vmi[j];
   shvlo[j] = vlo[j];
   tdg_sqr(shvhi[j],shvmi[j],shvlo[j],&prdhi[j],&prdmi[j],&prdlo[j]);
   __syncthreads();
   int powTwo = 1;                          // sum reduction
   for(int k=0; k < dimLog2; k++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < dim)
            tdg_inc(&prdhi[j],&prdmi[j],&prdlo[j],
                    prdhi[j+powTwo],prdmi[j+powTwo],prdlo[j+powTwo]);
      powTwo = powTwo*2;
      __syncthreads();
   }
   // thread 0 computes the sqrt of the inner product, others wait
   if(j == 0)
      tdg_sqrt(prdhi[0],prdmi[0],prdlo[0],&prdhi[0],&prdmi[0],&prdlo[0]); 
   if(j == 0)
   {
      *normhi = prdhi[0];
      *normmi = prdmi[0];
      *normlo = prdlo[0];
   }
   __syncthreads();
   tdg_div(shvhi[j],shvmi[j],shvlo[j],prdhi[0],prdmi[0],prdlo[0],
           &vhi[j],&vmi[j],&vlo[j]);
}

__global__ void medium_normalize_vector
 ( double *vhi, double *vmi, double *vlo, int dim, int rnd, int rndLog2,
   int BS, int BSLog2, double *normhi, double *normmi, double *normlo )
{
   int j = threadIdx.x;
   int powTwo;
   int vBSind = 0;

   __shared__ double shvhi[td_shmemsize];
   __shared__ double shvmi[td_shmemsize];
   __shared__ double shvlo[td_shmemsize];
   __shared__ double prdhi[td_shmemsize];
   __shared__ double prdmi[td_shmemsize];
   __shared__ double prdlo[td_shmemsize];
   __shared__ double sumshi[maxrounds];
   __shared__ double sumsmi[maxrounds];
   __shared__ double sumslo[maxrounds];

   for(int i=0; i<rnd; i++)
   {
      if(vBSind+j >= dim)       // at last round some threads are excluded
      {
         prdhi[j] = 0.0;
         prdmi[j] = 0.0;
         prdlo[j] = 0.0;
      }
      else
      {
         shvhi[j] = vhi[vBSind+j];  // reading of vector into shared memory
         shvmi[j] = vmi[vBSind+j];
         shvlo[j] = vlo[vBSind+j];
         tdg_sqr(shvhi[j],shvmi[j],shvlo[j],&prdhi[j],&prdmi[j],&prdlo[j]);
      }
      __syncthreads();
      powTwo = 1;                          // sum reduction
      for(int k=0; k < BSLog2; k++)
      {
         if((j%(powTwo*2)) == 0)
            if(j+powTwo < BS)
               tdg_inc(&prdhi[j],&prdmi[j],&prdlo[j],
                       prdhi[j+powTwo],prdmi[j+powTwo],prdlo[j+powTwo]);
         powTwo = powTwo*2;
         __syncthreads();
      }
      // thread 0 copies the sum of this round in sums[i], the others wait
      if(j == 0)
      {
         sumshi[i] = prdhi[0]; 
         sumsmi[i] = prdmi[0]; 
         sumslo[i] = prdlo[0]; 
      }
      __syncthreads();
      vBSind = vBSind + BS;
   }
   powTwo = 1;                          // sum reduction
   for(int k=0; k < rndLog2; k++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < rnd)
            tdg_inc(&sumshi[j],&sumsmi[j],&sumslo[j],
                    sumshi[j+powTwo],sumsmi[j+powTwo],sumslo[j+powTwo]);
      powTwo = powTwo*2;
      __syncthreads();
   }
   if(j == 0) tdg_sqrt(sumshi[0],sumsmi[0],sumslo[0],
                       &sumshi[0],&sumsmi[0],&sumslo[0]); 
   if(j == 0)
   {
      *normhi = sumshi[0];
      *normmi = sumsmi[0];
      *normlo = sumslo[0];
   }
   __syncthreads();
   vBSind = 0;
   for(int i=0; i<rnd; i++)
   {
      if(vBSind+j < dim)
      {
         shvhi[j] = vhi[vBSind+j];   // read into shared memory
         shvmi[j] = vmi[vBSind+j];   // and normalize the vector
         shvlo[j] = vlo[vBSind+j];
         tdg_div(shvhi[j],shvmi[j],shvlo[j],sumshi[0],sumsmi[0],sumslo[0],
                 &vhi[vBSind+j],&vmi[vBSind+j],&vlo[vBSind+j]);
      }
      __syncthreads();
      vBSind = vBSind + BS;
   }
}

__global__ void large_sum_the_squares
 ( double *vhi, double *vmi, double *vlo, int dim,
   double *sumshi, double *sumsmi, double *sumslo, int BS, int BSLog2 )
{
   const int i = blockIdx.x;
   const int j = threadIdx.x;
   const int k = i*BS + j;

   __shared__ double shvhi[td_shmemsize];
   __shared__ double shvmi[td_shmemsize];
   __shared__ double shvlo[td_shmemsize];
   __shared__ double prdhi[td_shmemsize];
   __shared__ double prdmi[td_shmemsize];
   __shared__ double prdlo[td_shmemsize];

   shvhi[j] = vhi[k];
   shvmi[j] = vmi[k];
   shvlo[j] = vlo[k];
   tdg_sqr(shvhi[j],shvmi[j],shvlo[j],&prdhi[j],&prdmi[j],&prdlo[j]);

   __syncthreads();

   int powTwo = 1;                          // sum reduction
   for(int L=0; L < BSLog2; L++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < BS) 
            tdg_inc(&prdhi[j],&prdmi[j],&prdlo[j],
                    prdhi[j+powTwo],prdmi[j+powTwo],prdlo[j+powTwo]);
      powTwo = powTwo*2;

      __syncthreads();
   }
   if(j == 0)                              // thread 0 writes the sum
   {
      sumshi[i] = prdhi[0];
      sumsmi[i] = prdmi[0];
      sumslo[i] = prdlo[0];
   }
}

__global__ void large_normalize_vector
 ( double *vhi, double *vmi, double *vlo, int dim,
   double *sumshi, double *sumsmi, double *sumslo,
   int nbsums, int nbsumsLog2, int BS,
   double *normhi, double *normmi, double *normlo )
{
   const int i = blockIdx.x;
   const int j = threadIdx.x;
   const int k = i*BS + j;

   __shared__ double shvhi[td_shmemsize];
   __shared__ double shvmi[td_shmemsize];
   __shared__ double shvlo[td_shmemsize];

   if(j < nbsums)
   {
      shvhi[j] = sumshi[j];
      shvmi[j] = sumsmi[j];
      shvlo[j] = sumslo[j];
   }
   __syncthreads();

   int powTwo = 1;                          // sum reduction
   for(int L=0; L < nbsumsLog2; L++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < nbsums)
            tdg_inc(&shvhi[j],&shvmi[j],&shvlo[j],
                    shvhi[j+powTwo],shvmi[j+powTwo],shvlo[j+powTwo]);
      powTwo = powTwo*2;

      __syncthreads();
   }
   __syncthreads();                    // every thread 0 of all blocks
   if(j == 0)                          // compute the 2-norm and assigns
   {                                   // to the output parameter
      tdg_sqrt(shvhi[0],shvmi[0],shvlo[0],normhi,normmi,normlo); 
   }
   __syncthreads();                    // to the output parameter

   if(k < dim)
   {
      shvhi[j] = vhi[k];
      shvmi[j] = vmi[k];
      shvlo[j] = vlo[k];
      tdg_div(shvhi[j],shvmi[j],shvlo[j],*normhi,*normmi,*normlo,
              &shvhi[j],&shvmi[j],&shvlo[j]);
      vhi[k] = shvhi[j];
      vmi[k] = shvmi[j];
      vlo[k] = shvlo[j];
   }
}

void GPU_norm
 ( double *vhi_h, double *vmi_h, double *vlo_h, int dim, int freq, int BS,
   double *normhi, double *normmi, double *normlo, int blocked )
{
   int BSLog2 = ceil(log2((double) BS)); // ceil for sum reduction

   double* vhi_d;                   // allocate for vector on device
   double* vmi_d;
   double* vlo_d;
   size_t size = dim*sizeof(double);
   hipMalloc((void**)&vhi_d,size);
   hipMalloc((void**)&vmi_d,size);
   hipMalloc((void**)&vlo_d,size);
   hipMemcpy(vhi_d,vhi_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vmi_d,vmi_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vlo_d,vlo_h,size,hipMemcpyHostToDevice);
   double* normhi_d;
   double* normmi_d;
   double* normlo_d;
   hipMalloc((void**)&normhi_d,sizeof(double));
   hipMalloc((void**)&normmi_d,sizeof(double));
   hipMalloc((void**)&normlo_d,sizeof(double));

   if(dim == BS)
   {
      for(int i=0; i<freq; i++)
         small_normalize_vector<<<1,BS>>>
            (vhi_d,vmi_d,vlo_d,dim,BSLog2,normhi_d,normmi_d,normlo_d);
   }
   else if(blocked == 0)
   {
      const int rf = ceil(((double) dim)/BS);
      const int rfLog2 = ceil(log2((double) rf));
      for(int i=0; i<freq; i++)
         medium_normalize_vector<<<1,BS>>>
            (vhi_d,vmi_d,vlo_d,dim,rf,rfLog2,BS,BSLog2,
             normhi_d,normmi_d,normlo_d);
   }
   else
   {
      const int nblocks = dim/BS;
      const int nblocksLog2 = ceil(log2((double) nblocks));
      double* sumshi_d; // sums of squares for each block
      double* sumsmi_d; // middle parts of sums of squares
      double* sumslo_d; // low parts of sums of squares
      size_t sums_size = nblocks*sizeof(double);
      hipMalloc((void**)&sumshi_d,sums_size);
      hipMalloc((void**)&sumsmi_d,sums_size);
      hipMalloc((void**)&sumslo_d,sums_size);
      for(int i=0; i<freq; i++)
      {
         large_sum_the_squares<<<nblocks,BS>>>
            (vhi_d,vmi_d,vlo_d,dim,sumshi_d,sumsmi_d,sumslo_d,BS,BSLog2);
         large_normalize_vector<<<nblocks,BS>>>
            (vhi_d,vmi_d,vlo_d,dim,sumshi_d,sumsmi_d,sumslo_d,
             nblocks,nblocksLog2,BS,normhi_d,normmi_d,normlo_d);
      }
   }
   hipMemcpy(vhi_h,vhi_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vmi_h,vmi_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vlo_h,vlo_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(normhi,normhi_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normmi,normmi_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normlo,normlo_d,sizeof(double),hipMemcpyDeviceToHost);
}
