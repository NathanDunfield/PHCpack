#include "hip/hip_runtime.h"
// The file triple_double_gpufun.cu defines the code for the functions
// specified in triple_double_gpufun.h

#include "double_double_gpufun.h"
#include "triple_double_gpufun.h"

/************************* renormalizations ***************************/

__device__ void tdg_fast_renorm
 ( double x0, double x1, double x2, double x3,
   double *r0, double *r1, double *r2 )
{
   double f0,f1,f2,f3,pr;
   int ptr;

   pr = ddg_quick_two_sum(x2,x3,&f3);
   pr = ddg_quick_two_sum(x1,pr,&f2);
   f0 = ddg_quick_two_sum(x0,pr,&f1);
   if(f1 == 0.0)
   {
      pr = f0;
      ptr = 0;
      *r0 = ddg_quick_two_sum(pr,f2,&pr);
   }
   else
   {
      *r0 = f0;
      pr = f1;
      ptr = 1;
      *r1 = ddg_quick_two_sum(pr,f2,&pr);
   }
   if(pr == 0.0)
   {
      if(ptr == 0)
         pr = *r0;
      else
         pr = *r1;
   }
   else
   {
      ptr = ptr + 1;
   }
   if(ptr == 0)
      *r0 = ddg_quick_two_sum(pr,f3,&pr);
   else if(ptr == 1)
      *r1 = ddg_quick_two_sum(pr,f3,&pr);
   else
      *r2 = ddg_quick_two_sum(pr,f3,&pr);

   if(pr == 0.0) 
   {
      if(ptr == 0)
         pr = *r0;
      else if(ptr == 1)
         pr = *r1;
      else
         pr = *r2;
    }
    else
    {
       ptr = ptr + 1;
    }
    if((ptr < 3) && (pr != 0.0))
    {
       if(ptr == 0)
          *r0 = pr;
       else if(ptr == 1)
          *r1 = pr;
       else
          *r2 = pr;
       ptr = ptr + 1;
    }
    if(ptr < 1)
    {
       *r0 = 0.0; *r1 = 0.0; *r2 = 0.0;
    }
    else if(ptr < 2)
    {
       *r1 = 0.0; *r2 = 0.0;
    }
    else if(ptr < 3)
    {
       *r2 = 0.0;
    }
}

__device__ void tdg_renorm_add1
 ( double x0, double x1, double x2, double y,
   double *r0, double *r1, double *r2 )
{
   double f0,f1,f2,f3,pr;
   int ptr;

   pr = ddg_two_sum(x2,y,&f3);
   pr = ddg_two_sum(x1,pr,&f2);
   f0 = ddg_two_sum(x0,pr,&f1);
   if(f1 == 0.0)
   {
      pr = f0;
      ptr = 0;
      *r0 = ddg_quick_two_sum(pr,f2,&pr);
   }
   else
   {
      *r0 = f0;
      pr = f1;
      ptr = 1;
      *r1 = ddg_quick_two_sum(pr,f2,&pr);
   }
   if(pr == 0.0)
   {
      if(ptr == 0)
         pr = *r0;
      else
         pr = *r1;
   }
   else
   {
      ptr = ptr + 1;
   }
   if(ptr == 0)
      *r0 = ddg_quick_two_sum(pr,f3,&pr);
   else if(ptr == 1)
      *r1 = ddg_quick_two_sum(pr,f3,&pr);
   else
      *r2 = ddg_quick_two_sum(pr,f3,&pr);

   if(pr == 0.0) 
   {
      if(ptr == 0)
         pr = *r0;
      else if(ptr == 1)
         pr = *r1;
      else
         pr = *r2;
    }
    else
    {
       ptr = ptr + 1;
    }
    if((ptr < 3) && (pr != 0.0))
    {
       if(ptr == 0)
          *r0 = pr;
       else if(ptr == 1)
          *r1 = pr;
       else
          *r2 = pr;
       ptr = ptr + 1;
    }
    if(ptr < 1)
    {
       *r0 = 0.0; *r1 = 0.0; *r2 = 0.0;
    }
    else if(ptr < 2)
    {
       *r1 = 0.0; *r2 = 0.0;
    }
    else if(ptr < 3)
    {
       *r2 = 0.0;
    }
}

/*************************** copy and abs ***************************/

__device__ void tdg_copy
 ( double a_hi, double a_mi, double a_lo,
   double *b_hi, double *b_mi, double *b_lo )
{
   *b_hi = a_hi;
   *b_mi = a_mi;
   *b_lo = a_lo;
}

__device__ void tdg_abs
 ( double a_hi, double a_mi, double a_lo,
   double *b_hi, double *b_mi, double *b_lo )
{
   if(a_hi < 0.0)
   {
      *b_hi = -a_hi;
      *b_mi = -a_mi;
      *b_lo = -a_lo;
   }
   else
   {
      *b_hi = a_hi;
      *b_mi = a_mi;
      *b_lo = a_lo;
   }
}

/************************** additions ********************************/

__device__ void tdg_add
 ( double a_hi, double a_mi, double a_lo,
   double b_hi, double b_mi, double b_lo,
   double *c_hi, double *c_mi, double *c_lo )
{
   double f0,f1,f2,f3,e;

   f2 = ddg_two_sum(a_lo,b_lo,&f3);
   f1 = ddg_two_sum(a_mi,b_mi,&e);
   f2 = ddg_two_sum(f2,e,&e);
   f3 += e;
   f0 = ddg_two_sum(a_hi,b_hi,&e);
   f1 = ddg_two_sum(f1,e,&e);
   f2 = ddg_two_sum(f2,e,&e);
   f3 += e;

   tdg_fast_renorm(f0,f1,f2,f3,c_hi,c_mi,c_lo);
}

__device__ void tdg_inc
 ( double *a_hi, double *a_mi, double *a_lo,
   double b_hi, double b_mi, double b_lo )
{
   double f0,f1,f2,f3,e;

   f2 = ddg_two_sum(*a_lo,b_lo,&f3);
   f1 = ddg_two_sum(*a_mi,b_mi,&e);
   f2 = ddg_two_sum(f2,e,&e);
   f3 += e;
   f0 = ddg_two_sum(*a_hi,b_hi,&e);
   f1 = ddg_two_sum(f1,e,&e);
   f2 = ddg_two_sum(f2,e,&e);
   f3 += e;

   tdg_fast_renorm(f0,f1,f2,f3,a_hi,a_mi,a_lo);
}

__device__ void tdg_inc_d
 ( double *a_hi, double *a_mi, double *a_lo, double b )
{
   tdg_renorm_add1(*a_hi,*a_mi,*a_lo,b,a_hi,a_mi,a_lo);
}

__device__ void tdg_minus ( double *a_hi, double *a_mi, double *a_lo )
{
   *a_hi = -(*a_hi);
   *a_mi = -(*a_mi);
   *a_lo = -(*a_lo);
}

__device__ void tdg_sub
 ( double a_hi, double a_mi, double a_lo,
   double b_hi, double b_mi, double b_lo,
   double *c_hi, double *c_mi, double *c_lo )
{
   tdg_copy(b_hi,b_mi,b_lo,c_hi,c_mi,c_lo);
   tdg_minus(c_hi,c_mi,c_lo);
   tdg_inc(c_hi,c_mi,c_lo,a_hi,a_mi,a_lo);
}

/***************** multiplications and division ********************/

__device__ void tdg_mul_pwr2
 ( double a_hi, double a_mi, double a_lo, double b,
   double *c_hi, double *c_mi, double *c_lo )
{
   *c_hi = a_hi*b;
   *c_mi = a_mi*b;
   *c_lo = a_lo*b;
}

__device__ void tdg_mul
 ( double a_hi, double a_mi, double a_lo,
   double b_hi, double b_mi, double b_lo,
   double *c_hi, double *c_mi, double *c_lo )
{
   double f0,f1,f2,f3,p,e;

   f3 = a_mi*b_lo + a_lo*b_mi;
   f2 = ddg_two_prod(a_hi,b_lo,&e);
   f3 += e;
   p = ddg_two_prod(a_mi,b_mi,&e);
   f3 += e;
   f2 = ddg_two_sum(f2,p,&e);
   f3 += e;
   p = ddg_two_prod(a_lo,b_hi,&e);
   f3 += e;
   f2 = ddg_two_sum(f2,p,&e);
   f3 += e;
   f1 = ddg_two_prod(a_hi,b_mi,&e);
   f2 = ddg_two_sum(f2,e,&e);
   f3 += e;
   p = ddg_two_prod(a_mi,b_hi,&e);
   f2 = ddg_two_sum(f2,e,&e);
   f3 += e;
   f1 = ddg_two_sum(f1,p,&e);
   f2 = ddg_two_sum(f2,e,&e);
   f3 += e;
   f0 = ddg_two_prod(a_hi,b_hi,&e);
   f1 = ddg_two_sum(f1,e,&e);
   f2 = ddg_two_sum(f2,e,&e);
   f3 += e;

   tdg_fast_renorm(f0,f1,f2,f3,c_hi,c_mi,c_lo);
}

__device__ void tdg_sqr
 ( double a_hi, double a_mi, double a_lo,
   double *c_hi, double *c_mi, double *c_lo )
{
   double f0,f1,f2,f3,p,e;

   f3 = 2.0*a_mi*a_lo;
   f2 = ddg_two_prod(a_hi,a_lo,&e);
   f3 += e;
   p = ddg_two_prod(a_mi,a_mi,&e);
   f3 += e;
   f2 = ddg_two_sum(f2,p,&e);
   f3 += e;
   p = ddg_two_prod(a_lo,a_hi,&e);
   f3 += e;
   f2 = ddg_two_sum(f2,p,&e);
   f3 += e;
   f1 = ddg_two_prod(a_hi,a_mi,&e);
   f2 = ddg_two_sum(f2,e,&e);
   f3 += e;
   p = ddg_two_prod(a_mi,a_hi,&e);
   f2 = ddg_two_sum(f2,e,&e);
   f3 += e;
   f1 = ddg_two_sum(f1,p,&e);
   f2 = ddg_two_sum(f2,e,&e);
   f3 += e;
   f0 = ddg_two_prod(a_hi,a_hi,&e);
   f1 = ddg_two_sum(f1,e,&e);
   f2 = ddg_two_sum(f2,e,&e);
   f3 += e;

   tdg_fast_renorm(f0,f1,f2,f3,c_hi,c_mi,c_lo);
}

__device__ void tdg_mul_td_d
 ( double a_hi, double a_mi, double a_lo, double b,
   double *c_hi, double *c_mi, double *c_lo )
{
   double f0,f1,f2,f3,e;

   f3 = 0.0;
   f2 = ddg_two_prod(a_lo,b,&e);
   f3 += e;
   f1 = ddg_two_prod(a_mi,b,&e);
   f2 = ddg_two_sum(f2,e,&e);
   f3 += e;
   f0 = ddg_two_prod(a_hi,b,&e);
   f1 = ddg_two_sum(f1,e,&e);
   f2 = ddg_two_sum(f2,e,&e);
   f3 += e;

   tdg_fast_renorm(f0,f1,f2,f3,c_hi,c_mi,c_lo);
}

__device__ void tdg_div
 ( double a_hi, double a_mi, double a_lo,
   double b_hi, double b_mi, double b_lo,
   double *c_hi, double *c_mi, double *c_lo )
{
   double acc_hi,acc_mi,acc_lo;
   double q0,q1,q2,q3;

   q0 = a_hi/b_hi;
   tdg_mul_td_d(b_hi,b_mi,b_lo,q0,&acc_hi,&acc_mi,&acc_lo);
   tdg_sub(a_hi,a_mi,a_lo,acc_hi,acc_mi,acc_lo,c_hi,c_mi,c_lo);

   q1 = *c_hi/b_hi;
   tdg_mul_td_d(b_hi,b_mi,b_lo,q1,&acc_hi,&acc_mi,&acc_lo);
   tdg_sub(*c_hi,*c_mi,*c_lo,acc_hi,acc_mi,acc_lo,c_hi,c_mi,c_lo);

   q2 = *c_hi/b_hi;
   tdg_mul_td_d(b_hi,b_mi,b_lo,q2,&acc_hi,&acc_mi,&acc_lo);
   tdg_sub(*c_hi,*c_mi,*c_lo,acc_hi,acc_mi,acc_lo,c_hi,c_mi,c_lo);

   q3 = *c_hi/b_hi;

   tdg_fast_renorm(q0,q1,q2,q3,c_hi,c_mi,c_lo);
}

/***************************** square root *****************************/

__device__ void tdg_sqrt
 ( double a_hi, double a_mi, double a_lo,
   double *b_hi, double *b_mi, double *b_lo )
{
   double z_hi,z_mi,z_lo;

   ddg_sqrt(a_hi,a_mi,b_hi,b_mi);
   tdg_sqr(*b_hi,*b_mi,0.0,&z_hi,&z_mi,&z_lo);
   tdg_inc(&z_hi,&z_mi,&z_lo,a_hi,a_mi,a_lo);
   tdg_div(z_hi,z_mi,z_lo,*b_hi,*b_mi,0.0,&z_hi,&z_mi,&z_lo);
   tdg_mul_pwr2(z_hi,z_mi,z_lo,0.5,b_hi,b_mi,b_lo);
}
