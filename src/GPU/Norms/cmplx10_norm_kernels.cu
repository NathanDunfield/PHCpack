#include "hip/hip_runtime.h"
// Defines code of the functions in cmplx10_norm_kernels.h,
// to compute the 2-norm and normalize a complex vector,
// in deca double precision,
// for vectors of small, medium, and large size.

#include "double_double_gpufun.cu"
#include "quad_double_gpufun.cu"
#include "octo_double_gpufun.cu"
#include "deca_double_gpufun.cu"
#include "cmplx10_norm_kernels.h"

__global__ void small_normalize_vector
 ( double *vrertb, double *vrerix, double *vrermi, double *vrerrg,
   double *vrerpk, double *vreltb, double *vrelix, double *vrelmi,
   double *vrelrg, double *vrelpk,
   double *vimrtb, double *vimrix, double *vimrmi, double *vimrrg,
   double *vimrpk, double *vimltb, double *vimlix, double *vimlmi,
   double *vimlrg, double *vimlpk, int dim, int dimLog2,
   double *normrtb, double *normrix, double *normrmi, double *normrrg,
   double *normrpk, double *normltb, double *normlix, double *normlmi,
   double *normlrg, double *normlpk )
{
   int j = threadIdx.x;

   __shared__ double shvrertb[da_shmemsize];
   __shared__ double shvrerix[da_shmemsize];
   __shared__ double shvrermi[da_shmemsize];
   __shared__ double shvrerrg[da_shmemsize];
   __shared__ double shvrerpk[da_shmemsize];
   __shared__ double shvreltb[da_shmemsize];
   __shared__ double shvrelix[da_shmemsize];
   __shared__ double shvrelmi[da_shmemsize];
   __shared__ double shvrelrg[da_shmemsize];
   __shared__ double shvrelpk[da_shmemsize];
   __shared__ double shvimrtb[da_shmemsize];
   __shared__ double shvimrix[da_shmemsize];
   __shared__ double shvimrmi[da_shmemsize];
   __shared__ double shvimrrg[da_shmemsize];
   __shared__ double shvimrpk[da_shmemsize];
   __shared__ double shvimltb[da_shmemsize];
   __shared__ double shvimlix[da_shmemsize];
   __shared__ double shvimlmi[da_shmemsize];
   __shared__ double shvimlrg[da_shmemsize];
   __shared__ double shvimlpk[da_shmemsize];
   __shared__ double prdrtb[da_shmemsize];
   __shared__ double prdrix[da_shmemsize];
   __shared__ double prdrmi[da_shmemsize];
   __shared__ double prdrrg[da_shmemsize];
   __shared__ double prdrpk[da_shmemsize];
   __shared__ double prdltb[da_shmemsize];
   __shared__ double prdlix[da_shmemsize];
   __shared__ double prdlmi[da_shmemsize];
   __shared__ double prdlrg[da_shmemsize];
   __shared__ double prdlpk[da_shmemsize];
   __shared__ double sumrtb[da_shmemsize];
   __shared__ double sumrix[da_shmemsize];
   __shared__ double sumrmi[da_shmemsize];
   __shared__ double sumrrg[da_shmemsize];
   __shared__ double sumrpk[da_shmemsize];
   __shared__ double sumltb[da_shmemsize];
   __shared__ double sumlix[da_shmemsize];
   __shared__ double sumlmi[da_shmemsize];
   __shared__ double sumlrg[da_shmemsize];
   __shared__ double sumlpk[da_shmemsize];

   shvrertb[j] = vrertb[j]; // reading real parts into shared memory
   shvrerix[j] = vrerix[j];
   shvrermi[j] = vrermi[j];
   shvrerrg[j] = vrerrg[j];
   shvrerpk[j] = vrerpk[j];
   shvreltb[j] = vreltb[j];
   shvrelix[j] = vrelix[j];
   shvrelmi[j] = vrelmi[j];
   shvrelrg[j] = vrelrg[j];
   shvrelpk[j] = vrelpk[j];
   shvimrtb[j] = vimrtb[j]; // reading imaginary parts into shared memory
   shvimrix[j] = vimrix[j];
   shvimrmi[j] = vimrmi[j];
   shvimrrg[j] = vimrrg[j];
   shvimrpk[j] = vimrpk[j];
   shvimltb[j] = vimltb[j];
   shvimlix[j] = vimlix[j];
   shvimlmi[j] = vimlmi[j];
   shvimlrg[j] = vimlrg[j];
   shvimlpk[j] = vimlpk[j];

   dag_sqr(shvrertb[j],shvrerix[j],shvrermi[j],shvrerrg[j],shvrerpk[j],
           shvreltb[j],shvrelix[j],shvrelmi[j],shvrelrg[j],shvrelpk[j],
            &sumrtb[j], &sumrix[j], &sumrmi[j], &sumrrg[j], &sumrpk[j],
            &sumltb[j], &sumlix[j], &sumlmi[j], &sumlrg[j], &sumlpk[j]);
   dag_sqr(shvimrtb[j],shvimrix[j],shvimrmi[j],shvimrrg[j],shvimrpk[j],
           shvimltb[j],shvimlix[j],shvimlmi[j],shvimlrg[j],shvimlpk[j],
            &prdrtb[j], &prdrix[j], &prdrmi[j], &prdrrg[j], &prdrpk[j],
            &prdltb[j], &prdlix[j], &prdlmi[j], &prdlrg[j], &prdlpk[j]);
   dag_inc(&sumrtb[j],&sumrix[j],&sumrmi[j],&sumrrg[j],&sumrpk[j],
           &sumltb[j],&sumlix[j],&sumlmi[j],&sumlrg[j],&sumlpk[j],
            prdrtb[j], prdrix[j], prdrmi[j], prdrrg[j], prdrpk[j],
            prdltb[j], prdlix[j], prdlmi[j], prdlrg[j], prdlpk[j]);

   __syncthreads();
   int powTwo = 1;                          // sum reduction
   for(int k=0; k < dimLog2; k++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < dim)
            dag_inc(&sumrtb[j],&sumrix[j],&sumrmi[j],&sumrrg[j],&sumrpk[j],
                    &sumltb[j],&sumlix[j],&sumlmi[j],&sumlrg[j],&sumlpk[j],
                     sumrtb[j+powTwo],sumrix[j+powTwo],sumrmi[j+powTwo],
                     sumrrg[j+powTwo],sumrpk[j+powTwo],
                     sumltb[j+powTwo],sumlix[j+powTwo],sumlmi[j+powTwo],
                     sumlrg[j+powTwo],sumlpk[j+powTwo]);
      powTwo = powTwo*2;
      __syncthreads();
   }
   // thread 0 computes the sqrt of the inner product, others wait
   if(j == 0)
   {
      dag_sqrt( sumrtb[0], sumrix[0], sumrmi[0], sumrrg[0], sumrpk[0],
                sumltb[0], sumlix[0], sumlmi[0], sumlrg[0], sumlpk[0],
               &sumrtb[0],&sumrix[0],&sumrmi[0],&sumrrg[0],&sumrpk[0],
               &sumltb[0],&sumlix[0],&sumlmi[0],&sumlrg[0],&sumlpk[0]); 
      *normrtb = sumrtb[0];
      *normrix = sumrix[0];
      *normrmi = sumrmi[0];
      *normrrg = sumrrg[0];
      *normrpk = sumrpk[0];
      *normltb = sumltb[0];
      *normlix = sumlix[0];
      *normlmi = sumlmi[0];
      *normlrg = sumlrg[0];
      *normlpk = sumlpk[0];
   }
   __syncthreads();
   dag_div(shvrertb[j],shvrerix[j],shvrermi[j],shvrerrg[j],shvrerpk[j],
           shvreltb[j],shvrelix[j],shvrelmi[j],shvrelrg[j],shvrelpk[j],
             sumrtb[0],  sumrix[0],  sumrmi[0],  sumrrg[0],  sumrpk[0],
             sumltb[0],  sumlix[0],  sumlmi[0],  sumlrg[0],  sumlpk[0],
            &vrertb[j], &vrerix[j], &vrermi[j], &vrerrg[j], &vrerpk[j],
            &vreltb[j], &vrelix[j], &vrelmi[j], &vrelrg[j], &vrelpk[j]);
   dag_div(shvimrtb[j],shvimrix[j],shvimrmi[j],shvimrrg[j],shvimrpk[j],
           shvimltb[j],shvimlix[j],shvimlmi[j],shvimlrg[j],shvimlpk[j],
             sumrtb[0],  sumrix[0],  sumrmi[0],  sumrrg[0],  sumrpk[0],
             sumltb[0],  sumlix[0],  sumlmi[0],  sumlrg[0],  sumlpk[0],
            &vimrtb[j], &vimrix[j], &vimrmi[j], &vimrrg[j], &vimrpk[j],
            &vimltb[j], &vimlix[j], &vimlmi[j], &vimlrg[j], &vimlpk[j]);
}

__global__ void medium_normalize_vector
 ( double *vrertb, double *vrerix, double *vrermi, double *vrerrg,
   double *vrerpk, double *vreltb, double *vrelix, double *vrelmi,
   double *vrelrg, double *vrelpk,
   double *vimrtb, double *vimrix, double *vimrmi, double *vimrrg,
   double *vimrpk, double *vimltb, double *vimlix, double *vimlmi,
   double *vimlrg, double *vimlpk,
   int dim, int rnd, int rndLog2, int BS, int BSLog2,
   double *normrtb, double *normrix, double *normrmi, double *normrrg,
   double *normrpk, double *normltb, double *normlix, double *normlmi,
   double *normlrg, double *normlpk )
{
   int j = threadIdx.x;
   int powTwo;
   int vBSind = 0;

   __shared__ double shvrertb[da_shmemsize];
   __shared__ double shvrerix[da_shmemsize];
   __shared__ double shvrermi[da_shmemsize];
   __shared__ double shvrerrg[da_shmemsize];
   __shared__ double shvrerpk[da_shmemsize];
   __shared__ double shvreltb[da_shmemsize];
   __shared__ double shvrelix[da_shmemsize];
   __shared__ double shvrelmi[da_shmemsize];
   __shared__ double shvrelrg[da_shmemsize];
   __shared__ double shvrelpk[da_shmemsize];
   __shared__ double shvimrtb[da_shmemsize];
   __shared__ double shvimrix[da_shmemsize];
   __shared__ double shvimrmi[da_shmemsize];
   __shared__ double shvimrrg[da_shmemsize];
   __shared__ double shvimrpk[da_shmemsize];
   __shared__ double shvimltb[da_shmemsize];
   __shared__ double shvimlix[da_shmemsize];
   __shared__ double shvimlmi[da_shmemsize];
   __shared__ double shvimlrg[da_shmemsize];
   __shared__ double shvimlpk[da_shmemsize];
   __shared__ double prdrtb[da_shmemsize];
   __shared__ double prdrix[da_shmemsize];
   __shared__ double prdrmi[da_shmemsize];
   __shared__ double prdrrg[da_shmemsize];
   __shared__ double prdrpk[da_shmemsize];
   __shared__ double prdltb[da_shmemsize];
   __shared__ double prdlix[da_shmemsize];
   __shared__ double prdlmi[da_shmemsize];
   __shared__ double prdlrg[da_shmemsize];
   __shared__ double prdlpk[da_shmemsize];
   __shared__ double accrtb[da_shmemsize];
   __shared__ double accrix[da_shmemsize];
   __shared__ double accrmi[da_shmemsize];
   __shared__ double accrrg[da_shmemsize];
   __shared__ double accrpk[da_shmemsize];
   __shared__ double accltb[da_shmemsize];
   __shared__ double acclix[da_shmemsize];
   __shared__ double acclmi[da_shmemsize];
   __shared__ double acclrg[da_shmemsize];
   __shared__ double acclpk[da_shmemsize];
   __shared__ double sumsrtb[maxrounds];
   __shared__ double sumsrix[maxrounds];
   __shared__ double sumsrmi[maxrounds];
   __shared__ double sumsrrg[maxrounds];
   __shared__ double sumsrpk[maxrounds];
   __shared__ double sumsltb[maxrounds];
   __shared__ double sumslix[maxrounds];
   __shared__ double sumslmi[maxrounds];
   __shared__ double sumslrg[maxrounds];
   __shared__ double sumslpk[maxrounds];

   for(int i=0; i<rnd; i++)
   {
      if(vBSind+j >= dim)       // at last round some threads are excluded
      {
         accrtb[j] = 0.0;
         accrix[j] = 0.0;
         accrmi[j] = 0.0;
         accrrg[j] = 0.0;
         accrpk[j] = 0.0;
         accltb[j] = 0.0;
         acclix[j] = 0.0;
         acclmi[j] = 0.0;
         acclrg[j] = 0.0;
         acclpk[j] = 0.0;
      }
      else
      {
         shvrertb[j] = vrertb[vBSind+j];  // reading into shared memory
         shvrerix[j] = vrerix[vBSind+j];
         shvrermi[j] = vrermi[vBSind+j];
         shvrerrg[j] = vrerrg[vBSind+j];
         shvrerpk[j] = vrerpk[vBSind+j];
         shvreltb[j] = vreltb[vBSind+j];
         shvrelix[j] = vrelix[vBSind+j];
         shvrelmi[j] = vrelmi[vBSind+j];
         shvrelrg[j] = vrelrg[vBSind+j];
         shvrelpk[j] = vrelpk[vBSind+j];
         shvimrtb[j] = vimrtb[vBSind+j]; 
         shvimrix[j] = vimrix[vBSind+j]; 
         shvimrmi[j] = vimrmi[vBSind+j]; 
         shvimrrg[j] = vimrrg[vBSind+j]; 
         shvimrpk[j] = vimrpk[vBSind+j]; 
         shvimltb[j] = vimltb[vBSind+j]; 
         shvimlix[j] = vimlix[vBSind+j]; 
         shvimlmi[j] = vimlmi[vBSind+j]; 
         shvimlrg[j] = vimlrg[vBSind+j]; 
         shvimlpk[j] = vimlpk[vBSind+j]; 

         dag_sqr(shvrertb[j],shvrerix[j],shvrermi[j],shvrerrg[j],shvrerpk[j],
                 shvreltb[j],shvrelix[j],shvrelmi[j],shvrelrg[j],shvrelpk[j],
                  &accrtb[j], &accrix[j], &accrmi[j], &accrrg[j], &accrpk[j],
                  &accltb[j], &acclix[j], &acclmi[j], &acclrg[j], &acclpk[j]);
         dag_sqr(shvimrtb[j],shvimrix[j],shvimrmi[j],shvimrrg[j],shvimrpk[j],
                 shvimltb[j],shvimlix[j],shvimlmi[j],shvimlrg[j],shvimlpk[j],
                  &prdrtb[j], &prdrix[j], &prdrmi[j], &prdrrg[j], &prdrpk[j],
                  &prdltb[j], &prdlix[j], &prdlmi[j], &prdlrg[j], &prdlpk[j]);
         dag_inc(&accrtb[j],&accrix[j],&accrmi[j],&accrrg[j],&accrpk[j],
                 &accltb[j],&acclix[j],&acclmi[j],&acclrg[j],&acclpk[j],
                  prdrtb[j], prdrix[j], prdrmi[j], prdrrg[j], prdrpk[j],
                  prdltb[j], prdlix[j], prdlmi[j], prdlrg[j], prdlpk[j]);
      }
      __syncthreads();
      powTwo = 1;                          // sum reduction
      for(int k=0; k < BSLog2; k++)
      {
         if((j%(powTwo*2)) == 0)
            if(j+powTwo < BS)
               dag_inc(&accrtb[j],&accrix[j],&accrmi[j],&accrrg[j],&accrpk[j],
                       &accltb[j],&acclix[j],&acclmi[j],&acclrg[j],&acclpk[j],
                        accrtb[j+powTwo],accrix[j+powTwo],accrmi[j+powTwo],
                        accrrg[j+powTwo],accrpk[j+powTwo],
                        accltb[j+powTwo],acclix[j+powTwo],acclmi[j+powTwo],
                        acclrg[j+powTwo],acclpk[j+powTwo]);
         powTwo = powTwo*2;
         __syncthreads();
      }
      // thread 0 copies the sum of this round in sums[i], the others wait
      if(j == 0)
      {
         sumsrtb[i] = accrtb[0]; 
         sumsrix[i] = accrix[0]; 
         sumsrmi[i] = accrmi[0]; 
         sumsrrg[i] = accrrg[0]; 
         sumsrpk[i] = accrpk[0]; 
         sumsltb[i] = accltb[0]; 
         sumslix[i] = acclix[0]; 
         sumslmi[i] = acclmi[0]; 
         sumslrg[i] = acclrg[0]; 
         sumslpk[i] = acclpk[0]; 
      }
      __syncthreads();
      vBSind = vBSind + BS;
   }
   powTwo = 1;                          // sum reduction
   for(int k=0; k < rndLog2; k++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < rnd)
            dag_inc
               (&sumsrtb[j],&sumsrix[j],&sumsrmi[j],&sumsrrg[j],&sumsrpk[j],
                &sumsltb[j],&sumslix[j],&sumslmi[j],&sumslrg[j],&sumslpk[j],
                sumsrtb[j+powTwo],sumsrix[j+powTwo],sumsrmi[j+powTwo],
                sumsrrg[j+powTwo],sumsrpk[j+powTwo],
                sumsltb[j+powTwo],sumslix[j+powTwo],sumslmi[j+powTwo],
                sumslrg[j+powTwo],sumslpk[j+powTwo]);
      powTwo = powTwo*2;
      __syncthreads();
   }
   if(j == 0)
   {
      dag_sqrt( sumsrtb[0], sumsrix[0], sumsrmi[0], sumsrrg[0], sumsrpk[0],
                sumsltb[0], sumslix[0], sumslmi[0], sumslrg[0], sumslpk[0],
               &sumsrtb[0],&sumsrix[0],&sumsrmi[0],&sumsrrg[0],&sumsrpk[0],
               &sumsltb[0],&sumslix[0],&sumslmi[0],&sumslrg[0],&sumslpk[0]);
      *normrtb = sumsrtb[0];
      *normrix = sumsrix[0];
      *normrmi = sumsrmi[0];
      *normrrg = sumsrrg[0];
      *normrpk = sumsrpk[0];
      *normltb = sumsltb[0];
      *normlix = sumslix[0];
      *normlmi = sumslmi[0];
      *normlrg = sumslrg[0];
      *normlpk = sumslpk[0];
   }
   __syncthreads();
   vBSind = 0;
   for(int i=0; i<rnd; i++)
   {
      if(vBSind+j < dim)
      {
         shvrertb[j] = vrertb[vBSind+j];     // read into shared memory
         shvrerix[j] = vrerix[vBSind+j];
         shvrermi[j] = vrermi[vBSind+j];
         shvrerrg[j] = vrerrg[vBSind+j];
         shvrerpk[j] = vrerpk[vBSind+j];
         shvreltb[j] = vreltb[vBSind+j];
         shvrelix[j] = vrelix[vBSind+j];
         shvrelmi[j] = vrelmi[vBSind+j];
         shvrelrg[j] = vrelrg[vBSind+j];
         shvrelpk[j] = vrelpk[vBSind+j];
         shvimrtb[j] = vimrtb[vBSind+j];
         shvimrix[j] = vimrix[vBSind+j];
         shvimrmi[j] = vimrmi[vBSind+j];
         shvimrrg[j] = vimrrg[vBSind+j];
         shvimrpk[j] = vimlpk[vBSind+j];
         shvimltb[j] = vimltb[vBSind+j];
         shvimlix[j] = vimlix[vBSind+j];
         shvimlmi[j] = vimlmi[vBSind+j];
         shvimlrg[j] = vimlrg[vBSind+j];
         shvimlpk[j] = vimlpk[vBSind+j];
         // normalize vector
         dag_div(shvrertb[j],shvrerix[j],shvrermi[j],shvrerrg[j],shvrerpk[j],
                 shvreltb[j],shvrelix[j],shvrelmi[j],shvrelrg[j],shvrelpk[j],
                  sumsrtb[0], sumsrix[0], sumsrmi[0], sumsrrg[0], sumsrpk[0],
                  sumsltb[0], sumslix[0], sumslmi[0], sumslrg[0], sumslpk[0],
                  &vrertb[vBSind+j],&vrerix[vBSind+j],&vrermi[vBSind+j],
                  &vrerrg[vBSind+j],&vrerpk[vBSind+j],
                  &vreltb[vBSind+j],&vrelix[vBSind+j],&vrelmi[vBSind+j],
                  &vrelrg[vBSind+j],&vrelpk[vBSind+j]);
         dag_div(shvimrtb[j],shvimrix[j],shvimrmi[j],shvimrrg[j],shvimrpk[j],
                 shvimltb[j],shvimlix[j],shvimlmi[j],shvimlrg[j],shvimlpk[j],
                  sumsrtb[0], sumsrix[0], sumsrmi[0], sumsrrg[0], sumsrpk[0],
                  sumsltb[0], sumslix[0], sumslmi[0], sumslrg[0], sumslpk[0],
                  &vimrtb[vBSind+j],&vimrix[vBSind+j],&vimrmi[vBSind+j],
                  &vimrrg[vBSind+j],&vimrpk[vBSind+j],
                  &vimltb[vBSind+j],&vimlix[vBSind+j],&vimlmi[vBSind+j],
                  &vimlrg[vBSind+j],&vimlpk[vBSind+j]);
      }
      __syncthreads();
      vBSind = vBSind + BS;
   }
}

__global__ void large_sum_the_squares
 ( double *vrertb, double *vrerix, double *vrermi, double *vrerrg,
   double *vrerpk, double *vreltb, double *vrelix, double *vrelmi,
   double *vrelrg, double *vrelpk,
   double *vimrtb, double *vimrix, double *vimrmi, double *vimrrg,
   double *vimrpk, double *vimltb, double *vimlix, double *vimlmi,
   double *vimlrg, double *vimlpk, int dim,
   double *sumsrtb, double *sumsrix, double *sumsrmi, double *sumsrrg,
   double *sumsrpk, double *sumsltb, double *sumslix, double *sumslmi,
   double *sumslrg, double *sumslpk, int BS, int BSLog2 )
{
   const int i = blockIdx.x;
   const int j = threadIdx.x;
   const int k = i*BS + j;

   __shared__ double shvrertb[da_shmemsize];
   __shared__ double shvrerix[da_shmemsize];
   __shared__ double shvrermi[da_shmemsize];
   __shared__ double shvrerrg[da_shmemsize];
   __shared__ double shvrerpk[da_shmemsize];
   __shared__ double shvreltb[da_shmemsize];
   __shared__ double shvrelix[da_shmemsize];
   __shared__ double shvrelmi[da_shmemsize];
   __shared__ double shvrelrg[da_shmemsize];
   __shared__ double shvrelpk[da_shmemsize];
   __shared__ double shvimrtb[da_shmemsize];
   __shared__ double shvimrix[da_shmemsize];
   __shared__ double shvimrmi[da_shmemsize];
   __shared__ double shvimrrg[da_shmemsize];
   __shared__ double shvimrpk[da_shmemsize];
   __shared__ double shvimltb[da_shmemsize];
   __shared__ double shvimlix[da_shmemsize];
   __shared__ double shvimlmi[da_shmemsize];
   __shared__ double shvimlrg[da_shmemsize];
   __shared__ double shvimlpk[da_shmemsize];
   __shared__ double prdrtb[da_shmemsize];
   __shared__ double prdrix[da_shmemsize];
   __shared__ double prdrmi[da_shmemsize];
   __shared__ double prdrrg[da_shmemsize];
   __shared__ double prdrpk[da_shmemsize];
   __shared__ double prdltb[da_shmemsize];
   __shared__ double prdlix[da_shmemsize];
   __shared__ double prdlmi[da_shmemsize];
   __shared__ double prdlrg[da_shmemsize];
   __shared__ double prdlpk[da_shmemsize];
   __shared__ double accrtb[da_shmemsize];
   __shared__ double accrix[da_shmemsize];
   __shared__ double accrmi[da_shmemsize];
   __shared__ double accrrg[da_shmemsize];
   __shared__ double accrpk[da_shmemsize];
   __shared__ double accltb[da_shmemsize];
   __shared__ double acclix[da_shmemsize];
   __shared__ double acclmi[da_shmemsize];
   __shared__ double acclrg[da_shmemsize];
   __shared__ double acclpk[da_shmemsize];

   shvrertb[j] = vrertb[k];
   shvrerix[j] = vrerix[k];
   shvrermi[j] = vrermi[k];
   shvrerrg[j] = vrerrg[k];
   shvrerpk[j] = vrerpk[k];
   shvreltb[j] = vreltb[k];
   shvrelix[j] = vrelix[k];
   shvrelmi[j] = vrelmi[k];
   shvrelrg[j] = vrelrg[k];
   shvrelpk[j] = vrelpk[k];
   shvimrtb[j] = vimrtb[k];
   shvimrix[j] = vimrix[k];
   shvimrmi[j] = vimrmi[k];
   shvimrrg[j] = vimrrg[k];
   shvimrpk[j] = vimrpk[k];
   shvimltb[j] = vimltb[k];
   shvimlix[j] = vimlix[k];
   shvimlmi[j] = vimlmi[k];
   shvimlrg[j] = vimlrg[k];
   shvimlpk[j] = vimlpk[k];

   dag_sqr(shvrertb[j],shvrerix[j],shvrermi[j],shvrerrg[j],shvrerpk[j],
           shvreltb[j],shvrelix[j],shvrelmi[j],shvrelrg[j],shvrelpk[j],
            &accrtb[j], &accrix[j], &accrmi[j], &accrrg[j], &accrpk[j],
            &accltb[j], &acclix[j], &acclmi[j], &acclrg[j], &acclpk[j]);
   dag_sqr(shvimrtb[j],shvimrix[j],shvimrmi[j],shvimrrg[j],shvimrpk[j],
           shvimltb[j],shvimlix[j],shvimlmi[j],shvimlrg[j],shvimlpk[j],
            &prdrtb[j], &prdrix[j], &prdrmi[j], &prdrrg[j], &prdrpk[j],
            &prdltb[j], &prdlix[j], &prdlmi[j], &prdlrg[j], &prdlpk[j]);
   dag_inc(&accrtb[j],&accrix[j],&accrmi[j],&accrrg[j],&accrpk[j],
           &accltb[j],&acclix[j],&acclmi[j],&acclrg[j],&acclpk[j],
            prdrtb[j], prdrix[j], prdrmi[j], prdrrg[j], prdrpk[j],
            prdltb[j], prdlix[j], prdlmi[j], prdlrg[j], prdlpk[j]);

   __syncthreads();

   int powTwo = 1;                          // sum reduction
   for(int L=0; L < BSLog2; L++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < BS)
            dag_inc(&accrtb[j],&accrix[j],&accrmi[j],&accrrg[j],&accrpk[j],
                    &accltb[j],&acclix[j],&acclmi[j],&acclrg[j],&acclpk[j],
                     accrtb[j+powTwo],accrix[j+powTwo],accrmi[j+powTwo],
                     accrrg[j+powTwo],accrpk[j+powTwo],
                     accltb[j+powTwo],acclix[j+powTwo],acclmi[j+powTwo],
                     acclrg[j+powTwo],acclpk[j+powTwo]);
      powTwo = powTwo*2;

      __syncthreads();
   }
   if(j == 0)                               // thread 0 writes the sum
   {
      sumsrtb[i] = accrtb[0];
      sumsrix[i] = accrix[0];
      sumsrmi[i] = accrmi[0];
      sumsrrg[i] = accrrg[0];
      sumsrpk[i] = accrpk[0];
      sumsltb[i] = accltb[0];
      sumslix[i] = acclix[0];
      sumslmi[i] = acclmi[0];
      sumslrg[i] = acclrg[0];
      sumslpk[i] = acclpk[0];
   }
}

__global__ void large_normalize_vector
 ( double *vrertb, double *vrerix, double *vrermi, double *vrerrg,
   double *vrerpk, double *vreltb, double *vrelix, double *vrelmi,
   double *vrelrg, double *vrelpk,
   double *vimrtb, double *vimrix, double *vimrmi, double *vimrrg,
   double *vimrpk, double *vimltb, double *vimlix, double *vimlmi,
   double *vimlrg, double *vimlpk, int dim,
   double *sumsrtb, double *sumsrix, double *sumsrmi, double *sumsrrg,
   double *sumsrpk, double *sumsltb, double *sumslix, double *sumslmi,
   double *sumslrg, double *sumslpk,
   int nbsums, int nbsumsLog2, int BS,
   double *normrtb, double *normrix, double *normrmi, double *normrrg,
   double *normrpk, double *normltb, double *normlix, double *normlmi,
   double *normlrg, double *normlpk )
{
   const int i = blockIdx.x;
   const int j = threadIdx.x;
   const int k = i*BS + j;

   __shared__ double shvrertb[da_shmemsize];
   __shared__ double shvrerix[da_shmemsize];
   __shared__ double shvrermi[da_shmemsize];
   __shared__ double shvrerrg[da_shmemsize];
   __shared__ double shvrerpk[da_shmemsize];
   __shared__ double shvreltb[da_shmemsize];
   __shared__ double shvrelix[da_shmemsize];
   __shared__ double shvrelmi[da_shmemsize];
   __shared__ double shvrelrg[da_shmemsize];
   __shared__ double shvrelpk[da_shmemsize];
   __shared__ double shvimrtb[da_shmemsize];
   __shared__ double shvimrix[da_shmemsize];
   __shared__ double shvimrmi[da_shmemsize];
   __shared__ double shvimrrg[da_shmemsize];
   __shared__ double shvimrpk[da_shmemsize];
   __shared__ double shvimltb[da_shmemsize];
   __shared__ double shvimlix[da_shmemsize];
   __shared__ double shvimlmi[da_shmemsize];
   __shared__ double shvimlrg[da_shmemsize];
   __shared__ double shvimlpk[da_shmemsize];

   if(j < nbsums)
   {
      shvrertb[j] = sumsrtb[j];
      shvrerix[j] = sumsrix[j];
      shvrermi[j] = sumsrmi[j];
      shvrerrg[j] = sumsrrg[j];
      shvrerpk[j] = sumsrpk[j];
      shvreltb[j] = sumsltb[j];
      shvrelix[j] = sumslix[j];
      shvrelmi[j] = sumslmi[j];
      shvrelrg[j] = sumslrg[j];
      shvrelpk[j] = sumslpk[j];
   }

   __syncthreads();

   int powTwo = 1;                          // sum reduction
   for(int L=0; L < nbsumsLog2; L++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < nbsums)
            dag_inc
               (&shvrertb[j],&shvrerix[j],&shvrermi[j],
                &shvrerrg[j],&shvrerpk[j],
                &shvreltb[j],&shvrelix[j],&shvrelmi[j],
                &shvrelrg[j],&shvrelpk[j],
                shvrertb[j+powTwo],shvrerix[j+powTwo],shvrermi[j+powTwo],
                shvrerrg[j+powTwo],shvrerpk[j+powTwo],
                shvreltb[j+powTwo],shvrelix[j+powTwo],shvrelmi[j+powTwo],
                shvrelrg[j+powTwo],shvrelpk[j+powTwo]);
      powTwo = powTwo*2;

      __syncthreads();
   }
   __syncthreads();                      // every thread 0 of all blocks
   if(j == 0)                            // compute the 2-norm and assigns
   {                                     // to the output parameter
      dag_sqrt(shvrertb[0],shvrerix[0],shvrermi[0],shvrerrg[0],shvrerpk[0],
               shvreltb[0],shvrelix[0],shvrelmi[0],shvrelrg[0],shvrelpk[0],
                normrtb,    normrix,    normrmi,    normrrg,    normrpk,
                normltb,    normlix,    normlmi,    normlrg,    normlpk);
   }
   __syncthreads();                    

   if(k < dim)
   {
      shvrertb[j] = vrertb[k];
      shvrerix[j] = vrerix[k];
      shvrermi[j] = vrermi[k];
      shvrerrg[j] = vrerrg[k];
      shvrerpk[j] = vrerpk[k];
      shvreltb[j] = vreltb[k];
      shvrelix[j] = vrelix[k];
      shvrelmi[j] = vrelmi[k];
      shvrelrg[j] = vrelrg[k];
      shvrelpk[j] = vrelpk[k];
      shvimrtb[j] = vimrtb[k];
      shvimrix[j] = vimrix[k];
      shvimrmi[j] = vimrmi[k];
      shvimrrg[j] = vimrrg[k];
      shvimrpk[j] = vimrpk[k];
      shvimltb[j] = vimltb[k];
      shvimlix[j] = vimlix[k];
      shvimlmi[j] = vimlmi[k];
      shvimlrg[j] = vimlrg[k];
      shvimlpk[j] = vimlpk[k];

      dag_div
         ( shvrertb[j], shvrerix[j], shvrermi[j], shvrerrg[j], shvrerpk[j],
           shvreltb[j], shvrelix[j], shvrelmi[j], shvrelrg[j], shvrelpk[j],
           *normrtb,    *normrix,    *normrmi,    *normrrg,    *normrpk,
           *normltb,    *normlix,    *normlmi,    *normlrg,    *normlpk,
          &shvrertb[j],&shvrerix[j],&shvrermi[j],&shvrerrg[j],&shvrerpk[j],
          &shvreltb[j],&shvrelix[j],&shvrelmi[j],&shvrelrg[j],&shvrelpk[j]);
      dag_div
         ( shvimrtb[j], shvimrix[j], shvimrmi[j], shvimrrg[j], shvimrpk[j],
           shvimltb[j], shvimlix[j], shvimlmi[j], shvimlrg[j], shvimlpk[j],
           *normrtb,    *normrix,    *normrmi,    *normrrg,    *normrpk,
           *normltb,    *normlix,    *normlmi,    *normlrg,    *normlpk,
          &shvimrtb[j],&shvimrix[j],&shvimrmi[j],&shvimrrg[j],&shvimrpk[j],
          &shvimltb[j],&shvimlix[j],&shvimlmi[j],&shvimlrg[j],&shvimlpk[j]);

      vrertb[k] = shvrertb[j];
      vrerix[k] = shvrerix[j];
      vrermi[k] = shvrermi[j];
      vrerrg[k] = shvrerrg[j];
      vrerpk[k] = shvrerpk[j];
      vreltb[k] = shvreltb[j];
      vrelix[k] = shvrelix[j];
      vrelmi[k] = shvrelmi[j];
      vrelrg[k] = shvrelrg[j];
      vrelpk[k] = shvrelpk[j];
      vimrtb[k] = shvimrtb[j];
      vimrix[k] = shvimrix[j];
      vimrmi[k] = shvimrmi[j];
      vimrrg[k] = shvimrrg[j];
      vimrpk[k] = shvimrpk[j];
      vimltb[k] = shvimltb[j];
      vimlix[k] = shvimlix[j];
      vimlmi[k] = shvimlmi[j];
      vimlrg[k] = shvimlrg[j];
      vimlpk[k] = shvimlpk[j];
   }
}

void GPU_norm
 ( double *vrertb_h, double *vrerix_h, double *vrermi_h, double *vrerrg_h,
   double *vrerpk_h, double *vreltb_h, double *vrelix_h, double *vrelmi_h,
   double *vrelrg_h, double *vrelpk_h,
   double *vimrtb_h, double *vimrix_h, double *vimrmi_h, double *vimrrg_h,
   double *vimrpk_h, double *vimltb_h, double *vimlix_h, double *vimlmi_h,
   double *vimlrg_h, double *vimlpk_h, int dim, int freq, int BS,
   double *normrtb, double *normrix, double *normrmi, double *normrrg,
   double *normrpk, double *normltb, double *normlix, double *normlmi,
   double *normlrg, double *normlpk, int blocked )
{
   int BSLog2 = ceil(log2((double) BS)); // ceil for sum reduction

   double* vrertb_d;           // highest real parts on device
   double* vrerix_d;           // second highest real parts on device
   double* vrermi_d;           // third highest real parts on device
   double* vrerrg_d;           // fourth highest real parts on device
   double* vrerpk_d;           // fifth highest real parts on device
   double* vreltb_d;           // fifth lowest real parts on device
   double* vrelix_d;           // fourth lowest real parts on device
   double* vrelmi_d;           // third second lowest real parts on device
   double* vrelrg_d;           // second lowest real parts on device
   double* vrelpk_d;           // lowest real parts on device
   double* vimrtb_d;           // highest imaginary parts on device
   double* vimrix_d;           // second highest imaginary parts on device
   double* vimrmi_d;           // third highest imaginary parts on device
   double* vimrrg_d;           // fourth highest imaginary parts on device
   double* vimrpk_d;           // fifth highest imaginary parts on device
   double* vimltb_d;           // fifth lowest imaginary parts on device
   double* vimlix_d;           // fourth lowest imaginary parts on device
   double* vimlmi_d;           // third lowest imaginary parts on device
   double* vimlrg_d;           // second lowest imaginary parts on device
   double* vimlpk_d;           // lowest imaginary parts on device
   size_t size = dim*sizeof(double);
   hipMalloc((void**)&vrertb_d,size);
   hipMalloc((void**)&vrerix_d,size);
   hipMalloc((void**)&vrermi_d,size);
   hipMalloc((void**)&vrerrg_d,size);
   hipMalloc((void**)&vrerpk_d,size);
   hipMalloc((void**)&vreltb_d,size);
   hipMalloc((void**)&vrelix_d,size);
   hipMalloc((void**)&vrelmi_d,size);
   hipMalloc((void**)&vrelrg_d,size);
   hipMalloc((void**)&vrelpk_d,size);
   hipMalloc((void**)&vimrtb_d,size);
   hipMalloc((void**)&vimrix_d,size);
   hipMalloc((void**)&vimrmi_d,size);
   hipMalloc((void**)&vimrrg_d,size);
   hipMalloc((void**)&vimrpk_d,size);
   hipMalloc((void**)&vimltb_d,size);
   hipMalloc((void**)&vimlix_d,size);
   hipMalloc((void**)&vimlmi_d,size);
   hipMalloc((void**)&vimlrg_d,size);
   hipMalloc((void**)&vimlpk_d,size);
   hipMemcpy(vrertb_d,vrertb_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrerix_d,vrerix_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrermi_d,vrermi_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrerrg_d,vrerrg_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrerpk_d,vrerpk_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vreltb_d,vreltb_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrelix_d,vrelix_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrelmi_d,vrelmi_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrelrg_d,vrelrg_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrelpk_d,vrelpk_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimrtb_d,vimrtb_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimrix_d,vimrix_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimrmi_d,vimrmi_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimrrg_d,vimrrg_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimrpk_d,vimrpk_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimltb_d,vimltb_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimlix_d,vimlix_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimlmi_d,vimlmi_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimlrg_d,vimlrg_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimlpk_d,vimlpk_h,size,hipMemcpyHostToDevice);
   double* normrtb_d;
   double* normrix_d;
   double* normrmi_d;
   double* normrrg_d;
   double* normrpk_d;
   double* normltb_d;
   double* normlix_d;
   double* normlmi_d;
   double* normlrg_d;
   double* normlpk_d;
   hipMalloc((void**)&normrtb_d,sizeof(double));
   hipMalloc((void**)&normrix_d,sizeof(double));
   hipMalloc((void**)&normrmi_d,sizeof(double));
   hipMalloc((void**)&normrrg_d,sizeof(double));
   hipMalloc((void**)&normrpk_d,sizeof(double));
   hipMalloc((void**)&normltb_d,sizeof(double));
   hipMalloc((void**)&normlix_d,sizeof(double));
   hipMalloc((void**)&normlmi_d,sizeof(double));
   hipMalloc((void**)&normlrg_d,sizeof(double));
   hipMalloc((void**)&normlpk_d,sizeof(double));

   if(dim == BS)
   {
      for(int i=0; i<freq; i++)
         small_normalize_vector<<<1,BS>>>
            (vrertb_d,vrerix_d,vrermi_d,vrerrg_d,vrerpk_d,
             vreltb_d,vrelix_d,vrelmi_d,vrelrg_d,vrelpk_d,
             vimrtb_d,vimrix_d,vimrmi_d,vimrrg_d,vimrpk_d,
             vimltb_d,vimlix_d,vimlmi_d,vimlrg_d,vimlpk_d,
             dim,BSLog2,
             normrtb_d,normrix_d,normrmi_d,normrrg_d,normrpk_d,
             normltb_d,normlix_d,normlmi_d,normlrg_d,normlpk_d);
   }
   else if(blocked == 0)
   {
      int rf = ceil(((double) dim)/BS);
      int rfLog2 = ceil(log2((double) rf));
      for(int i=0; i<freq; i++)
         medium_normalize_vector<<<1,BS>>>
            (vrertb_d,vrerix_d,vrermi_d,vrerrg_d,vrerpk_d,
             vreltb_d,vrelix_d,vrelmi_d,vrelrg_d,vrelpk_d,
             vimrtb_d,vimrix_d,vimrmi_d,vimrrg_d,vimrpk_d,
             vimltb_d,vimlix_d,vimlmi_d,vimlrg_d,vimlpk_d,dim,
             rf,rfLog2,BS,BSLog2,
             normrtb_d,normrix_d,normrmi_d,normrrg_d,normrpk_d,
             normltb_d,normlix_d,normlmi_d,normlrg_d,normlpk_d);
   }
   else
   {
      const int nblocks = dim/BS;
      const int nblocksLog2 = ceil(log2((double) nblocks));
      double* sumsrtb_d; // highest parts of sums of squares for each block
      double* sumsrix_d; // 2nd highest parts of sums of squares
      double* sumsrmi_d; // middle parts of sums of squares
      double* sumsrrg_d; // 2nd lowest parts of sums of squares for each block
      double* sumsrpk_d; // lowest parts of sums of squares for each block
      double* sumsltb_d; // highest parts of sums of squares for each block
      double* sumslix_d; // 2nd highest parts of sums of squares
      double* sumslmi_d; // middle parts of sums of squares
      double* sumslrg_d; // 2nd lowest parts of sums of squares for each block
      double* sumslpk_d; // lowest parts of sums of squares for each block
      size_t sums_size = nblocks*sizeof(double);
      hipMalloc((void**)&sumsrtb_d,sums_size);
      hipMalloc((void**)&sumsrix_d,sums_size);
      hipMalloc((void**)&sumsrmi_d,sums_size);
      hipMalloc((void**)&sumsrrg_d,sums_size);
      hipMalloc((void**)&sumsrpk_d,sums_size);
      hipMalloc((void**)&sumsltb_d,sums_size);
      hipMalloc((void**)&sumslix_d,sums_size);
      hipMalloc((void**)&sumslmi_d,sums_size);
      hipMalloc((void**)&sumslrg_d,sums_size);
      hipMalloc((void**)&sumslpk_d,sums_size);
      for(int i=0; i<freq; i++)
      {
         large_sum_the_squares<<<nblocks,BS>>>
            (vrertb_d,vrerix_d,vrermi_d,vrerrg_d,vrerpk_d,
             vreltb_d,vrelix_d,vrelmi_d,vrelrg_d,vrelpk_d,
             vimrtb_d,vimrix_d,vimrmi_d,vimrrg_d,vimrpk_d,
             vimltb_d,vimlix_d,vimlmi_d,vimlrg_d,vimlpk_d,dim,
             sumsrtb_d,sumsrix_d,sumsrmi_d,sumsrrg_d,sumsrpk_d,
             sumsltb_d,sumslix_d,sumslmi_d,sumslrg_d,sumslpk_d,BS,BSLog2);
         large_normalize_vector<<<nblocks,BS>>>
            (vrertb_d,vrerix_d,vrermi_d,vrerrg_d,vrerpk_d,
             vreltb_d,vrelix_d,vrelmi_d,vrelrg_d,vrelpk_d,
             vimrtb_d,vimrix_d,vimrmi_d,vimrrg_d,vimrpk_d,
             vimltb_d,vimlix_d,vimlmi_d,vimlrg_d,vimlpk_d,dim,
             sumsrtb_d,sumsrix_d,sumsrmi_d,sumsrrg_d,sumsrpk_d,
             sumsltb_d,sumslix_d,sumslmi_d,sumslrg_d,sumslpk_d,
             nblocks,nblocksLog2,BS,
             normrtb_d,normrix_d,normrmi_d,normrrg_d,normrpk_d,
             normltb_d,normlix_d,normlmi_d,normlrg_d,normlpk_d);
      }
   }
   hipMemcpy(vrertb_h,vrertb_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrerix_h,vrerix_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrermi_h,vrermi_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrerrg_h,vrerrg_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrerpk_h,vrerpk_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vreltb_h,vreltb_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrelix_h,vrelix_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrelmi_h,vrelmi_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrelrg_h,vrelrg_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrelpk_h,vrelpk_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimrtb_h,vimrtb_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimrix_h,vimrix_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimrmi_h,vimrmi_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimrrg_h,vimrrg_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimrpk_h,vimrpk_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimltb_h,vimltb_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimlix_h,vimlix_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimlmi_h,vimlmi_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimlrg_h,vimlrg_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimlpk_h,vimlpk_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(normrtb,normrtb_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normrix,normrix_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normrmi,normrmi_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normrrg,normrrg_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normrpk,normrpk_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normltb,normltb_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normlix,normlix_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normlmi,normlmi_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normlrg,normlrg_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normlpk,normlpk_d,sizeof(double),hipMemcpyDeviceToHost);
}
