#include "hip/hip_runtime.h"
// Defines code of the functions in cmplx10_norm_kernels.h,
// to compute the 2-norm and normalize a complex vector,
// in deca double precision,
// for vectors of small, medium, and large size.

#include "double_double_gpufun.cu"
#include "quad_double_gpufun.cu"
#include "octo_double_gpufun.cu"
#include "deca_double_gpufun.cu"
#include "cmplx10_norm_kernels.h"

__global__ void small_normalize_vector
 ( double *vrertb, double *vrerix, double *vrermi, double *vrerrg,
   double *vrerpk, double *vreltb, double *vrelix, double *vrelmi,
   double *vrelrg, double *vrelpk,
   double *vimrtb, double *vimrix, double *vimrmi, double *vimrrg,
   double *vimrpk, double *vimltb, double *vimlix, double *vimlmi,
   double *vimlrg, double *vimlpk, int dim, int dimLog2,
   double *normrtb, double *normrix, double *normrmi, double *normrrg,
   double *normrpk, double *normltb, double *normlix, double *normlmi,
   double *normlrg, double *normlpk )
{
   int j = threadIdx.x;

   __shared__ double shvrertb[da_shmemsize];
   __shared__ double shvrerix[da_shmemsize];
   __shared__ double shvrermi[da_shmemsize];
   __shared__ double shvrerrg[da_shmemsize];
   __shared__ double shvrerpk[da_shmemsize];
   __shared__ double shvreltb[da_shmemsize];
   __shared__ double shvrelix[da_shmemsize];
   __shared__ double shvrelmi[da_shmemsize];
   __shared__ double shvrelrg[da_shmemsize];
   __shared__ double shvrelpk[da_shmemsize];
   __shared__ double shvimrtb[da_shmemsize];
   __shared__ double shvimrix[da_shmemsize];
   __shared__ double shvimrmi[da_shmemsize];
   __shared__ double shvimrrg[da_shmemsize];
   __shared__ double shvimrpk[da_shmemsize];
   __shared__ double shvimltb[da_shmemsize];
   __shared__ double shvimlix[da_shmemsize];
   __shared__ double shvimlmi[da_shmemsize];
   __shared__ double shvimlrg[da_shmemsize];
   __shared__ double shvimlpk[da_shmemsize];
   __shared__ double prdrtb[da_shmemsize];
   __shared__ double prdrix[da_shmemsize];
   __shared__ double prdrmi[da_shmemsize];
   __shared__ double prdrrg[da_shmemsize];
   __shared__ double prdrpk[da_shmemsize];
   __shared__ double prdltb[da_shmemsize];
   __shared__ double prdlix[da_shmemsize];
   __shared__ double prdlmi[da_shmemsize];
   __shared__ double prdlrg[da_shmemsize];
   __shared__ double prdlpk[da_shmemsize];
   __shared__ double sumrtb[da_shmemsize];
   __shared__ double sumrix[da_shmemsize];
   __shared__ double sumrmi[da_shmemsize];
   __shared__ double sumrrg[da_shmemsize];
   __shared__ double sumrpk[da_shmemsize];
   __shared__ double sumltb[da_shmemsize];
   __shared__ double sumlix[da_shmemsize];
   __shared__ double sumlmi[da_shmemsize];
   __shared__ double sumlrg[da_shmemsize];
   __shared__ double sumlpk[da_shmemsize];

   shvrertb[j] = vrertb[j]; // reading real parts into shared memory
   shvrerix[j] = vrerix[j];
   shvrermi[j] = vrermi[j];
   shvrerrg[j] = vrerrg[j];
   shvrerpk[j] = vrerpk[j];
   shvreltb[j] = vreltb[j];
   shvrelix[j] = vrelix[j];
   shvrelmi[j] = vrelmi[j];
   shvrelrg[j] = vrelrg[j];
   shvrelpk[j] = vrelpk[j];
   shvimrtb[j] = vimrtb[j]; // reading imaginary parts into shared memory
   shvimrix[j] = vimrix[j];
   shvimrmi[j] = vimrmi[j];
   shvimrrg[j] = vimrrg[j];
   shvimrpk[j] = vimrpk[j];
   shvimltb[j] = vimltb[j];
   shvimlix[j] = vimlix[j];
   shvimlmi[j] = vimlmi[j];
   shvimlrg[j] = vimlrg[j];
   shvimlpk[j] = vimlpk[j];

   dag_sqr(shvrertb[j],shvrerix[j],shvrermi[j],shvrerrg[j],shvrerpk[j],
           shvreltb[j],shvrelix[j],shvrelmi[j],shvrelrg[j],shvrelpk[j],
            &sumrtb[j], &sumrix[j], &sumrmi[j], &sumrrg[j], &sumrpk[j],
            &sumltb[j], &sumlix[j], &sumlmi[j], &sumlrg[j], &sumlpk[j]);
   dag_sqr(shvimrtb[j],shvimrix[j],shvimrmi[j],shvimrrg[j],shvimrpk[j],
           shvimltb[j],shvimlix[j],shvimlmi[j],shvimlrg[j],shvimlpk[j],
            &prdrtb[j], &prdrix[j], &prdrmi[j], &prdrrg[j], &prdrpk[j],
            &prdltb[j], &prdlix[j], &prdlmi[j], &prdlrg[j], &prdlpk[j]);
   dag_inc(&sumrtb[j],&sumrix[j],&sumrmi[j],&sumrrg[j],&sumrpk[j],
           &sumltb[j],&sumlix[j],&sumlmi[j],&sumlrg[j],&sumlpk[j],
            prdrtb[j], prdrix[j], prdrmi[j], prdrrg[j], prdrpk[j],
            prdltb[j], prdlix[j], prdlmi[j], prdlrg[j], prdlpk[j]);

   __syncthreads();
   int powTwo = 1;                          // sum reduction
   for(int k=0; k < dimLog2; k++)
   {
      if((j%(powTwo*2)) == 0)
         if(j+powTwo < dim)
            dag_inc(&sumrtb[j],&sumrix[j],&sumrmi[j],&sumrrg[j],&sumrpk[j],
                    &sumltb[j],&sumlix[j],&sumlmi[j],&sumlrg[j],&sumlpk[j],
                     sumrtb[j+powTwo],sumrix[j+powTwo],sumrmi[j+powTwo],
                     sumrrg[j+powTwo],sumrpk[j+powTwo],
                     sumltb[j+powTwo],sumlix[j+powTwo],sumlmi[j+powTwo],
                     sumlrg[j+powTwo],sumlpk[j+powTwo]);
      powTwo = powTwo*2;
      __syncthreads();
   }
   // thread 0 computes the sqrt of the inner product, others wait
   if(j == 0)
      dag_sqrt( sumrtb[0], sumrix[0], sumrmi[0], sumrrg[0], sumrpk[0],
                sumltb[0], sumlix[0], sumlmi[0], sumlrg[0], sumlpk[0],
               &sumrtb[0],&sumrix[0],&sumrmi[0],&sumrrg[0],&sumrpk[0],
               &sumltb[0],&sumlix[0],&sumlmi[0],&sumlrg[0],&sumlpk[0]); 
   if(j == 0)
   {
      *normrtb = sumrtb[0];
      *normrix = sumrix[0];
      *normrmi = sumrmi[0];
      *normrrg = sumrrg[0];
      *normrpk = sumrpk[0];
      *normltb = sumltb[0];
      *normlix = sumlix[0];
      *normlmi = sumlmi[0];
      *normlrg = sumlrg[0];
      *normlpk = sumlpk[0];
   }
   __syncthreads();
   dag_div(shvrertb[j],shvrerix[j],shvrermi[j],shvrerrg[j],shvrerpk[j],
           shvreltb[j],shvrelix[j],shvrelmi[j],shvrelrg[j],shvrelpk[j],
             sumrtb[0],  sumrix[0],  sumrmi[0],  sumrrg[0],  sumrpk[0],
             sumltb[0],  sumlix[0],  sumlmi[0],  sumlrg[0],  sumlpk[0],
            &vrertb[j], &vrerix[j], &vrermi[j], &vrerrg[j], &vrerpk[j],
            &vreltb[j], &vrelix[j], &vrelmi[j], &vrelrg[j], &vrelpk[j]);
   dag_div(shvimrtb[j],shvimrix[j],shvimrmi[j],shvimrrg[j],shvimrpk[j],
           shvimltb[j],shvimlix[j],shvimlmi[j],shvimlrg[j],shvimlpk[j],
             sumrtb[0],  sumrix[0],  sumrmi[0],  sumrrg[0],  sumrpk[0],
             sumltb[0],  sumlix[0],  sumlmi[0],  sumlrg[0],  sumlpk[0],
            &vimrtb[j], &vimrix[j], &vimrmi[j], &vimrrg[j], &vimrpk[j],
            &vimltb[j], &vimlix[j], &vimlmi[j], &vimlrg[j], &vimlpk[j]);
}

__global__ void medium_normalize_vector
 ( double *vrertb, double *vrerix, double *vrermi, double *vrerrg,
   double *vrerpk, double *vreltb, double *vrelix, double *vrelmi,
   double *vrelrg, double *vrelpk,
   double *vimrtb, double *vimrix, double *vimrmi, double *vimrrg,
   double *vimrpk, double *vimltb, double *vimlix, double *vimlmi,
   double *vimlrg, double *vimlpk,
   int dim, int rnd, int rndLog2, int BS, int BSLog2,
   double *normrtb, double *normrix, double *normrmi, double *normrrg,
   double *normrpk, double *normltb, double *normlix, double *normlmi,
   double *normlrg, double *normlpk )
{
   *normrtb = 0.0;
}

__global__ void large_sum_the_squares
 ( double *vrertb, double *vrerix, double *vrermi, double *vrerrg,
   double *vrerpk, double *vreltb, double *vrelix, double *vrelmi,
   double *vrelrg, double *vrelpk,
   double *vimrtb, double *vimrix, double *vimrmi, double *vimrrg,
   double *vimrpk, double *vimltb, double *vimlix, double *vimlmi,
   double *vimlrg, double *vimlpk, int dim,
   double *sumsrtb, double *sumsrix, double *sumsrmi, double *sumsrrg,
   double *sumsrpk, double *sumsltb, double *sumslix, double *sumslmi,
   double *sumslrg, double *sumslpk, int BS, int BSLog2 )
{
   *sumsrtb = 0.0;
}

__global__ void large_normalize_vector
 ( double *vrertb, double *vrerix, double *vrermi, double *vrerrg,
   double *vrerpk, double *vreltb, double *vrelix, double *vrelmi,
   double *vrelrg, double *vrelpk,
   double *vimrtb, double *vimrix, double *vimrmi, double *vimrrg,
   double *vimrpk, double *vimltb, double *vimlix, double *vimlmi,
   double *vimlrg, double *vimlpk, int dim,
   double *sumsrtb, double *sumsrix, double *sumsrmi, double *sumsrrg,
   double *sumsrpk, double *sumsltb, double *sumslix, double *sumslmi,
   double *sumslrg, double *sumslpk,
   int nbsums, int nbsumsLog2, int BS,
   double *normrtb, double *normrix, double *normrmi, double *normrrg,
   double *normrpk, double *normltb, double *normlix, double *normlmi,
   double *normlrg, double *normlpk )
{
   *normrtb = 0.0;
}

void GPU_norm
 ( double *vrertb_h, double *vrerix_h, double *vrermi_h, double *vrerrg_h,
   double *vrerpk_h, double *vreltb_h, double *vrelix_h, double *vrelmi_h,
   double *vrelrg_h, double *vrelpk_h,
   double *vimrtb_h, double *vimrix_h, double *vimrmi_h, double *vimrrg_h,
   double *vimrpk_h, double *vimltb_h, double *vimlix_h, double *vimlmi_h,
   double *vimlrg_h, double *vimlpk_h, int dim, int freq, int BS,
   double *normrtb, double *normrix, double *normrmi, double *normrrg,
   double *normrpk, double *normltb, double *normlix, double *normlmi,
   double *normlrg, double *normlpk, int blocked )
{
   int BSLog2 = ceil(log2((double) BS)); // ceil for sum reduction

   double* vrertb_d;           // highest real parts on device
   double* vrerix_d;           // second highest real parts on device
   double* vrermi_d;           // third highest real parts on device
   double* vrerrg_d;           // fourth highest real parts on device
   double* vrerpk_d;           // fifth highest real parts on device
   double* vreltb_d;           // fifth lowest real parts on device
   double* vrelix_d;           // fourth lowest real parts on device
   double* vrelmi_d;           // third second lowest real parts on device
   double* vrelrg_d;           // second lowest real parts on device
   double* vrelpk_d;           // lowest real parts on device
   double* vimrtb_d;           // highest imaginary parts on device
   double* vimrix_d;           // second highest imaginary parts on device
   double* vimrmi_d;           // third highest imaginary parts on device
   double* vimrrg_d;           // fourth highest imaginary parts on device
   double* vimrpk_d;           // fifth highest imaginary parts on device
   double* vimltb_d;           // fifth lowest imaginary parts on device
   double* vimlix_d;           // fourth lowest imaginary parts on device
   double* vimlmi_d;           // third lowest imaginary parts on device
   double* vimlrg_d;           // second lowest imaginary parts on device
   double* vimlpk_d;           // lowest imaginary parts on device
   size_t size = dim*sizeof(double);
   hipMalloc((void**)&vrertb_d,size);
   hipMalloc((void**)&vrerix_d,size);
   hipMalloc((void**)&vrermi_d,size);
   hipMalloc((void**)&vrerrg_d,size);
   hipMalloc((void**)&vrerpk_d,size);
   hipMalloc((void**)&vreltb_d,size);
   hipMalloc((void**)&vrelix_d,size);
   hipMalloc((void**)&vrelmi_d,size);
   hipMalloc((void**)&vrelrg_d,size);
   hipMalloc((void**)&vrelpk_d,size);
   hipMalloc((void**)&vimrtb_d,size);
   hipMalloc((void**)&vimrix_d,size);
   hipMalloc((void**)&vimrmi_d,size);
   hipMalloc((void**)&vimrrg_d,size);
   hipMalloc((void**)&vimrpk_d,size);
   hipMalloc((void**)&vimltb_d,size);
   hipMalloc((void**)&vimlix_d,size);
   hipMalloc((void**)&vimlmi_d,size);
   hipMalloc((void**)&vimlrg_d,size);
   hipMalloc((void**)&vimlpk_d,size);
   hipMemcpy(vrertb_d,vrertb_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrerix_d,vrerix_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrermi_d,vrermi_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrerrg_d,vrerrg_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrerpk_d,vrerpk_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vreltb_d,vreltb_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrelix_d,vrelix_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrelmi_d,vrelmi_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrelrg_d,vrelrg_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vrelpk_d,vrelpk_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimrtb_d,vimrtb_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimrix_d,vimrix_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimrmi_d,vimrmi_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimrrg_d,vimrrg_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimrpk_d,vimrpk_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimltb_d,vimltb_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimlix_d,vimlix_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimlmi_d,vimlmi_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimlrg_d,vimlrg_h,size,hipMemcpyHostToDevice);
   hipMemcpy(vimlpk_d,vimlpk_h,size,hipMemcpyHostToDevice);
   double* normrtb_d;
   double* normrix_d;
   double* normrmi_d;
   double* normrrg_d;
   double* normrpk_d;
   double* normltb_d;
   double* normlix_d;
   double* normlmi_d;
   double* normlrg_d;
   double* normlpk_d;
   hipMalloc((void**)&normrtb_d,sizeof(double));
   hipMalloc((void**)&normrix_d,sizeof(double));
   hipMalloc((void**)&normrmi_d,sizeof(double));
   hipMalloc((void**)&normrrg_d,sizeof(double));
   hipMalloc((void**)&normrpk_d,sizeof(double));
   hipMalloc((void**)&normltb_d,sizeof(double));
   hipMalloc((void**)&normlix_d,sizeof(double));
   hipMalloc((void**)&normlmi_d,sizeof(double));
   hipMalloc((void**)&normlrg_d,sizeof(double));
   hipMalloc((void**)&normlpk_d,sizeof(double));

   if(dim == BS)
   {
      for(int i=0; i<freq; i++)
         small_normalize_vector<<<1,BS>>>
            (vrertb_d,vrerix_d,vrermi_d,vrerrg_d,vrerpk_d,
             vreltb_d,vrelix_d,vrelmi_d,vrelrg_d,vrelpk_d,
             vimrtb_d,vimrix_d,vimrmi_d,vimrrg_d,vimrpk_d,
             vimltb_d,vimlix_d,vimlmi_d,vimlrg_d,vimlpk_d,
             dim,BSLog2,
             normrtb_d,normrix_d,normrmi_d,normrrg_d,normrpk_d,
             normltb_d,normlix_d,normlmi_d,normlrg_d,normlpk_d);
   }
   else if(blocked == 0)
   {
      int rf = ceil(((double) dim)/BS);
      int rfLog2 = ceil(log2((double) rf));
      for(int i=0; i<freq; i++)
         medium_normalize_vector<<<1,BS>>>
            (vrertb_d,vrerix_d,vrermi_d,vrerrg_d,vrerpk_d,
             vreltb_d,vrelix_d,vrelmi_d,vrelrg_d,vrelpk_d,
             vimrtb_d,vimrix_d,vimrmi_d,vimrrg_d,vimrpk_d,
             vimltb_d,vimlix_d,vimlmi_d,vimlrg_d,vimlpk_d,dim,
             rf,rfLog2,BS,BSLog2,
             normrtb_d,normrix_d,normrmi_d,normrrg_d,normrpk_d,
             normltb_d,normlix_d,normlmi_d,normlrg_d,normlpk_d);
   }
   else
   {
      const int nblocks = dim/BS;
      const int nblocksLog2 = ceil(log2((double) nblocks));
      double* sumsrtb_d; // highest parts of sums of squares for each block
      double* sumsrix_d; // 2nd highest parts of sums of squares
      double* sumsrmi_d; // middle parts of sums of squares
      double* sumsrrg_d; // 2nd lowest parts of sums of squares for each block
      double* sumsrpk_d; // lowest parts of sums of squares for each block
      double* sumsltb_d; // highest parts of sums of squares for each block
      double* sumslix_d; // 2nd highest parts of sums of squares
      double* sumslmi_d; // middle parts of sums of squares
      double* sumslrg_d; // 2nd lowest parts of sums of squares for each block
      double* sumslpk_d; // lowest parts of sums of squares for each block
      size_t sums_size = nblocks*sizeof(double);
      hipMalloc((void**)&sumsrtb_d,sums_size);
      hipMalloc((void**)&sumsrix_d,sums_size);
      hipMalloc((void**)&sumsrmi_d,sums_size);
      hipMalloc((void**)&sumsrrg_d,sums_size);
      hipMalloc((void**)&sumsrpk_d,sums_size);
      hipMalloc((void**)&sumsltb_d,sums_size);
      hipMalloc((void**)&sumslix_d,sums_size);
      hipMalloc((void**)&sumslmi_d,sums_size);
      hipMalloc((void**)&sumslrg_d,sums_size);
      hipMalloc((void**)&sumslpk_d,sums_size);
      for(int i=0; i<freq; i++)
      {
         large_sum_the_squares<<<nblocks,BS>>>
            (vrertb_d,vrerix_d,vrermi_d,vrerrg_d,vrerpk_d,
             vreltb_d,vrelix_d,vrelmi_d,vrelrg_d,vrelpk_d,
             vimrtb_d,vimrix_d,vimrmi_d,vimrrg_d,vimrpk_d,
             vimltb_d,vimlix_d,vimlmi_d,vimlrg_d,vimlpk_d,dim,
             sumsrtb_d,sumsrix_d,sumsrmi_d,sumsrrg_d,sumsrpk_d,
             sumsltb_d,sumslix_d,sumslmi_d,sumslrg_d,sumslpk_d,BS,BSLog2);
         large_normalize_vector<<<nblocks,BS>>>
            (vrertb_d,vrerix_d,vrermi_d,vrerrg_d,vrerpk_d,
             vreltb_d,vrelix_d,vrelmi_d,vrelrg_d,vrelpk_d,
             vimrtb_d,vimrix_d,vimrmi_d,vimrrg_d,vimrpk_d,
             vimltb_d,vimlix_d,vimlmi_d,vimlrg_d,vimlpk_d,dim,
             sumsrtb_d,sumsrix_d,sumsrmi_d,sumsrrg_d,sumsrpk_d,
             sumsltb_d,sumslix_d,sumslmi_d,sumslrg_d,sumslpk_d,
             nblocks,nblocksLog2,BS,
             normrtb_d,normrix_d,normrmi_d,normrrg_d,normrpk_d,
             normltb_d,normlix_d,normlmi_d,normlrg_d,normlpk_d);
      }
   }
   hipMemcpy(vrertb_h,vrertb_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrerix_h,vrerix_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrermi_h,vrermi_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrerrg_h,vrerrg_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrerpk_h,vrerpk_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vreltb_h,vreltb_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrelix_h,vrelix_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrelmi_h,vrelmi_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrelrg_h,vrelrg_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vrelpk_h,vrelpk_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimrtb_h,vimrtb_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimrix_h,vimrix_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimrmi_h,vimrmi_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimrrg_h,vimrrg_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimrpk_h,vimrpk_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimltb_h,vimltb_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimlix_h,vimlix_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimlmi_h,vimlmi_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimlrg_h,vimlrg_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(vimlpk_h,vimlpk_d,size,hipMemcpyDeviceToHost);
   hipMemcpy(normrtb,normrtb_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normrix,normrix_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normrmi,normrmi_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normrrg,normrrg_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normrpk,normrpk_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normltb,normltb_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normlix,normlix_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normlmi,normlmi_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normlrg,normlrg_d,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(normlpk,normlpk_d,sizeof(double),hipMemcpyDeviceToHost);
}
